#include "hip/hip_runtime.h"
/*
PARALLEL KERNELS are classified as simple and special

(^_^) All simple kernels are here 

Some of the kernels in parallel simplex are specialized and slightly complicated,
they are stored in a separate module for cleanliness, follow the
usage in uv_model_parallel.cu (aka parent) file. 

FYI Directly reviewing this file wouldn't make sense. If you see a kernel in parent
You'll either find it here or there's a comment that would take you to the 
appropriate place
*/

#include <iostream>

#include "parallel_structs.h"

#ifndef KERNELS
#define KERNELS

// ##################################################
// PREPROCESS and POSTPROCESS  >>
// ##################################################

/*
Kernel to convert float cost matrix to the MatrixCell objects
*/
__global__ void createCostMatrix(MatrixCell *d_costMtx, float * d_costs_ptr, int n_supplies, int n_demands)
{

    int d = blockIdx.x * blockDim.x + threadIdx.x;
    int s = blockIdx.y * blockDim.y + threadIdx.y;

    if (s < n_supplies && d < n_demands)
    {
        int id = s * n_demands + d;
        MatrixCell _c = {.row = s, .col = d, .cost = d_costs_ptr[id]};
        d_costMtx[id] = _c;
    }
}

/*
Generate a tree on the global memory using the initial set of feasible flows
*/
__global__ void create_initial_tree(flowInformation * d_flows_ptr, int * d_adjMtx_ptr, float * d_flowMtx_ptr,
    int numSupplies, int numDemands)
{
    
    int V = numSupplies+numDemands;
    int gid = blockIdx.x*blockDim.x + threadIdx.x;

    if (gid < V - 1) {
    
        flowInformation _this_flow = d_flows_ptr[gid];
        int row = _this_flow.source;
        int column =  _this_flow.destination;
        int idx = TREE_LOOKUP(row, numSupplies+column, V); // Index in adjacency matrix
        float _qty = 1.0*_this_flow.qty;
        if (_qty==0){
            // Handling degeneracy - Flow purturbation
            _qty=epsilon;
        }
        d_flowMtx_ptr[gid] = _qty;
        d_adjMtx_ptr[idx] = gid+1;
    }
}

/*
Reverse operation of generating a tree from the feasible flows - unordered allocation
*/
__global__ void retrieve_final_tree(flowInformation * d_flows_ptr, int * d_adjMtx_ptr, float * d_flowMtx_ptr,
        int numSupplies, int numDemands) 
{

    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;
    int V = numSupplies+numDemands;
    
    // Upper triangle scope of adj matrix
    if (col_indx < V && col_indx >= numSupplies && row_indx < numSupplies) {
        
        // Check if this is a flow edge - 
        int gid = TREE_LOOKUP(row_indx, col_indx, V);
        int flow_id = d_adjMtx_ptr[gid];
        if (flow_id > 0) {

            flowInformation _this_flow;
            _this_flow.qty = round(d_flowMtx_ptr[flow_id - 1]);
            _this_flow.source = row_indx;
            _this_flow.destination = col_indx - numSupplies;
            d_flows_ptr[flow_id - 1] = _this_flow;

        }
    }
}


/* 
Transfer flows on device and prepare an adjacency and flow matrix using the flows from IBFS
In case of sequencial pivoting - one would need a copy of adjMatrix on the host to traverse the graph
*/
__host__ void create_IBF_tree_on_host_device(flowInformation * feasible_flows,
    int ** d_adjMtx_ptr, int ** h_adjMtx_ptr, float ** d_flowMtx_ptr, float ** h_flowMtx_ptr, 
    int numSupplies, int numDemands) 
{
    int V = numSupplies+numDemands;
    int _utm_entries = (V*(V+1))/2; // Number of entries in upper triangular matrix 

    gpuErrchk(hipMalloc((void **) d_adjMtx_ptr, sizeof(int)*_utm_entries)); 
    thrust::fill(thrust::device, *d_adjMtx_ptr, (*d_adjMtx_ptr) + _utm_entries, 0);

    gpuErrchk(hipMalloc((void **) d_flowMtx_ptr, sizeof(float)*(V-1)));
    thrust::fill(thrust::device, *d_flowMtx_ptr, (*d_flowMtx_ptr) + (V-1), 0);

    // Make a replica of feasible flows on device
    flowInformation * d_flows_ptr;
    gpuErrchk(hipMalloc((void **) &d_flows_ptr, sizeof(flowInformation)*(V-1)));
    gpuErrchk(hipMemcpy(d_flows_ptr, feasible_flows, sizeof(flowInformation)*(V-1), hipMemcpyHostToDevice));

    // Small kernel to parallely create a tree using the flows
    create_initial_tree <<< ceil(1.0*(V-1)/blockSize), blockSize >>> (d_flows_ptr, *d_adjMtx_ptr, *d_flowMtx_ptr, numSupplies, numDemands);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
    // Now device_flows are useless; 
    // All information about graph is now contained within d_adjMatrix, d_flowMatrix on device =>
    gpuErrchk(hipFree(d_flows_ptr));
    
    // Make a copy on host >>
    *h_adjMtx_ptr = (int *) malloc(sizeof(int)*(_utm_entries));
    gpuErrchk(hipMemcpy(*h_adjMtx_ptr, *d_adjMtx_ptr, sizeof(int)*(_utm_entries), hipMemcpyDeviceToHost));
    *h_flowMtx_ptr = (float *) malloc(sizeof(float)*(V-1));
    gpuErrchk(hipMemcpy(*h_flowMtx_ptr, *d_flowMtx_ptr, sizeof(float)*(V-1), hipMemcpyDeviceToHost));
}

/*
Given a feasible tree on device, load a feasible solution to transportation problem on the host
*/
__host__ void retrieve_solution_on_current_tree(flowInformation * feasible_flows, int * d_adjMtx_ptr, float * d_flowMtx_ptr, 
    int &active_flows, int numSupplies, int numDemands)
{
    // Recreate device flows using the current adjMatrix
    flowInformation default_flow;
    default_flow.qty = 0;

    flowInformation * d_flows_ptr;
    gpuErrchk(hipMalloc((void **) &d_flows_ptr, sizeof(flowInformation)*(numSupplies*numDemands)));
    thrust::fill(thrust::device, d_flows_ptr, d_flows_ptr + (numSupplies*numDemands), default_flow);

    dim3 __blockDim(blockSize, blockSize, 1);
    int grid_size = ceil(1.0*(numSupplies+numDemands)/blockSize); // VxV threads
    dim3 __gridDim(grid_size, grid_size, 1);
    retrieve_final_tree <<< __gridDim, __blockDim >>> (d_flows_ptr, d_adjMtx_ptr, d_flowMtx_ptr, numSupplies, numDemands);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
    // Copy the (flows > 0) back on the host >>
    auto flow_end = thrust::remove_if(thrust::device,
        d_flows_ptr, d_flows_ptr + (numSupplies*numDemands), is_zero());
    int flow_count = flow_end - d_flows_ptr;
    // Update active flows in result 
    active_flows = flow_count;
    gpuErrchk(hipMemcpy(feasible_flows, d_flows_ptr, (flow_count)*sizeof(flowInformation), hipMemcpyDeviceToHost));

}


// ##################################################
// SOLVING DUAL >>
// ##################################################

/*
APPROACH 1 :
Kernels concerned with solving the UV System using a BFS Traversal Approach
*/

__global__ void copy_row_shadow_prices(Variable * U_vars, float * u_vars_ptr, int numSupplies) 
{    
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    if (gid < numSupplies) {
        u_vars_ptr[gid] = U_vars[gid].value;
    }
}

__global__ void copy_col_shadow_prices(Variable * V_vars, float * v_vars_ptr, int numDemands) 
{
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    if (gid < numDemands) {
        v_vars_ptr[gid] = V_vars[gid].value;
    }
}

__global__ void initialize_U_vars(Variable * U_vars, int numSupplies) {
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    Variable default_var;
    if (gid < numSupplies) {
        U_vars[gid] = default_var;
    }
}

__global__ void initialize_V_vars(Variable * V_vars, int numDemands) {
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    Variable default_var;
    if (gid < numDemands) {
        V_vars[gid] = default_var;
    }
}

/*
Breadth First Traversal on UV
*/
__global__ void assign_next(int * d_adjMtx_ptr, float * d_costs_ptr, 
    Variable *u_vars, Variable *v_vars, int numSupplies, int numDemands) {
    
    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;
    int V = numSupplies + numDemands;

    // Within the scope of the adj matrix
    if (row_indx < numSupplies && col_indx < numDemands) {
        // Check if these are adjacent - (checks in upper triangular matrix, because row < adj-col-index)
        int indx = TREE_LOOKUP(row_indx, col_indx + numSupplies, V); // Adjusted destination vertex ID
        if (d_adjMtx_ptr[indx] > 0) {

            Variable u_i = u_vars[row_indx];
            Variable v_j = v_vars[col_indx];
            
            // Check if any of the u or v has not been assigned and adjacent is assigned - then assign it
            if (u_vars[row_indx].assigned && (!v_vars[col_indx].assigned)) {
                // In this case >> v_j = c_ij - u_i
                Variable var;
                var = d_costs_ptr[row_indx*numDemands+col_indx] - u_vars[row_indx].value;
                // var.assigned = true;
                v_vars[col_indx] = var;
            }
            else if ((!u_vars[row_indx].assigned) && v_vars[col_indx].assigned) {
                // In this case >> u_j = c_ij - v_j
                Variable var;
                var = d_costs_ptr[row_indx*numDemands+col_indx] -  v_vars[col_indx].value;
                // var.assigned = true;
                u_vars[row_indx] = var;
            }
        }
    }
}

// Credits: https://github.com/siddharths2710/cuda_bfs/blob/master/cuda_bfs/kernel.cu
__global__ void CUDA_BFS_KERNEL(int * start, int * length, int *Ea, bool * Fa, bool * Xa, 
        float * variables, float * d_costs_ptr, bool * done, int numSupplies, int numDemands, int V)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > V)
		*done = false;

	if (Fa[id] == true && Xa[id] == false)
	{
		// printf("%d ", id); //This printf gives the order of vertices in BFS	
		Fa[id] = false;
		Xa[id] = true;
		__syncthreads(); 
		int k = 0;
		int start_ptr = start[id];
		int end_ptr = start_ptr + length[id];
		for (int i = start_ptr; i < end_ptr; i++) 
		{
			int nid = Ea[i];
			if (Xa[nid] == false)
			{       
                int row_indx = min(nid, id);
                int col_indx = max(nid, id) - numSupplies;
				variables[nid] = d_costs_ptr[row_indx*numDemands+col_indx] - variables[id];
				Fa[nid] = true;
				*done = false;
			}
		}
	}
}

__global__ void determine_length(int * length, int * d_adjMtx_ptr, int V) {
        int L = 0;
        int i = blockIdx.x *blockDim.x + threadIdx.x;
        // No data re-use (this is a straight fwd kernel)
        if (i < V) 
        {    
                for (int j=0; j<V; j++) {
                        int idx = TREE_LOOKUP(i, j, V);
                        if (d_adjMtx_ptr[idx] > 0) {
                                L++;
                        }
                }
                length[i+1] = L;
                length[0] = 0;
        }
}

__global__ void fill_Ea(int * start, int * Ea, int * d_adjMtx_ptr, int V, int numSupplies) {
        int i = blockIdx.x*blockDim.x + threadIdx.x;
        int offset = start[i];
        int L = 0;
        if (i < V) {
                for (int j=0; j<V; j++) {
                        int idx = TREE_LOOKUP(i, j, V);
                        if (d_adjMtx_ptr[idx] > 0) {
                                Ea[offset + L] = j;
                                L++;
                        }
                }
        }
}


/*
APPROACH 2:
Kernels concerned with solving the UV System using a using a matrix solver
*/

// Custom Fill kernel for csr row pointers
__global__ void fill_csr_offset (int * d_csr_offsets, int length) {
        
        int idx = blockIdx.x*blockDim.x + threadIdx.x;
        if (idx < length) {
                if (idx == 0) {
                        d_csr_offsets[idx] = 0;
                }
                else {
                        d_csr_offsets[idx] = 2*idx - 1; 
                }
        }
}

/*
Create a dense linear system in parallel by looking at current feasible tree 
*/
__global__ void initialize_dense_u_v_system(float * d_A, float * d_b, int * d_adjMtx_ptr, 
    float * d_costs_ptr, int numSupplies, int numDemands) {
        
    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;
    int V = numSupplies + numDemands;

    if (row_indx < numSupplies && col_indx < numDemands) {
        int indx = TREE_LOOKUP(row_indx, col_indx + numSupplies, V); // Adjusted destination vertex ID
        int flow_indx = d_adjMtx_ptr[indx];
        if (flow_indx > 0) {
            // This is a flow - flow_indx = row_number, u = row_number, v = col_number
            d_A[flow_indx * V + row_indx] = 1;
            d_A[flow_indx * V + numSupplies + col_indx] = 1;
            d_b[flow_indx] = d_costs_ptr[row_indx*numDemands + col_indx];
        }
    }
}

/*
Create a sparse linear system in parallel by looking at current feasible tree 
*/
__global__ void initialize_sparse_u_v_system(int * d_csr_columns, float * d_b, int * d_adjMtx_ptr, 
    float * d_costs_ptr, int numSupplies, int numDemands) {
        
    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;
    int V = numSupplies + numDemands;

    if (row_indx < numSupplies && col_indx < numDemands) {
        int indx = TREE_LOOKUP(row_indx, col_indx + numSupplies, V); // Adjusted destination vertex ID
        int flow_indx = d_adjMtx_ptr[indx];
        if (flow_indx > 0) {
            // This is a flow - flow_indx = row_number, u = row_number, v = col_number
            d_csr_columns[2*flow_indx-1] = row_indx;
            d_csr_columns[2*flow_indx] = numSupplies + col_indx;
            d_b[flow_indx] = d_costs_ptr[row_indx*numDemands + col_indx];
        }
    }
}

/*
Load the solution of system to the appropriate place
*/
__global__ void retrieve_uv_solution(float * d_x, float * u_vars_ptr, float * v_vars_ptr, int numSupplies, int numDemands) 
{
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    int V = numSupplies + numDemands;
    if (gid < V) {
        if (gid < numSupplies) {
            u_vars_ptr[gid] = d_x[gid];
        } 
        else {
            v_vars_ptr[gid - numSupplies] = d_x[gid];
        }
    }
}

// ##################################################
// COMPUTING REDUCED COSTS >>
// ##################################################


/*
Kernel to compute Reduced Costs in the transportation table
*/
__global__ void computeReducedCosts(float * u_vars_ptr, float * v_vars_ptr, float * d_costs_ptr, float * d_reducedCosts_ptr, 
    int numSupplies, int numDemands)
{

        int row_indx = blockIdx.y*blockDim.y + threadIdx.y;
        int col_indx = blockIdx.x*blockDim.x + threadIdx.x;

        if (row_indx < numSupplies && col_indx < numDemands) {
            // r =  C_ij - (u_i + v_j);
            float r = d_costs_ptr[row_indx*numDemands+col_indx] - u_vars_ptr[row_indx] - v_vars_ptr[col_indx];
            d_reducedCosts_ptr[row_indx*numDemands+col_indx] = r;
        }
}

#endif