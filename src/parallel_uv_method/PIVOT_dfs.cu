#include "hip/hip_runtime.h"
#include "PIVOT_dfs.h"

/* 
Setup necessary resources for pivoting 
these resources are static and to be shared/overwritten between iterations
*/
__host__ void initialize_device_PIVOT(int ** backtracker, stackNode ** stack, bool ** visited, 
    int ** depth, float ** loop_minimum, int ** loop_min_from, int ** loop_min_to, int ** loop_min_id,
    vertex_conflicts ** v_conflicts, int numSupplies, int numDemands) {

    int V = numSupplies + numDemands;

    if (PIVOTING_STRATEGY=="sequencial_dfs") {

        // Pivoting requires some book-keeping (for the DFS procedure)
        // BOOK 1: Stores the routes discovered for each thread -
        *backtracker = (int *) malloc(sizeof(int)*V);    
        // BOOK 2: Stores the runtime stack for DFS running on each thread
        *stack = (stackNode *) malloc(sizeof(stackNode)*V);    
        // BOOK 3: Keeps a track if any vertex was visited during DFS for each thread
        *visited = (bool *) malloc(sizeof(bool)*V);
    }
    
    else if (PIVOTING_STRATEGY == "parallel_dfs") {

        // Allocate appropriate resources, Specific to parallel pivot >>
        int num_threads_launching = NUM_THREADS_LAUNCHING(numSupplies, numDemands, PARALLEL_PIVOT_IDEA);
        gpuErrchk(hipMalloc((void **) backtracker, num_threads_launching * V * sizeof(int)));
        gpuErrchk(hipMalloc((void **) stack, num_threads_launching * V * sizeof(stackNode)));
        gpuErrchk(hipMalloc((void **) visited, num_threads_launching * V * sizeof(bool)));
        gpuErrchk(hipMalloc((void **) depth, num_threads_launching * sizeof(int)));
        gpuErrchk(hipMalloc((void **) v_conflicts, numSupplies * numDemands * sizeof(vertex_conflicts)));

        // In hybrid pivoting this may not be required >> 

        // gpuErrchk(hipMalloc((void **) loop_minimum, num_threads_launching * sizeof(float)));
        // gpuErrchk(hipMalloc((void **) loop_min_from, num_threads_launching * sizeof(int)));
        // gpuErrchk(hipMalloc((void **) loop_min_to, num_threads_launching * sizeof(int)));
        // gpuErrchk(hipMalloc((void **) loop_min_id, num_threads_launching * sizeof(int)));
    }
}

/* 
Free up acquired resources for pivoting on host device 
*/
__host__ void terminate_device_PIVOT(int * backtracker, stackNode * stack, bool * visited, 
    int * depth, float * loop_minimum, int * loop_min_from, int * loop_min_to, int * loop_min_id,
    vertex_conflicts * v_conflicts) {

    if (PIVOTING_STRATEGY == "sequencial_dfs") {
        
        free(backtracker);
        free(stack);
        free(visited);
    
    }

    else if (PIVOTING_STRATEGY == "parallel_dfs")
    {

        // Free up space >>
        gpuErrchk(hipFree(backtracker));
        gpuErrchk(hipFree(stack));
        gpuErrchk(hipFree(visited));
        gpuErrchk(hipFree(depth));
        gpuErrchk(hipFree(v_conflicts));

        // gpuErrchk(hipFree(loop_minimum));
        // gpuErrchk(hipFree(loop_min_from));
        // gpuErrchk(hipFree(loop_min_to));
        // gpuErrchk(hipFree(loop_min_id)); 
    }
}

/*
Push a node in the provided stack
*/
__host__ __device__ void stack_push(stackNode * stack, int &stack_top, int vtx, int depth)
{
    stack_top++;
    stackNode node = {.index = vtx, .depth = depth};
    stack[stack_top] = node;
}

/*
Pop a node from the provided stack
*/
__host__ __device__ stackNode stack_pop(stackNode * stack, int &stack_top)
{
    stackNode vtx;
    vtx = stack[stack_top];
    stack_top--;
    return vtx;
}

/*
Perform depth first search looking for route to execute the pivot
*/
__host__ __device__ void perform_dfs_sequencial_on_i(int * adjMtx, int * vertex_start, int * vertex_degree, int * adjVertices, 
        stackNode * stack, int * backtracker, bool * visited, 
        int * depth, int starting_vertex, int target_vertex, int V)
{   
    
    int key, current_depth = 1, stack_top = -1;
    stackNode current_vertex;
    stack_push(stack, stack_top, starting_vertex, current_depth);

    while(!(stack_top == -1))
    {
        current_vertex = stack_pop(stack, stack_top);

        // check if current vtx has been already visited in this search
        if (!visited[current_vertex.index])
        {
            // if not visited: >> 
            //  - mark this as visited 
            //  - see if current_vertex is adj to the starting point, 
            //        if not - queue the vertices that are adjacent to current vertex, increment depth
            visited[current_vertex.index]=true;

            // Do the book-keeping
            current_depth = current_vertex.depth + 1;
            backtracker[current_vertex.depth] = current_vertex.index;

            // check if target point is adjacent
            key = TREE_LOOKUP(target_vertex, current_vertex.index, V);
            if (adjMtx[key] > 0 && current_depth > 1)
            {
                // Leads back to origin - this completes the cycle - exit the loop
                *depth = current_depth;
                break;
            }
            else
            {
                // Append the ajacent nodes in stack
                int _s = vertex_start[current_vertex.index];
                for (int j = _s; j < _s + vertex_degree[current_vertex.index]; j++)
                {
                    stack_push(stack, stack_top, adjVertices[j], current_depth);
                }
            }
            
        }
        // else - move to next vertex : pop_next, Before that >>
        // Iterations have explored the childeren and now going up in the recursion tree 
        // to something that is still pending to be explored -
        if (stack_top == -1)
        {
            *depth=1;
        }
    }
}


/*
Replaces the exiting basic flow with entering non basic flow
Does the necessary adjustments on the variables on device memory
*/
__host__ void exit_i_and_enter_j(int * d_adjMtx_ptr, float * d_flowMtx_ptr, int exit_src, int exit_dest, 
        int enter_src, int enter_dest, int min_flow_indx, float min_flow, int V) {
            
    int id;
    int null_value = 0;
    int new_value = min_flow_indx + 1;

    // Set value for exiting in d
    id = TREE_LOOKUP(exit_src, exit_dest, V);
    gpuErrchk(hipMemcpy(&d_adjMtx_ptr[id], &null_value, sizeof(int), hipMemcpyHostToDevice));

    // Set value for entering to the appropriate
    id = TREE_LOOKUP(enter_src, enter_dest, V);
    gpuErrchk(hipMemcpy(&d_adjMtx_ptr[id], &new_value, sizeof(int), hipMemcpyHostToDevice));

    // The flow would have become zero - update it again
    // gpuErrchk(hipMemcpy(&d_flowMtx_ptr[min_flow_indx], &min_flow, sizeof(float), hipMemcpyHostToDevice));

}

/*
Do a copy from new value to device pointer
*/
__host__ void modify_flowMtx_on_device(float * d_flowMtx_ptr, int id, float new_value) {
    gpuErrchk(hipMemcpy(&d_flowMtx_ptr[id], &new_value, sizeof(float), hipMemcpyHostToDevice));
}

__host__ void do_flow_adjustment_on_host_device(int * h_adjMtx_ptr, float * h_flowMtx_ptr, 
        int * d_adjMtx_ptr, float * d_flowMtx_ptr, int * backtracker, float min_flow, int min_from, int min_to, int min_flow_id,
        int pivot_row, int pivot_col, int depth, int V, int numSupplies, int numDemands) {

    
    /* *************************** 
        DEBUG UTILITY // Print the discovered loop and pivoting parameters
    **************************** */
    // std::cout<<"Pivot Row : "<<pivot_row<<std::endl;
    // std::cout<<"Pivot Col : "<<pivot_col<<std::endl;
    // std::cout<<" ************** LOOP"<<std::endl;
    // std::cout<<"Depth : "<<depth<<" : ";
    // for (int j = 0; j < depth + 1; j++) {
    //     std::cout<<backtracker[j]<<" ";
    // }
    // std::cout<<std::endl<<" ************** PIVOT"<<std::endl;
    // std::cout<<"Min flow : "<<min_flow<<std::endl;
    // std::cout<<"Min from : "<<min_from<<std::endl;
    // std::cout<<"Min to : "<<min_to<<std::endl;
    // std::cout<<"Min index : "<<min_flow_id<<std::endl;
 
    int _from, _to, id;
    float _flow;

    for (int i=1; i<depth; i++) 
    {
        _from = backtracker[i];
        _to = backtracker[i+1];
        id = h_adjMtx_ptr[TREE_LOOKUP(_from, _to, V)] - 1;
        _flow = ((int) pow(-1, (int)i%2))*min_flow;
        h_flowMtx_ptr[id] += _flow;
        // modify_flowMtx_on_device(d_flowMtx_ptr, id, h_flowMtx_ptr[id]);
    }

    // Do the replacment between exiting i - entering j on both host and device
    // Remove edge
    id = TREE_LOOKUP(min_from, min_to, V);
    h_adjMtx_ptr[id] = 0;
    // Insert edge
    id = TREE_LOOKUP(pivot_row, pivot_col+ numSupplies, V);
    h_adjMtx_ptr[id] = min_flow_id + 1;
    // Update new flow 
    h_flowMtx_ptr[min_flow_id] = min_flow;

    // Communicate device about the removal and addition of an extry-exit variable pair
    exit_i_and_enter_j(d_adjMtx_ptr, d_flowMtx_ptr, 
        min_from, min_to, 
        pivot_row, pivot_col + numSupplies, 
        min_flow_id, min_flow, V);
}

__host__ void execute_pivot_on_host_device(int * h_adjMtx_ptr, float * h_flowMtx_ptr, 
        int * d_adjMtx_ptr, float * d_flowMtx_ptr, int * backtracker, 
        int pivot_row, int pivot_col, int depth, int V, int numSupplies, int numDemands) {

    // *******************************************
    // STEP: Performing the pivot operation 
        // Step 1 - Find the Minimum flow
        // Step 2 - Adjust the Flow
    // *******************************************
            
    int id, _from = -1, _to = -1, min_flow_id = -1, min_from = -1, min_to = -1;
    float _flow, min_flow = INT_MAX;

    // ########### STEP 1 | Finding the minimum flow >>
    // Traverse the loop find the minimum flow that could be increased
    // on the incoming edge >> 
    for (int i=0; i<depth; i++) 
    {
        if (i%2==1) 
        {
            _from = backtracker[i];
            _to = backtracker[i+1];
            id = h_adjMtx_ptr[TREE_LOOKUP(_from, _to, V)] - 1;
            _flow = h_flowMtx_ptr[id];
            
            if (_flow < min_flow) 
            {
                min_flow = _flow;
                min_flow_id = id;
                min_from = _from;
                min_to = _to;
            }
        }
    }

    // ########### STEP 2 | Executing the flow adjustment >>

    // Skip the first edge (entering edge)
    // Exiting Edge will become automatically zero (min_from, min_to)
    // Note - minflow value is zero if there's a degenerate pivot!
    do_flow_adjustment_on_host_device(h_adjMtx_ptr, h_flowMtx_ptr, d_adjMtx_ptr, d_flowMtx_ptr, backtracker,
            min_flow, min_from, min_to, min_flow_id,
            pivot_row, pivot_col, depth, V, numSupplies, numDemands);

}

__host__ void perform_a_sequencial_pivot(int * backtracker, stackNode * stack, bool * visited,
    int * h_vertex_start, int * h_vertex_degree, int * h_adjVertices, 
    int * h_adjMtx_ptr, float * h_flowMtx_ptr, 
    int * d_vertex_start, int * d_vertex_degree, int * d_adjVertices,
    int * d_adjMtx_ptr, float * d_flowMtx_ptr,
    bool &result, int pivot_row, int pivot_col, 
    double &dfs_time, double &resolve_time, double &adjustment_time,
    int numSupplies, int numDemands) {
    
    // std::cout<<"Pivot Row : "<<pivot_row<<std::endl;
    // std::cout<<"Pivot Col : "<<pivot_col<<std::endl;

    auto _pivot_start = std::chrono::high_resolution_clock::now();
    auto _pivot_end = std::chrono::high_resolution_clock::now();
    auto _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    // *******************************************
    // STEP: Traverse tree and find a cycle
    // *******************************************
    int V = numSupplies+numDemands;
    int _depth = 1; // Stores length of cycle discovered for each thread
    backtracker[0] = pivot_row;
    memset(visited, 0, V*sizeof(bool));

    // Find a path by performing DFS from pivot_col reaching pivot row to complete cycle >>
    // SEQUENCIAL PROCEDURE to find An incoming edge to vertex = pivot_row from vertex = numSupplies + pivot_col        
    _pivot_start = std::chrono::high_resolution_clock::now();

    perform_dfs_sequencial_on_i(h_adjMtx_ptr, h_vertex_start, h_vertex_degree, h_adjVertices, 
        stack, backtracker, visited, &_depth, 
        pivot_col+numSupplies, pivot_row, V);
    
    _pivot_end = std::chrono::high_resolution_clock::now();
    _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    dfs_time += _pivot_duration.count();

    // If loop not discovered >>
    _pivot_start = std::chrono::high_resolution_clock::now();

    if (_depth <= 1) {
        
        std::cout<<" !! Error !! : Pivot cannot be performed, this is probably not a tree but forest!"<<std::endl;
        std::cout<<"Solution IS NOT OPTIMAL!"<<std::endl;
        result = true;
        return;
    
    }
    // Cycle was discovered and stored in backtracker array
    else {
        // *******************************************
        // STEP : Performing the pivot operation 
        // *******************************************

        backtracker[_depth] = pivot_row;

        // std::cout<<"Printing Cycle :: [ ";
        // for (int i=0; i<= _depth; i++){
        //     std::cout<<backtracker[i]<<", ";         
        // }
        // std::cout<<"]"<<std::endl;
        // exit(0);

        execute_pivot_on_host_device(h_adjMtx_ptr, h_flowMtx_ptr, d_adjMtx_ptr, d_flowMtx_ptr, backtracker,
            pivot_row, pivot_col, _depth, V, numSupplies, numDemands);
    }
    _pivot_end = std::chrono::high_resolution_clock::now();
    _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    adjustment_time += _pivot_duration.count();
}

/*
KERNEL 1 =>
Parallel version of DFS on Device -
On a negative reduced cost cell find a alternating path that improves the objective function
*/
__global__ void find_loops(MatrixCell * d_reducedCosts_ptr, int * d_adjMtx_ptr, float * d_flowMtx_ptr, 
        int * d_vertex_start, int * d_vertex_degree, int * d_adjVertices,
        stackNode * stack, bool * visited, int * backtracker, int * depth, int numSupplies, int numDemands, int bound) {

    int local_id = blockIdx.x*blockDim.x + threadIdx.x;
    MatrixCell c = d_reducedCosts_ptr[local_id];

    // Check bounds and if this reduced cost is negative
    // Bound is number of parallel pivots that need to be performed
    if (local_id < bound && c.cost < -10e-3 ) { 

        int V = numSupplies + numDemands;
        int offset = V * local_id;
        int local_row = c.row;
        int local_col = c.col;
        int _depth = 1;
        backtracker[offset] = local_row;
        
        // then pivot row is - local_row
        // and  pivot col is - local_col

        perform_dfs_sequencial_on_i(d_adjMtx_ptr, d_vertex_start, d_vertex_degree, d_adjVertices,
                &stack[offset], &backtracker[offset], 
                &visited[offset], &_depth, local_col + numSupplies, local_row, V);
    
        if (_depth > 1) {
    
            // A loop was found - complete the book-keeping
            backtracker[offset + _depth] = local_row;
    
            // Update depth and savings for referncing in subsequent kernel //
            depth[local_id] = _depth;
    
        }
        // else depth[local_id] = 0 (remains default)
    }
}


/*
Fetch and view discovered cycles 
Function: Copy depth, backtrack from device and print
*/
__host__ void __debug_utility_1(MatrixCell * d_reducedCosts_ptr, int * backtracker, int * depth,  
    int iteration, int numSupplies, int numDemands, int num_threads_launching) 
{
    std::cout<<"DEBUG UTIITY - 1 | Viewing Discovered Loops"<<std::endl;
    int V = numSupplies + numDemands;

    int * h_backtracker = (int *) malloc(num_threads_launching * V * sizeof(int));
    int * h_depth = (int *) malloc(num_threads_launching * sizeof(int));
    MatrixCell * h_reduced_costs = (MatrixCell *) malloc(num_threads_launching * sizeof(MatrixCell));

    int num_cycles = 0;
    
    hipMemcpy(h_backtracker, backtracker, num_threads_launching * V * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_depth, depth, num_threads_launching * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_reduced_costs, d_reducedCosts_ptr, num_threads_launching * sizeof(MatrixCell), hipMemcpyDeviceToHost);

    for (int i=0; i < num_threads_launching; i++) {
        int offset = V*i;
        if (h_depth[i] > 0) {
            std::cout<<"Iteration : "<<iteration<<" : Thread : "<<i<<" : Depth : "<<h_depth[i]<<" : ";
            for (int j = 0; j <= h_depth[i]; j++) {
                std::cout<<h_backtracker[offset+j]<<" ";
            }
            std::cout<<std::endl;
            // std::cout<<"\t Loop Minimum = "<<h_loop_minimum[i]<<" From :"<<h_loop_min_from[i]<<" To : "<<h_loop_min_to[i]<<std::endl;
            std::cout<<"\t Reduced Cost Row = "<<h_reduced_costs[i].row<<std::endl;
            std::cout<<"\t Reduced Cost Col = "<<h_reduced_costs[i].col<<std::endl;
            std::cout<<"\t Reduced Cost = "<<h_reduced_costs[i].cost<<std::endl;
            num_cycles++;
        }
    }

    free(h_backtracker);
    free(h_depth);
    free(h_reduced_costs);

    std::cout<<"\n"<<num_cycles<<" cycles were discovered!"<<std::endl;
    // *********************** END OF DEBUG UTILITY - 1 *************** //
}

/*
Fetch and view v_owner and v_savings 
Function: Copy arrays from device and print
*/
__host__ void __debug_utility_2(vertex_conflicts * v_conflicts, int numSupplies, int numDemands)
{
    std::cout<<"DEBUG UTIITY - 2 | Viewing Loop Owners"<<std::endl;

    vertex_conflicts * h_v_savings = (vertex_conflicts *) malloc(numSupplies * numDemands * sizeof(vertex_conflicts));
    hipMemcpy(h_v_savings, v_conflicts,  numSupplies * numDemands * sizeof(vertex_conflicts), hipMemcpyDeviceToHost);

    for (int i=0; i < numSupplies; i++) {
        for (int j=0; j < numDemands; j++) {
            std::cout << "Edge - (" <<i<<", "<<j+numSupplies<< ") by Thread : " << h_v_savings[i*numDemands + j].ints[1]<< std::endl;
        }
    }
    // *********************** END OF DEBUG UTILITY - 2 *************** //
}

__host__ void __debug_utility_3(int * backtracker, int * depth, 
    int numSupplies, int numDemands, int num_threads_launching) 
{
    // // *********************** DEBUG UTILITY - 3 *************** //
    // // Fetch and view the loops that do not conflict and maximize savings 
    
    std::cout<<"DEBUG UTIITY - 3 | Viewing Non-Conflicting loops"<<std::endl;
    int V = numSupplies + numDemands;
    int * h_backtracker = (int *) malloc(num_threads_launching * V * sizeof(int));
    int * h_depth = (int *) malloc(num_threads_launching * sizeof(int));
    int num_cycles = 0;
    
    hipMemcpy(h_backtracker, backtracker, num_threads_launching * V * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_depth, depth, num_threads_launching * sizeof(int), hipMemcpyDeviceToHost);

    for (int i=0; i < num_threads_launching; i++) {
        int offset = V*i;
       if (h_depth[i] > 0){
            std::cout<<"Thread : "<<i<<" : Depth : "<<h_depth[i]<<" : ";
            for (int j = 0; j < h_depth[i]; j++) {
                std::cout<<h_backtracker[offset+j]<<" ";
            }
            std::cout<<std::endl;
           num_cycles++;
       }
    }

    free(h_backtracker);
    free(h_depth);

    std::cout<<"\n"<<num_cycles<<" non conflicting cycles were discovered!"<<std::endl;
    // *********************** END OF DEBUG UTILITY - 3 *************** //
}

/*
API to execute parallel pivot
*/
__host__ void perform_a_parallel_pivot(int * backtracker, stackNode * stack, bool * visited,
    int * h_adjMtx_ptr, float * h_flowMtx_ptr, int * d_adjMtx_ptr, float * d_flowMtx_ptr, 
    int * d_vertex_start, int * d_vertex_degree, int * d_adjVertices,
    bool &result, MatrixCell * d_reducedCosts_ptr, int * depth, 
    float * loop_minimum, int * loop_min_from, int * loop_min_to, int * loop_min_id, 
    vertex_conflicts * v_conflicts,
    double &dfs_time, double &resolve_time, double &adjustment_time,
    int iteration, int numSupplies, int numDemands) {
    
    auto _pivot_start = std::chrono::high_resolution_clock::now();
    auto _pivot_end = std::chrono::high_resolution_clock::now();
    auto _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    /*
    Strategy is to execute multiple pivots at the same time
    Resolve conflicts through a barrier
        
    KERNEL 1: Go to all the cells with negative reduced costs -> find the pivots -> evaluate savings
    KERNEL 2: Scan through the discovered loops, for each vertex - atomically update the bet's on each vertex in the loops
    */

    int V = numSupplies + numDemands;
    int num_threads_launching = NUM_THREADS_LAUNCHING(numSupplies, numDemands, PARALLEL_PIVOT_IDEA);
    //std::cout<<"Num Threads = "<<num_threads_launching<<std::endl;

    // Discover Cycles
    _pivot_start = std::chrono::high_resolution_clock::now();
    
    dim3 __blockDim(blockSize, 1, 1);
    dim3 __gridDim(ceil(1.0*num_threads_launching/blockSize), 1, 1);
    
    thrust::fill(thrust::device, depth, depth + (num_threads_launching), 0);
    thrust::fill(thrust::device, visited, visited + (V * num_threads_launching), false);

    find_loops <<<__gridDim, __blockDim>>> (d_reducedCosts_ptr, d_adjMtx_ptr, d_flowMtx_ptr, // Lookups 
        d_vertex_start, d_vertex_degree, d_adjVertices,
        stack, visited,  // Intermediates
        backtracker, depth, // book-keeping
        numSupplies, numDemands, num_threads_launching); // Params
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize()); 
        // xxxxxx - Barrier 1 - xxxxxx
    
    // DEBUG UTILITY 1 ::
    // __debug_utility_1(d_reducedCosts_ptr, backtracker, depth, iteration, 
    // numSupplies, numDemands, num_threads_launching);

    _pivot_end = std::chrono::high_resolution_clock::now();
    _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    dfs_time += _pivot_duration.count();
    
    /* ******************************
        Multi-Pivot Method
    ******************************* */

    // Copy Discovered cycles to host and sequencially execute pivots on the host 
    // Making sure no edge is used twice // 
    bool * edge_visited = (bool *) malloc(numSupplies*numDemands*sizeof(bool)); 
    
    int * h_backtracker = (int *) malloc(num_threads_launching * V * sizeof(int));
    int * h_depth = (int *) malloc(num_threads_launching * sizeof(int));
    MatrixCell * h_reduced_costs = (MatrixCell *) malloc(num_threads_launching * sizeof(MatrixCell));

    _pivot_start = std::chrono::high_resolution_clock::now();

    int num_cycles_pivoted = 0;
    thrust::fill(thrust::host, edge_visited, edge_visited + (numSupplies*numDemands), false);
    
    hipMemcpy(h_backtracker, backtracker, num_threads_launching * V * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_depth, depth, num_threads_launching * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_reduced_costs, d_reducedCosts_ptr, num_threads_launching * sizeof(MatrixCell), hipMemcpyDeviceToHost);
    
    // In the running workflow, we start with most negative reduced cost and proceed thereafter 
    for (int i=0; i < num_threads_launching; i++) {

        int offset = V*i;
        if (h_depth[i] > 0) {
            // check if all the edges are available >> 
            bool cycle_valid = true; 
            int _edge_from, _edge_to, _id;

            for (int j = 0; j <= h_depth[i]-1; j++) {
                
                _edge_from = h_backtracker[offset+j] - numSupplies*(j%2);
                _edge_to = h_backtracker[offset+j+1] - numSupplies*((j+1)%2);
                _id = (_edge_from*numDemands + _edge_to)*((j+1)%2) + (_edge_to*numDemands + _edge_from)*(j%2);
                cycle_valid = (cycle_valid && !(edge_visited[_id]));
                // No need to check further if already found an edge that has been used
                if (!cycle_valid) {
                    // std::cout<<"break"<<std::endl;
                    break;
                }
            }

            if (cycle_valid) {

                // Mark edges in thie cycles as used >>
                #pragma omp parallel
                #pragma omp for
                for (int j = 0; j <= h_depth[i]-1; j++) {
                
                    _edge_from = h_backtracker[offset+j] - numSupplies*(j%2);
                    _edge_to = h_backtracker[offset+j+1] - numSupplies*((j+1)%2);
                    _id = (_edge_from*numDemands + _edge_to)*((j+1)%2) + (_edge_to*numDemands + _edge_from)*(j%2);
                    edge_visited[_id] = true;
                
                }

                #pragma omp barrier

                execute_pivot_on_host_device(h_adjMtx_ptr, h_flowMtx_ptr, d_adjMtx_ptr, d_flowMtx_ptr, 
                    &h_backtracker[offset], h_reduced_costs[i].row, h_reduced_costs[i].col, h_depth[i], 
                    V, numSupplies, numDemands);
                
                num_cycles_pivoted++;

            }
        }
    }

    _pivot_end = std::chrono::high_resolution_clock::now();
    _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    adjustment_time += _pivot_duration.count();

    free(h_backtracker);
    free(h_depth);
    free(h_reduced_costs);

    // End of multi-pivot
    std::cout<<"Iteration : "<<iteration<<" | Number of cycles pivoted : "<<num_cycles_pivoted<<std::endl;

}


// ***********************************************************************

__host__ void _debug_print_APSP(int * d_adjMtx, int * d_pathMtx, int V) {

    int * h_adjMtx_copy = (int *) malloc(sizeof(int)*V*V);
    int * h_pathMtx = (int *) malloc(sizeof(int)*V*V);

    gpuErrchk(hipMemcpy(h_adjMtx_copy, d_adjMtx, V*V*sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_pathMtx, d_pathMtx, V*V*sizeof(int), hipMemcpyDeviceToHost));
	
    std::cout<<" ********* Distances >>"<<std::endl;
    for (int i=0; i<V; i++) {
        std::cout<<i<<" : ";
        for (int j=0; j<V; j++) {
            std::cout<<h_adjMtx_copy[i*V + j]<<", ";
        }
        std::cout<<std::endl;
    }
    std::cout<<" ********* Path >>"<<std::endl;
    for (int i=0; i<V; i++) {
        std::cout<<i<<" : ";
        for (int j=0; j<V; j++) {
            std::cout<<h_pathMtx[i*V + j]<<", ";
        }
        std::cout<<std::endl;
    }
    std::cout << "All point shortest path printed!"<<std::endl;
}

/*
Step 1: Find all point to all points shortest distance with Floyd Warshall using naive implementation 
    of Floyd Warshall algorithm in CUDA

- Step 2: For all negative reduced costs find the paths
- Step 3: Find edge disjoint paths among the ones obtained in 2
- Step 4: Perfrom flow adjustment on the paths
*/
__host__ void perform_a_parallel_pivot_floyd_warshall(int * backtracker, stackNode * stack, bool * visited,
    int * h_adjMtx_ptr, float * h_flowMtx_ptr, int * d_adjMtx_ptr, float * d_flowMtx_ptr, 
    int * d_vertex_start, int * d_vertex_degree, int * d_adjVertices,
    bool &result, MatrixCell * d_reducedCosts_ptr, int * depth, 
    float * loop_minimum, int * loop_min_from, int * loop_min_to, int * loop_min_id, 
    vertex_conflicts * v_conflicts,
    double &dfs_time, double &resolve_time, double &adjustment_time,
    int iteration, int numSupplies, int numDemands) {
    
    auto _pivot_start = std::chrono::high_resolution_clock::now();
    auto _pivot_end = std::chrono::high_resolution_clock::now();
    auto _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    /*
    Strategy is to execute multiple pivots at the same time
    Resolve conflicts through a barrier
        
    KERNEL 1: Go to all the cells with negative reduced costs -> find the pivots -> evaluate savings
    KERNEL 2: Scan through the discovered loops, for each vertex - atomically update the bet's on each vertex in the loops
    */

    int V = numSupplies + numDemands;
    int num_threads_launching = NUM_THREADS_LAUNCHING(numSupplies, numDemands, PARALLEL_PIVOT_IDEA);
    std::cout<<"Num Threads = "<<num_threads_launching<<std::endl;

    // Discover Cycles
    
    int * d_adjMtx_ptr_copy, * d_pathMtx;
    
    int _utm_entries = V*V;  // (V*(V+1))/2; // Number of entries in upper triangular matrix 

    // Allocate memory for floyd warshall
	gpuErrchk(hipMalloc((void **) &d_adjMtx_ptr_copy, _utm_entries*sizeof(int)));
	gpuErrchk(hipMalloc((void **) &d_pathMtx, _utm_entries*sizeof(int)));
	
    _pivot_start = std::chrono::high_resolution_clock::now();

	// Make a copy of adjacency matrix to make depth
    // IDEA: run my_signum all at once to get rid of that in the floyd warshall kernel - insted of memcpy run a kernel	
	thrust::fill(thrust::device, d_pathMtx, d_pathMtx + _utm_entries, -1);
    
    dim3 dimBlock(blockSize, blockSize, 1);
    dim3 dimGrid(ceil(1.0*V/blockSize),ceil(1.0*V/blockSize),1);
    
    fill_adjMtx <<< dimGrid, dimBlock >>> (d_adjMtx_ptr_copy, d_adjMtx_ptr, d_pathMtx, V);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // Initialize the grid and block dimensions here
    dim3 dimGrid2((V - 1) / blockSize + 1, (V - 1) / blockSize + 1, 1);
    dim3 dimBlock2(blockSize, blockSize, 1);

    // /* hipFuncSetCacheConfig(reinterpret_cast<const void*>(_naive_fw_kernel), hipFuncCachePreferL1); */
    for (int vertex = 0; vertex < V; ++vertex) {
        _naive_floyd_warshall_kernel <<< dimGrid2, dimBlock2 >>> (vertex, V, d_adjMtx_ptr_copy, d_pathMtx);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }

    int * done;
    

    // DEBUG UTILITY >>
    // _debug_print_APSP(d_adjMtx_ptr_copy, d_pathMtx, V);

    _pivot_end = std::chrono::high_resolution_clock::now();
    _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    dfs_time += _pivot_duration.count();

    std::cout<<"APSP Completed in "<<dfs_time<<" microseconds"<<std::endl;
    exit(0);

}




/******************** DUMP ***********************







/*
The Novel Conflict Selector >>
Reference: https://stackoverflow.com/questions/17411493/how-can-i-implement-a-custom-atomic-function-involving-several-variables 
*/
__device__ unsigned long long int atomicMinAuxillary(unsigned long long int* address, float val1, int val2)
{
    vertex_conflicts loc, loctest;
    loc.floats[0] = val1;
    loc.ints[1] = val2;
    loctest.ulong = *address;
    while (val1  < loctest.floats[0] || (val1 == loctest.floats[0] && val2 < loctest.ints[1])) {
        // condition and tie-braker (bland's rule)
        loctest.ulong = atomicCAS(address, loctest.ulong,  loc.ulong);
    } 
    return loctest.ulong;
}

/*
Kernel 2 :: Step 1
Resolve conflicts |  Search for vertices that fall under conflicting loops
*/
__global__ void resolve_conflicts_step_1(int * depth, int * backtracker, MatrixCell * d_reducedCosts_ptr, 
        vertex_conflicts * v_conflicts, int numSupplies, int numDemands, int bound) {
            
    int local_id = blockIdx.y*blockDim.y + threadIdx.y;
    int V = numSupplies + numDemands;
    int offset = V * local_id;
    
    if (local_id < bound) {

        // Check if this is a cell along which pivoting is performing
        int _depth = depth[local_id];
        // Real loops exists on this edge (cell) -
        if (_depth > 0) {

            // Find Savings
            float r = d_reducedCosts_ptr[local_id].cost;
            int _edge_from, _edge_to, _id;
            // If this loop is pivoted then this is the savings you get 

            for (int i=0; i<_depth-1; i++) {

                _edge_from = backtracker[offset+i] - numSupplies*(i%2);
                _edge_to = backtracker[offset+i+1] - numSupplies*((i+1)%2);
                
                /* 
                Atomically make the comparison and assign

                Essentially the following is performed in an atomic sense
                if (_savings < v_savings[_vtx]) {
                    v_savings[_vtx] = _savings;
                    v_owner[_vtx] = local_id;
                } 
                */

                _id = (_edge_from*numDemands + _edge_to)*((i+1)%2) + (_edge_to*numDemands + _edge_from)*(i%2);
                atomicMinAuxillary(&(v_conflicts[_id].ulong), r, local_id);
            }
        }
    }
}

/*
Resolve conflicts | Step 2 : Kill threads => discard the loops => Set depth = 0
*/
__global__ void resolve_conflicts_step_2(int * depth, int * backtracker, vertex_conflicts * v_conflicts, 
        int numSupplies, int numDemands, int num_threads_launching) {
            
    int local_id = blockIdx.x*blockDim.x + threadIdx.x;
    int V = numSupplies + numDemands;
    int offset = V * local_id;
    
    if (local_id < num_threads_launching) {
        // Check if this is a cell along which pivoting is performing
        int _depth = depth[local_id];
        // Real loops exists on this edge (cell) -
        if (_depth > 0) {
            // Check continuity along all vertices in loop if v_owner is this thread itself >>
            bool _continuity = true;
            int _edge_from, _edge_to, _id, i=0;
            while (i < _depth-1 && _continuity) {
                _edge_from = backtracker[offset+i] - numSupplies*(i%2);
                _edge_to = backtracker[offset+i+1] - numSupplies*((i+1)%2);
                _id = (_edge_from*numDemands + _edge_to)*((i+1)%2) + (_edge_to*numDemands + _edge_from)*(i%2);
                _continuity = (_continuity && (v_conflicts[_id].ints[1] == local_id));
                i++;
            }

            if (!_continuity) { // Kill this thread in this case
                depth[local_id] = 0;
            }
        }
    }
}

/*
Kernel to execute the flow adjustments in parallel >>
*/
__global__ void run_flow_adjustments(int * d_adjMtx_ptr, float * d_flowMtx_ptr, int * depth, 
            int * backtracker, float * loop_minimum,
            int * loop_min_from, int * loop_min_to, int * loop_min_id,
            int numSupplies, int numDemands) {
            
    int local_row = blockIdx.y*blockDim.y + threadIdx.y;
    int local_col = blockIdx.x*blockDim.x + threadIdx.x;
    int local_id = local_row*numDemands + local_col;
    int V = numSupplies + numDemands;
    int offset = V * local_id;
    
    if (local_row < numSupplies && local_col < numDemands) {
        // Check if this is a cell along which pivoting is performing
        int _depth = depth[local_id];
        // Real loops exists on this edge (cell) -
        if (_depth > 0) {

            int _from, _to, id, j=-1, min_from = loop_min_from[local_id], 
                min_to = loop_min_to[local_id], min_flow_id = loop_min_id[local_id];
            float _flow, _min_flow = loop_minimum[local_id];

            for (int i=1; i<_depth; i++)
            {
                _from = backtracker[offset+i];
                _to = backtracker[offset+i+1];
                id = d_adjMtx_ptr[TREE_LOOKUP(_from, _to, V)] - 1;
                _flow = j*_min_flow;
                d_flowMtx_ptr[id] += _flow;
                j *= -1;
            }

            // Do the replacment between exiting i - entering j on both host and device
            // Remove edge
            id = TREE_LOOKUP(min_from, min_to, V);
            d_adjMtx_ptr[id] = 0;
            // Insert edge
            id = TREE_LOOKUP(local_row, local_col+numSupplies, V);
            d_adjMtx_ptr[id] = min_flow_id + 1;
            // Update new flow 
            d_flowMtx_ptr[min_flow_id] = _min_flow;
        }
    }
}

/*

// Resolve Conflicts >> 
    _pivot_start = std::chrono::high_resolution_clock::now();
    // std::cout<<"Parallel Pivoiting : Discovered Loops!"<<std::endl;
    // std::cout<<"Parallel Pivoiting : Resolving Conflicts | Running Step 1 (Discover conflicts) ..."<<std::endl;        
    vertex_conflicts _vtx_conflict_default;
    _vtx_conflict_default.floats[0] = FLT_MAX;
    _vtx_conflict_default.ints[1] = -1;
    thrust::fill(thrust::device, v_conflicts, v_conflicts + (numSupplies*numDemands), _vtx_conflict_default);

    resolve_conflicts_step_1 <<<__gridDim, __blockDim>>> (depth, backtracker, d_reducedCosts_ptr, 
        v_conflicts, numSupplies, numDemands, num_threads_launching);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
        // xxxxxx - Barrier 2 - xxxxxx

    // DEBUG UTILITY 2 ::
    // __debug_utility_2(v_conflicts, numSupplies, numDemands);
    
    // std::cout<<"Parallel Pivoiting : Completed Step 1 | Running Step 2 (Resolve Conflicts) ..."<<std::endl;
    resolve_conflicts_step_2 <<<__gridDim, __blockDim>>> (depth, backtracker, v_conflicts, numSupplies, numDemands, num_threads_launching);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    // xxxxxx - Barrier 3 - xxxxxx

    // DEBUG UTILITY 3 ::
    __debug_utility_3(backtracker, depth, numSupplies, numDemands, num_threads_launching);
    exit(0);
    _pivot_end = std::chrono::high_resolution_clock::now();
    _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    resolve_time += _pivot_duration.count();

    // std::cout<<"Parallel Pivoiting : Conflicts Resolved | Running flow adjustments ..."<<std::endl;        
    // Check if any conflicting pivots still exist >>
    _pivot_start = std::chrono::high_resolution_clock::now();
    int _conflict_flag = thrust::reduce(thrust::device, depth, depth + (numSupplies*numDemands), 0);
    if (_conflict_flag > 0) {
        
        // METHOD 1 : RUN ADJUSTMENTS IN PARALLEL
        if (PARALLEL_PIVOTING_METHOD=="pure") {
            std::cout<<"THIS PIVOTING METHOD IS OUT DATED, TRY - hybrid!"<<std::endl;
            exit(-1);
            run_flow_adjustments <<<__gridDim, __blockDim>>> (d_adjMtx_ptr, d_flowMtx_ptr, depth, backtracker, loop_minimum, 
                loop_min_from, loop_min_to, loop_min_id, numSupplies, numDemands);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
                // xxxxxx - Barrier 4 - xxxxxx
        }

        // METHOD 2 : RUN FLOW ADJUSTMENTS IN SEQ on host (for all independent loops)
        else if (PARALLEL_PIVOTING_METHOD=="hybrid") {
            
            int * _h_depth = (int *) malloc(num_threads_launching * sizeof(int));
            int * _h_backtracker = (int *) malloc(sizeof(int)*V);
            float min_flow = INT_MAX, _flow;
            int min_from, min_to, min_flow_id, _from, _to, id;
            gpuErrchk(hipMemcpy(_h_depth, depth, num_threads_launching * sizeof(int), hipMemcpyDeviceToHost));
            
            for (int i=0; i < (num_threads_launching); i++) {
                
                if (_h_depth[i] > 0) {

                    int offset = V*i;
                    int _pivot_row =  i/numDemands;
                    int _pivot_col = i - (_pivot_row*numDemands);

                    gpuErrchk(hipMemcpy(_h_backtracker, &backtracker[offset], (_h_depth[i]+1)*sizeof(int), hipMemcpyDeviceToHost));
        
                    for (int j=0; j<_h_depth[i]; j++) 
                    {
                        if (j%2==1)
                        {
                            _from = _h_backtracker[j];
                            _to = _h_backtracker[j+1];
                            id = h_adjMtx_ptr[TREE_LOOKUP(_from, _to, V)] - 1;
                            _flow = h_flowMtx_ptr[id];
                            if (_flow < min_flow) 
                            {
                                min_flow = _flow;
                                min_flow_id = id;
                                min_from = _from;
                                min_to = _to;
                            }
                        }
                    }
                    
                    // std::cout<<"Min flow :"<<min_flow<<std::endl;
                    // std::cout<<"Min from :"<<min_from<<std::endl;
                    // std::cout<<"Min to :"<<min_to<<std::endl;
                    // std::cout<<"Min id :"<<min_flow_id<<std::endl;

                    do_flow_adjustment_on_host_device(h_adjMtx_ptr, h_flowMtx_ptr, d_adjMtx_ptr, d_flowMtx_ptr, _h_backtracker,
                        min_flow, min_from, min_to, min_flow_id,
                        _pivot_row, _pivot_col, _h_depth[i], V, numSupplies, numDemands);
                }
            }

            free(_h_depth);
            free(_h_backtracker);
        }

        else {
            std::cout<<"Invalid Parallel Pivoting Method!"<<std::endl;
            exit(-1);
        }
    }

    else 
    {
        std::cout<<"No independent cycles found!"<<std::endl;
    }
    _pivot_end = std::chrono::high_resolution_clock::now();
    _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    adjustment_time += _pivot_duration.count();
*/
