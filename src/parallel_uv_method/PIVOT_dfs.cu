#include "hip/hip_runtime.h"
#include "PIVOT_dfs.h"


__host__ void Initialize_pivoting() {
    if (PIVOTING_STRATEGY == "parallel") {

        // Allocate appropriate resources >>
        int num_threads_launching = data->numSupplies*data->numDemands;
        hipMalloc((void **) &backtracker, num_threads_launching * V * sizeof(int));
        hipMalloc((void **) &depth, num_threads_launching * sizeof(int));
        hipMalloc((void **) &loop_minimum, num_threads_launching * sizeof(float));
        hipMalloc((void **) &loop_min_from, num_threads_launching * sizeof(int));
        hipMalloc((void **) &loop_min_to, num_threads_launching * sizeof(int));
        hipMalloc((void **) &loop_min_id, num_threads_launching * sizeof(int));
        hipMalloc((void **) &stack, num_threads_launching * V * sizeof(stackNode));
        hipMalloc((void **) &visited, num_threads_launching * V * sizeof(bool));
        hipMalloc((void **) &v_conflicts, V * sizeof(vertex_conflicts));
        _vtx_conflict_default.floats[0] = FLT_MAX;
        _vtx_conflict_default.ints[1] = -1;
        std::cout<<"\tParallel Pivoting : Allocated Resources on Device"<<std::endl;
    
    }

}

__host__ void terminate_PIVOT() {
    if (PIVOTING_STRATEGY == "parallel")
     {
        // Free up space >>
        hipFree(stack);
        hipFree(visited);
        hipFree(v_conflicts);
        hipFree(backtracker);
        hipFree(depth);
        hipFree(loop_minimum);
        hipFree(loop_min_from);
        hipFree(loop_min_to);
    }
}

/*
Push a node in the provided stack
*/
__host__ __device__ void stack_push(stackNode * stack, int &stack_top, int vtx, int depth)
{
    stack_top++;
    stackNode node = {.index = vtx, .depth = depth};
    stack[stack_top] = node;
}

/*
Pop a node from the provided stack
*/
__host__ __device__ stackNode stack_pop(stackNode * stack, int &stack_top)
{
    stackNode vtx;
    vtx = stack[stack_top];
    stack_top--;
    return vtx;
}

/*
Perform depth first search looking for route to execute the pivot
*/
__host__ __device__ void perform_dfs_sequencial_on_i(int * adjMtx, stackNode * stack, int * backtracker, bool * visited, 
        int * depth, int starting_vertex, int target_vertex, int V)
{   
    
    int key, current_depth = 1, stack_top = -1;
    stackNode current_vertex;
    stack_push(stack, stack_top, starting_vertex, current_depth);

    while(!(stack_top == -1))
    {
        current_vertex = stack_pop(stack, stack_top);

        // check if current vtx has been already visited in this search
        if (!visited[current_vertex.index])
        {
            // if not visited: >> 
            //  - mark this as visited 
            //  - see if current_vertex is adj to the starting point, 
            //        if not - queue the vertices that are adjacent to current vertex, increment depth
            visited[current_vertex.index]=true;

            // Do the book-keeping
            current_depth = current_vertex.depth + 1;
            backtracker[current_vertex.depth] = current_vertex.index;

            // check if target point is adjacent
            key = TREE_LOOKUP(target_vertex, current_vertex.index, V);
            if (adjMtx[key] > 0 && current_depth > 1)
            {
                // Leads back to origin - this completes the cycle - exit the loop
                // std::cout<<"Loop Breaks"<<std::endl;
                *depth = current_depth;
                break;
            }
            else
            {
                // Append the ajacent nodes in stack
                for(int j=0; j < V; j++)
                {
                    key = TREE_LOOKUP(current_vertex.index, j, V);
                    // queue neighbors
                    if(adjMtx[key] > 0)
                    {
                        stack_push(stack, stack_top, j, current_depth);
                    }
                }
            }
        }
        // else - move to next vertex : pop_next, Before that >>
        // Iterations have explored the childeren and now going up in the recursion tree 
        // to something that is still pending to be explored -
        if (stack_top == -1)
        {
            *depth=1;
        }
    }
}

/*
Do a copy from new value to device pointer
*/
__host__ void modify_flowMtx_on_device(float * d_flowMtx_ptr, int id, float new_value) {
    hipMemcpy(&d_flowMtx_ptr[id], &new_value, sizeof(float), hipMemcpyHostToDevice);
}

/*
Replaces the exiting basic flow with entering non basic flow
Does the necessary adjustments on the variables on device memory
*/
__host__ void exit_i_and_enter_j(int * d_adjMtx_ptr, float * d_flowMtx_ptr, int exit_src, int exit_dest, 
        int enter_src, int enter_dest, int min_flow_indx, float min_flow, int V) {
            
    int id;
    int null_value = 0;
    int new_value = min_flow_indx + 1;

    // Set value for exiting in d
    id = TREE_LOOKUP(exit_src, exit_dest, V);
    hipMemcpy(&d_adjMtx_ptr[id], &null_value, sizeof(int), hipMemcpyHostToDevice);

    // Set value for entering to the appropriate
    id = TREE_LOOKUP(enter_src, enter_dest, V);
    hipMemcpy(&d_adjMtx_ptr[id], &new_value, sizeof(int), hipMemcpyHostToDevice);

    // The flow would have become zero - update it again
    hipMemcpy(&d_flowMtx_ptr[min_flow_indx], &min_flow, sizeof(float), hipMemcpyHostToDevice);

}

/*
Parallel version of DFS on Device
*/
__global__ void find_loops_and_savings(float * d_reducedCosts_ptr, int * d_adjMtx_ptr, float * d_flowMtx_ptr, 
        stackNode * stack, bool * visited, int * backtracker, int * depth, float * loop_minimum,
        int * loop_min_from, int * loop_min_to, int * loop_min_id,
        int numSupplies, int numDemands) {

    int local_row = blockIdx.y*blockDim.y + threadIdx.y;
    int local_col = blockIdx.x*blockDim.x + threadIdx.x;
    int local_id = local_row*numDemands + local_col;
    int V = numSupplies + numDemands;
    int offset = V * local_id;

    if (local_row < numSupplies && local_col < numDemands) {
        float r = d_reducedCosts_ptr[local_id];    
        // Check if this reduced cost is negative
        if (r < 0) {
            
            int _depth = 1;
            backtracker[offset + 0] = local_row;
            // then pivot row is - local_row
            // and  pivot col is - local_col
            perform_dfs_sequencial_on_i(d_adjMtx_ptr, &stack[offset], &backtracker[offset], 
                &visited[offset], &_depth, numSupplies + local_col, local_row, V);
            
            if (!(_depth <= 1)) {
                
                // A loop was found - complete the book-keeping
                backtracker[offset + _depth] = local_row;

                // Traverse the loop find the minimum flow that could be increased
                // on the incoming edge >>
                int id, _from = -1, _to = -1, _min_flow_id = -1, _min_from = -1, _min_to = -1;
                float _flow, min_flow = FLT_MAX;

                for (int i=0; i<_depth; i++) 
                {
                    if (i%2==1)
                    {
                        _from = backtracker[offset+i];
                        _to = backtracker[offset+i+1];
                        id = d_adjMtx_ptr[TREE_LOOKUP(_from, _to, V)] - 1;
                        _flow = d_flowMtx_ptr[id];
                        
                        if (_flow < min_flow) 
                        {
                            min_flow = _flow;
                            _min_flow_id = id;
                            _min_from = _from;
                            _min_to = _to;
                        }
                    }
                }

                // Update depth and savings for referncing in subsequent kernel //
                depth[local_id] = _depth;
                loop_minimum[local_id] = min_flow;
                loop_min_from[local_id] = _min_from;
                loop_min_to[local_id] = _min_to;
                loop_min_id[local_id] = _min_flow_id;
            }
            // Otherwise depth[local_id] = 0 (remains default)
        }
    }
}

/*
The Novel Conflict Selector >>
Reference: https://stackoverflow.com/questions/17411493/how-can-i-implement-a-custom-atomic-function-involving-several-variables 
*/

__device__ unsigned long long int atomicMinAuxillary(unsigned long long int* address, float val1, int val2)
{
    vertex_conflicts loc, loctest;
    loc.floats[0] = val1;
    loc.ints[1] = val2;
    loctest.ulong = *address;
    while (val1  < loctest.floats[0] || (val1 == loctest.floats[0] && val2 < loctest.ints[1])) {
        // condition and tie-braker (bland's rule)
        loctest.ulong = atomicCAS(address, loctest.ulong,  loc.ulong);
    } 
    return loctest.ulong;
}

/*
Resolve conflicts | Step 1 : Search for vertices that fall under conflicting loops
*/
__global__ void resolve_conflicts_step_1(int * depth, int * backtracker, float * loop_minimum, float * d_reducedCosts_ptr, 
        vertex_conflicts * v_conflicts, int numSupplies, int numDemands) {
            
    int local_row = blockIdx.y*blockDim.y + threadIdx.y;
    int local_col = blockIdx.x*blockDim.x + threadIdx.x;
    int local_id = local_row*numDemands + local_col;
    int V = numSupplies + numDemands;
    int offset = V * local_id;
    
    if (local_row < numSupplies && local_col < numDemands) {
        // Check if this is a cell along which pivoting is performing
        int _depth = depth[local_id];
        // Real loops exists on this edge (cell) -
        if (_depth > 0) {

            // Find Savings
            float r = d_reducedCosts_ptr[local_id];
            // If this loop is pivoted then this is the savings you get
            float _savings = r*loop_minimum[local_id]; 
            int _vtx;

            for (int i=0; i<_depth; i++) {
                _vtx = backtracker[offset+i];
                // Atomically make the comparison and assign
                /* Essentially the following is performed in an atomic sense
                if (_savings < v_savings[_vtx]) {
                    v_savings[_vtx] = _savings;
                    v_owner[_vtx] = local_id;
                } 
                */   
                atomicMinAuxillary(&(v_conflicts[_vtx].ulong), _savings, local_id);
            }
        }
    }
}

/*
Resolve conflicts | Step 2 : Kill threads => discard the loops => Set depth = 0
*/
__global__ void resolve_conflicts_step_2(int * depth, int * backtracker, vertex_conflicts * v_conflicts, int numSupplies, int numDemands) {
            
    int local_row = blockIdx.y*blockDim.y + threadIdx.y;
    int local_col = blockIdx.x*blockDim.x + threadIdx.x;
    int local_id = local_row*numDemands + local_col;
    int V = numSupplies + numDemands;
    int offset = V * local_id;
    
    if (local_row < numSupplies && local_col < numDemands) {
        // Check if this is a cell along which pivoting is performing
        int _depth = depth[local_id];
        // Real loops exists on this edge (cell) -
        if (_depth > 0) {
            // Check continuity along all vertices in loop if v_owner is this thread itself >>
            bool _continuity = true;
            int _vtx, i=0;
            while (i < _depth && _continuity) {
                _vtx = backtracker[offset+i];
                _continuity = (_continuity && (v_conflicts[_vtx].ints[1] == local_id));
                i++;
            }
            if (!_continuity) { // Kill this thread in this case
                depth[local_id] = 0;
            }
        }
    }
}

/*
Kernel to execute the flow adjustments in parallel >>
*/
__global__ void run_flow_adjustments(int * d_adjMtx_ptr, float * d_flowMtx_ptr, int * depth, 
            int * backtracker, float * loop_minimum,
            int * loop_min_from, int * loop_min_to, int * loop_min_id,
            int numSupplies, int numDemands) {
            
    int local_row = blockIdx.y*blockDim.y + threadIdx.y;
    int local_col = blockIdx.x*blockDim.x + threadIdx.x;
    int local_id = local_row*numDemands + local_col;
    int V = numSupplies + numDemands;
    int offset = V * local_id;
    
    if (local_row < numSupplies && local_col < numDemands) {
        // Check if this is a cell along which pivoting is performing
        int _depth = depth[local_id];
        // Real loops exists on this edge (cell) -
        if (_depth > 0) {

            int _from, _to, id, j=-1, min_from = loop_min_from[local_id], 
                min_to = loop_min_to[local_id], min_flow_id = loop_min_id[local_id];
            float _flow, _min_flow = loop_minimum[local_id];

            for (int i=1; i<_depth; i++) 
            {
                _from = backtracker[offset+i];
                _to = backtracker[offset+i+1];
                id = d_adjMtx_ptr[TREE_LOOKUP(_from, _to, V)] - 1;
                _flow = j*_min_flow;
                d_flowMtx_ptr[id] += _flow;
                j *= -1;
            }

            // Do the replacment between exiting i - entering j on both host and device
            // Remove edge
            id = TREE_LOOKUP(min_from, min_to, V);
            d_adjMtx_ptr[id] = 0;
            // Insert edge
            id = TREE_LOOKUP(local_row, local_col+numSupplies, V);
            d_adjMtx_ptr[id] = min_flow_id + 1;
            // Update new flow 
            d_flowMtx_ptr[min_flow_id] = _min_flow;
        }
    }
}

__host__ void do_flow_adjustment_on_host(int * h_adjMtx_ptr, float * h_flowMtx_ptr, 
        int * d_adjMtx_ptr, float * d_flowMtx_ptr, int * backtracker, float min_flow, int min_from, int min_to, int min_flow_id,
        int pivot_row, int pivot_col, int depth, int V, int numSupplies, int numDemands) {

    // std::cout<<"Depth : "<<depth<<" : ";
    // for (int j = 0; j < depth + 1; j++) {
    //     std::cout<<backtracker[j]<<" ";
    // }
    // std::cout<<std::endl;
    // std::cout<<"Min flow : "<<min_flow<<std::endl;
    // std::cout<<"Min from : "<<min_from<<std::endl;
    // std::cout<<"Min to : "<<min_to<<std::endl;
    // std::cout<<"Min id : "<<min_flow_id<<std::endl;
    // std::cout<<"Pivot Row : "<<pivot_row<<std::endl;
    // std::cout<<"Pivot Col : "<<pivot_col<<std::endl;

    int j=-1, _from, _to, id;
    float _flow;
    for (int i=1; i<depth; i++) 
    {
        _from = backtracker[i];
        _to = backtracker[i+1];
        id = h_adjMtx_ptr[TREE_LOOKUP(_from, _to, V)] - 1;
        _flow = j*min_flow;
        h_flowMtx_ptr[id] += _flow;
        modify_flowMtx_on_device(d_flowMtx_ptr, id, h_flowMtx_ptr[id]);
        j *= -1;
    }

    // Do the replacment between exiting i - entering j on both host and device
    // Remove edge
    id = TREE_LOOKUP(min_from, min_to, V);
    h_adjMtx_ptr[id] = 0;
    // Insert edge
    id = TREE_LOOKUP(pivot_row, pivot_col+ numSupplies, V);
    h_adjMtx_ptr[id] = min_flow_id + 1;
    // Update new flow 
    h_flowMtx_ptr[min_flow_id] = min_flow;

    exit_i_and_enter_j(d_adjMtx_ptr, d_flowMtx_ptr, 
        min_from, min_to, 
        pivot_row, pivot_col + numSupplies, 
        min_flow_id, min_flow, V);
}

__host__ void execute_pivot_on_host(int * h_adjMtx_ptr, float * h_flowMtx_ptr, 
        int * d_adjMtx_ptr, float * d_flowMtx_ptr, int * backtracker, 
        int pivot_row, int pivot_col, int depth, int V, int numSupplies, int numDemands) {

    // *******************************************
    // STEP: Performing the pivot operation 
        // Step 1 - Find the Minimum flow
        // Step 2 - Adjust the Flow
    // *******************************************
            
    int id, _from = -1, _to = -1, min_flow_id = -1, min_from = -1, min_to = -1;
    float _flow, min_flow = INT_MAX;

    // ########### STEP 1 | Finding the minimum flow >>
    // Traverse the loop find the minimum flow that could be increased
    // on the incoming edge >> 
    for (int i=0; i<depth; i++) 
    {
        if (i%2==1) 
        {
            _from = backtracker[i];
            _to = backtracker[i+1];
            id = h_adjMtx_ptr[TREE_LOOKUP(_from, _to, V)] - 1;
            _flow = h_flowMtx_ptr[id];
            
            if (_flow < min_flow) 
            {
                min_flow = _flow;
                min_flow_id = id;
                min_from = _from;
                min_to = _to;
            }
        }
    }

    // ########### STEP 2 | Executing the flow adjustment >>

    // Skip the first edge (entering edge)
    // Exiting Edge will become automatically zero (min_from, min_to)
    // Note - minflow value is zero if there's a degenerate pivot!
    do_flow_adjustment_on_host(h_adjMtx_ptr, h_flowMtx_ptr, d_adjMtx_ptr, d_flowMtx_ptr, backtracker,
            min_flow, min_from, min_to, min_flow_id,
            pivot_row, pivot_col, depth, V, numSupplies, numDemands);

}

void perform_a_sequencial_pivot() {

    // An incoming edge from vertex = pivot_row to vertex = numSupplies + pivot_col
            
            // *******************************************
            // STEP 2: If not, Traverse tree and find a route
            // *******************************************

            // BOOK 1: Stores the routes discovered for each thread -
            int * backtracker = (int *) malloc(sizeof(int)*V);
            
            // BOOK 2: Stores the runtime stack for DFS running on each thread
            stackNode * stack = (stackNode *) malloc(sizeof(stackNode)*V);
            
            // BOOK 3: Keeps a track if any vertex was visited during DFS for each thread
            bool * visited = (bool *) malloc(sizeof(bool)*V);
            memset(visited, 0, V*sizeof(bool));

            // BOOK 4: Stores length of routes discovered for each thread
            int depth;

            // SEQUENCIAL PROCEDURE >>
            backtracker[0] = pivot_row;
            depth = 1;
            
            // Find a loop by performing DFS from pivot_col upto pivot row >>
            perform_dfs_sequencial_on_i(h_adjMtx_ptr, stack, backtracker, visited, &depth, 
                pivot_col+data->numSupplies, pivot_row, V);
            
            // If loop not discovered >>
            if (depth <= 1) {
                std::cout<<" !! Error !! : Degenerate pivot cannot be performed, this is probably not a tree but forest!"<<std::endl;
                std::cout<<"Solution IS NOT OPTIMAL!"<<std::endl;
                // view_uvra();
                // std::cout<<"From : "<<pivot_row<<" | To : "<<pivot_col+data->numSupplies<<std::endl;
                result = true;
                return;
            }

            backtracker[depth] = pivot_row;

            // *******************************************
            // STEP 3: Performing the pivot operation 
            // *******************************************
            // std::cout<<"Pivot Row : "<<pivot_row<<std::endl;
            // std::cout<<"Pivot Col : "<<pivot_col<<std::endl;

            execute_pivot_on_host(h_adjMtx_ptr, h_flowMtx_ptr, d_adjMtx_ptr, d_flowMtx_ptr, backtracker,
            pivot_row, pivot_col, depth, V, data->numSupplies, data->numDemands);

            free(backtracker);
            free(stack);
            free(visited);

}



void perform_a_parallel_pivot() {
    /*
            Strategy is to execute multiple pivots at the same time
            Resolve conflicts through a barrier
            
            KERNEL 1: Go to all the cells with negative reduced costs -> find the pivots -> evaluate savings
            KERNEL 2: 
            */

            dim3 __blockDim(blockSize, blockSize, 1);
            dim3 __gridDim(ceil(1.0*data->numDemands/blockSize), ceil(1.0*data->numSupplies/blockSize), 1);
    
            thrust::fill(thrust::device, depth, depth + data->numSupplies*data->numDemands, 0);
            thrust::fill(thrust::device, visited, visited + (V * data->numSupplies*data->numDemands), false);

            find_loops_and_savings <<<__gridDim, __blockDim>>> (d_reducedCosts_ptr, d_adjMtx_ptr, d_flowMtx_ptr, // Lookups 
                    stack, visited, backtracker,  // Intermediates
                    depth, loop_minimum,  // Outputs
                    loop_min_from, loop_min_to, loop_min_id, // Book-keeping sake
                    data->numSupplies, data->numDemands); // Params
            hipDeviceSynchronize(); // xxxxxx - Barrier 1 - xxxxxx
            
            // *********************** DEBUG UTILITY - 1 *************** //
            // Fetch and view dicsovered cycles 
            // Function: Copy depth, backtrack from device and print
            
            // std::cout<<"DEBUG UTIITY - 1 | Viewing Discovered Loops"<<std::endl;
            // int num_threads_launching = data->numSupplies*data->numDemands;

            // int * h_backtracker = (int *) malloc(num_threads_launching * V * sizeof(int));
            // int * h_depth = (int *) malloc(num_threads_launching * sizeof(int));
            // float * h_loop_minimum = (float *) malloc(num_threads_launching * sizeof(float));
            // int * h_loop_min_from = (int *) malloc(num_threads_launching * sizeof(int));
            // int * h_loop_min_to = (int *) malloc(num_threads_launching * sizeof(int));
            // int * h_loop_min_id = (int *) malloc(num_threads_launching * sizeof(int));
            // h_reduced_costs = (float *) malloc(num_threads_launching * sizeof(float));

            // int num_cycles = 0;
            
            // hipMemcpy(h_backtracker, backtracker, num_threads_launching * V * sizeof(int), hipMemcpyDeviceToHost);
            // hipMemcpy(h_depth, depth, num_threads_launching * sizeof(int), hipMemcpyDeviceToHost);
            // hipMemcpy(h_loop_minimum, loop_minimum, num_threads_launching * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(h_loop_min_from, loop_min_from, num_threads_launching * sizeof(int), hipMemcpyDeviceToHost);
            // hipMemcpy(h_loop_min_to, loop_min_to, num_threads_launching * sizeof(int), hipMemcpyDeviceToHost);
            // hipMemcpy(h_loop_min_id, loop_min_id, num_threads_launching * sizeof(int), hipMemcpyDeviceToHost);
            // hipMemcpy(h_reduced_costs, d_reducedCosts_ptr, num_threads_launching * sizeof(float), hipMemcpyDeviceToHost);

            // for (int i=0; i < num_threads_launching; i++) {
            //     int offset = V*i;
            //     if (h_depth[i] > 0){
            //         std::cout<<"Thread - "<<i<<" : Depth : "<<h_depth[i]<<" : ";
            //         for (int j = 0; j <= h_depth[i]; j++) {
            //             std::cout<<h_backtracker[offset+j]<<" ";
            //         }
            //         std::cout<<std::endl;
            //         std::cout<<"\t Loop Minimum = "<<h_loop_minimum[i]<<" From :"<<h_loop_min_from[i]<<" To : "<<h_loop_min_to[i]<<std::endl;
            //         std::cout<<"\t Reduced Costs = "<<h_reduced_costs[i]<<std::endl;
            //         num_cycles++;
            //     }
            // }

            // free(h_backtracker);
            // free(h_depth);
            // free(h_loop_minimum);
            // free(h_loop_min_from);
            // free(h_loop_min_to);
            // free(h_loop_min_id);
            // free(h_reduced_costs);

            // std::cout<<"\n"<<num_cycles<<" cycles were discovered!"<<std::endl;

            // *********************** END OF DEBUG UTILITY - 1 *************** //

            // std::cout<<"Parallel Pivoiting : Discovered Loops!"<<std::endl;

            // Resolve Conflicts >>

            thrust::fill(thrust::device, v_conflicts, v_conflicts + V, _vtx_conflict_default);

            // std::cout<<"Parallel Pivoiting : Resolving Conflicts | Running Step 1 (Discover conflicts) ..."<<std::endl;
            
            resolve_conflicts_step_1 <<<__gridDim, __blockDim>>> (depth, backtracker, loop_minimum, d_reducedCosts_ptr, 
                    v_conflicts, data->numSupplies, data->numDemands);
            hipDeviceSynchronize(); // xxxxxx - Barrier 2 - xxxxxx
            
            // *********************** DEBUG UTILITY - 2 *************** //
            // Fetch and view v_owner and v_savings 
            // Function: Copy arrays from device and print
            
            // std::cout<<"DEBUG UTIITY - 2 | Viewing Loop Owners"<<std::endl;

            // vertex_conflicts * h_v_savings = (vertex_conflicts *) malloc(V * sizeof(vertex_conflicts));
            // hipMemcpy(h_v_savings, v_conflicts,  V * sizeof(vertex_conflicts), hipMemcpyDeviceToHost);

            // for (int i=0; i < V; i++) {
            //     std::cout << "Vertex - " << i << " by Thread : " << h_v_savings[i].ints[1]<< std::endl;
            // }

            // *********************** END OF DEBUG UTILITY - 2 *************** //
            
            // std::cout<<"Parallel Pivoiting : Completed Step 1 | Running Step 2 (Resolve Conflicts) ..."<<std::endl;
            
            resolve_conflicts_step_2 <<<__gridDim, __blockDim>>> (depth, backtracker, v_conflicts, data->numSupplies, data->numDemands);
            hipDeviceSynchronize(); // xxxxxx - Barrier 3 - xxxxxx

            // // *********************** DEBUG UTILITY - 3 *************** //
            // // Fetch and view the loops that do not conflict and maximize savings 
            
            // std::cout<<"DEBUG UTIITY - 3 | Viewing Non-Conflicting loops"<<std::endl;

            // int num_threads_launching2 = data->numSupplies*data->numDemands;
            // int * h_backtracker2 = (int *) malloc(num_threads_launching2 * V * sizeof(int));
            // int * h_depth2 = (int *) malloc(num_threads_launching2 * sizeof(int));
            // float * h_loop_minimum2 = (float *) malloc(num_threads_launching2 * sizeof(float));
            // int num_cycles2 = 0;
            
            // hipMemcpy(h_backtracker2, backtracker, num_threads_launching2 * V * sizeof(int), hipMemcpyDeviceToHost);
            // hipMemcpy(h_depth2, depth, num_threads_launching2 * sizeof(int), hipMemcpyDeviceToHost);
            // hipMemcpy(h_loop_minimum2, loop_minimum, num_threads_launching2 * sizeof(float), hipMemcpyDeviceToHost);

            // for (int i=0; i < num_threads_launching2; i++) {
            //     int offset2 = V*i;
            //    if (h_depth2[i] > 0){
            //         std::cout<<"Thread - "<<i<<" : Depth : "<<h_depth2[i]<<" : ";
            //         for (int j = 0; j < h_depth2[i]; j++) {
            //             std::cout<<h_backtracker2[offset2+j]<<" ";
            //         }
            //         std::cout<<std::endl;
            //         std::cout<<"\t Loop Minimum = "<<h_loop_minimum2[i]<<std::endl;
            //        num_cycles2++;
            //    }
            // }

            // free(h_backtracker2);
            // free(h_depth2);
            // free(h_loop_minimum2);

            // std::cout<<"\n"<<num_cycles2<<" non conflicting cycles were discovered!"<<std::endl;

            // *********************** END OF DEBUG UTILITY - 3 *************** //

            // std::cout<<"Parallel Pivoiting : Conflicts Resolved | Running flow adjustments ..."<<std::endl;
            
            // Check if any conflicting pivots exist
            int _conflict_flag = thrust::reduce(thrust::device, depth, depth + data->numSupplies*data->numDemands, 0);
            if (_conflict_flag > 0) {
                
                // METHOD 1 : RUN ADJUSTMENTS IN PARALLEL
                run_flow_adjustments <<<__gridDim, __blockDim>>> (d_adjMtx_ptr, d_flowMtx_ptr, depth, backtracker, loop_minimum, 
                    loop_min_from, loop_min_to, loop_min_id, data->numSupplies, data->numDemands);
                hipDeviceSynchronize(); // xxxxxx - Barrier 4 - xxxxxx

                // METHOD 2 : RUN FLOW ADJUSTMENTS IN SEQ on host (for all independent loops)
                // int * _h_depth = (int *) malloc(data->numSupplies*data->numDemands * sizeof(int));
                // int * _h_backtracker = (int *) malloc(sizeof(int)*V);
                // float min_flow;
                // int min_from, min_to, min_flow_id;
                // hipMemcpy(_h_depth, depth, data->numSupplies*data->numDemands * sizeof(int), hipMemcpyDeviceToHost);
                
                // for (int i=0; i < (data->numSupplies*data->numDemands); i++) {
                //     if (_h_depth[i] > 0) {

                //         int offset = V*i;
                //         pivot_row =  i/data->numDemands;
                //         pivot_col = i - (pivot_row*data->numDemands);

                //         hipMemcpy(_h_backtracker, &backtracker[offset], (_h_depth[i]+1)*sizeof(int), hipMemcpyDeviceToHost);
                //         hipMemcpy(&min_flow, &loop_minimum[i], sizeof(float), hipMemcpyDeviceToHost);
                //         hipMemcpy(&min_from, &loop_min_from[i], sizeof(int), hipMemcpyDeviceToHost);
                //         hipMemcpy(&min_to, &loop_min_to[i], sizeof(int), hipMemcpyDeviceToHost);
                //         hipMemcpy(&min_flow_id, &loop_min_id[i], sizeof(int), hipMemcpyDeviceToHost);
                        
                //         do_flow_adjustment_on_host(h_adjMtx_ptr, h_flowMtx_ptr, d_adjMtx_ptr, d_flowMtx_ptr, _h_backtracker,
                //             min_flow, min_from, min_to, min_flow_id,
                //             pivot_row, pivot_col, _h_depth[i], V, data->numSupplies, data->numDemands);
                        
                //         // std::cout<<"Adjusted!"<<std::endl;
                //    }
                // }
                // free(_h_depth);
                // free(_h_backtracker);
        else 
        {
            std::cout<<"No independent cycles found!"<<std::endl;
        }
}