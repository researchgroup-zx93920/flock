#include "hip/hip_runtime.h"
#include "PIVOT_dfs.h"

/* 
Setup necessary resources for pivoting 
these resources are static and to be shared/overwritten between iterations
*/
__host__ void pivotMalloc(PivotHandler &pivot, int numSupplies, int numDemands) {

    int V = numSupplies + numDemands;

    if (PIVOTING_STRATEGY=="sequencial_dfs") {

        // Pivoting requires some book-keeping (for the DFS procedure)
        // BOOK 1: Stores the routes discovered for each thread -
        pivot.backtracker = (int *) malloc(sizeof(int)*V);    
        // BOOK 2: Stores the runtime stack for DFS running on each thread
        pivot.stack = (stackNode *) malloc(sizeof(stackNode)*V);    
        // BOOK 3: Keeps a track if any vertex was visited during DFS for each thread
        pivot.visited = (bool *) malloc(sizeof(bool)*V);
    }
    
    else if (PIVOTING_STRATEGY == "parallel_dfs") {

        // Allocate appropriate resources, Specific to parallel pivot >>
        int num_threads_launching = NUM_THREADS_LAUNCHING(numSupplies, numDemands, PARALLEL_PIVOT_IDEA);
        // BOOK 1: Stores the routes discovered for each thread
        gpuErrchk(hipMalloc((void **) &pivot.backtracker, num_threads_launching * V * sizeof(int)));
        // BOOK 2: Stores the runtime stack for DFS running on each thread
        gpuErrchk(hipMalloc((void **) &pivot.stack, num_threads_launching * V * sizeof(stackNode)));
        // BOOK 3: Keeps a track if any vertex was visited during DFS for each thread
        gpuErrchk(hipMalloc((void **) &pivot.visited, num_threads_launching * V * sizeof(bool)));
        // BOOK 4: Stores the length of path discovered by each thread through DFS
        gpuErrchk(hipMalloc((void **) &pivot.depth, num_threads_launching * sizeof(int)));
        
        // Following is temporarily removed 
        // gpuErrchk(hipMalloc((void **) &pivot.v_conflicts, numSupplies * numDemands * sizeof(vertex_conflicts)));
    }
}

/* 
Free up acquired resources for pivoting on host device 
*/
__host__ void pivotFree(PivotHandler &pivot) {

    if (PIVOTING_STRATEGY == "sequencial_dfs") {
        
        free(pivot.backtracker);
        free(pivot.stack);
        free(pivot.visited);
    
    }

    else if (PIVOTING_STRATEGY == "parallel_dfs")
    {
        // Free up space >>
        gpuErrchk(hipFree(pivot.backtracker));
        gpuErrchk(hipFree(pivot.stack));
        gpuErrchk(hipFree(pivot.visited));
        gpuErrchk(hipFree(pivot.depth));

    }
}

/*
Push a node in the provided stack
*/
__host__ __device__ void stack_push(stackNode * stack, int &stack_top, int vtx, int depth)
{
    stack_top++;
    stackNode node = {.index = vtx, .depth = depth};
    stack[stack_top] = node;
}

/*
Pop a node from the provided stack
*/
__host__ __device__ stackNode stack_pop(stackNode * stack, int &stack_top)
{
    stackNode vtx;
    vtx = stack[stack_top];
    stack_top--;
    return vtx;
}

/*
Perform depth first search looking for route to execute the pivot
*/
__host__ __device__ void perform_dfs_sequencial_on_i(int * adjMtx, int * vertex_start, int * vertex_degree, int * adjVertices, 
        stackNode * stack, int * backtracker, bool * visited, 
        int * depth, int starting_vertex, int target_vertex, int V)
{   
    
    int key, current_depth = 1, stack_top = -1;
    stackNode current_vertex;
    stack_push(stack, stack_top, starting_vertex, current_depth);

    while(!(stack_top == -1))
    {
        current_vertex = stack_pop(stack, stack_top);

        // check if current vtx has been already visited in this search
        if (!visited[current_vertex.index])
        {
            // if not visited: >> 
            //  - mark this as visited 
            //  - see if current_vertex is adj to the starting point, 
            //        if not - queue the vertices that are adjacent to current vertex, increment depth
            visited[current_vertex.index]=true;

            // Do the book-keeping
            current_depth = current_vertex.depth + 1;
            backtracker[current_vertex.depth] = current_vertex.index;

            // check if target point is adjacent
            key = TREE_LOOKUP(target_vertex, current_vertex.index, V);
            if (adjMtx[key] > 0 && current_depth > 1)
            {
                // Leads back to origin - this completes the cycle - exit the loop
                *depth = current_depth;
                break;
            }
            else
            {
                // Append the ajacent nodes in stack
                int _s = vertex_start[current_vertex.index];
                for (int j = _s; j < _s + vertex_degree[current_vertex.index]; j++)
                {
                    stack_push(stack, stack_top, adjVertices[j], current_depth);
                }
            }
            
        }
        // else - move to next vertex : pop_next, Before that >>
        // Iterations have explored the childeren and now going up in the recursion tree 
        // to something that is still pending to be explored -
        if (stack_top == -1)
        {
            *depth=1;
        }
    }
}


/*
Replaces the exiting basic flow with entering non basic flow
Does the necessary adjustments on the variables on device memory
*/
__host__ void exit_i_and_enter_j(int * d_adjMtx_ptr, float * d_flowMtx_ptr, int exit_src, int exit_dest, 
        int enter_src, int enter_dest, int min_flow_indx, float min_flow, int V) {
            
    int id;
    int null_value = 0;
    int new_value = min_flow_indx + 1;

    // Set value for exiting in d
    id = TREE_LOOKUP(exit_src, exit_dest, V);
    gpuErrchk(hipMemcpy(&d_adjMtx_ptr[id], &null_value, sizeof(int), hipMemcpyHostToDevice));

    // Set value for entering to the appropriate
    id = TREE_LOOKUP(enter_src, enter_dest, V);
    gpuErrchk(hipMemcpy(&d_adjMtx_ptr[id], &new_value, sizeof(int), hipMemcpyHostToDevice));

    // The flow would have become zero - update it again
    // gpuErrchk(hipMemcpy(&d_flowMtx_ptr[min_flow_indx], &min_flow, sizeof(float), hipMemcpyHostToDevice));

}

/*
Do a copy from new value to device pointer
*/
__host__ void modify_flowMtx_on_device(float * d_flowMtx_ptr, int id, float new_value) {
    gpuErrchk(hipMemcpy(&d_flowMtx_ptr[id], &new_value, sizeof(float), hipMemcpyHostToDevice));
}

__host__ void do_flow_adjustment_on_host_device(int * h_adjMtx_ptr, float * h_flowMtx_ptr, 
        int * d_adjMtx_ptr, float * d_flowMtx_ptr, int * backtracker, float min_flow, int min_from, int min_to, int min_flow_id,
        int pivot_row, int pivot_col, int depth, int V, int numSupplies, int numDemands) {

    
    /* *************************** 
        DEBUG UTILITY // Print the discovered loop and pivoting parameters
    **************************** */
    // std::cout<<"Pivot Row : "<<pivot_row<<std::endl;
    // std::cout<<"Pivot Col : "<<pivot_col<<std::endl;
    // std::cout<<" ************** LOOP"<<std::endl;
    // std::cout<<"Depth : "<<depth<<" : ";
    // for (int j = 0; j < depth + 1; j++) {
    //     std::cout<<backtracker[j]<<" ";
    // }
    // std::cout<<std::endl<<" ************** PIVOT"<<std::endl;
    // std::cout<<"Min flow : "<<min_flow<<std::endl;
    // std::cout<<"Min from : "<<min_from<<std::endl;
    // std::cout<<"Min to : "<<min_to<<std::endl;
    // std::cout<<"Min index : "<<min_flow_id<<std::endl;
 
    int _from, _to, id;
    float _flow;

    for (int i=1; i<depth; i++) 
    {
        _from = backtracker[i];
        _to = backtracker[i+1];
        id = h_adjMtx_ptr[TREE_LOOKUP(_from, _to, V)] - 1;
        _flow = ((int) pow(-1, (int)i%2))*min_flow;
        h_flowMtx_ptr[id] += _flow;
        // modify_flowMtx_on_device(d_flowMtx_ptr, id, h_flowMtx_ptr[id]);
    }

    // Do the replacment between exiting i - entering j on both host and device
    // Remove edge
    id = TREE_LOOKUP(min_from, min_to, V);
    h_adjMtx_ptr[id] = 0;
    // Insert edge
    id = TREE_LOOKUP(pivot_row, pivot_col+ numSupplies, V);
    h_adjMtx_ptr[id] = min_flow_id + 1;
    // Update new flow 
    h_flowMtx_ptr[min_flow_id] = min_flow;

    // Communicate device about the removal and addition of an extry-exit variable pair
    exit_i_and_enter_j(d_adjMtx_ptr, d_flowMtx_ptr, 
        min_from, min_to, 
        pivot_row, pivot_col + numSupplies, 
        min_flow_id, min_flow, V);
}

__host__ void execute_pivot_on_host_device(int * h_adjMtx_ptr, float * h_flowMtx_ptr, 
        int * d_adjMtx_ptr, float * d_flowMtx_ptr, int * backtracker, 
        int pivot_row, int pivot_col, int depth, int V, int numSupplies, int numDemands) {

    // *******************************************
    // STEP: Performing the pivot operation 
        // Step 1 - Find the Minimum flow
        // Step 2 - Adjust the Flow
    // *******************************************
            
    int id, _from = -1, _to = -1, min_flow_id = -1, min_from = -1, min_to = -1;
    float _flow, min_flow = INT_MAX;

    // ########### STEP 1 | Finding the minimum flow >>
    // Traverse the loop find the minimum flow that could be increased
    // on the incoming edge >> 
    for (int i=0; i<depth; i++) 
    {
        if (i%2==1) 
        {
            _from = backtracker[i];
            _to = backtracker[i+1];
            id = h_adjMtx_ptr[TREE_LOOKUP(_from, _to, V)] - 1;
            _flow = h_flowMtx_ptr[id];
            
            if (_flow < min_flow) 
            {
                min_flow = _flow;
                min_flow_id = id;
                min_from = _from;
                min_to = _to;
            }
        }
    }

    // ########### STEP 2 | Executing the flow adjustment >>

    // Skip the first edge (entering edge)
    // Exiting Edge will become automatically zero (min_from, min_to)
    // Note - minflow value is zero if there's a degenerate pivot!
    do_flow_adjustment_on_host_device(h_adjMtx_ptr, h_flowMtx_ptr, d_adjMtx_ptr, d_flowMtx_ptr, backtracker,
            min_flow, min_from, min_to, min_flow_id,
            pivot_row, pivot_col, depth, V, numSupplies, numDemands);

}

/*
Pivoting Operation in Transport Simplex. A pivot is complete in following 3 Steps
    Step 1: Check if already optimal 
    Step 2: If not, Traverse tree and find a route (using DFS)
    Step 3: Perform the pivot and adjust the flow
    Step 4/0: Repeat!
*/
__host__ void perform_a_sequencial_pivot(PivotHandler &pivot, PivotTimer &timer,
    Graph &graph, MatrixCell * d_reducedCosts_ptr, bool &result, int numSupplies, int numDemands) {

    MatrixCell min_reduced_cost;
    
    // Find index of most negative reduced cost negative reduced cost >>
    int min_indx = thrust::min_element(thrust::device,
                d_reducedCosts_ptr, d_reducedCosts_ptr + (numSupplies*numDemands), compareCells()) - d_reducedCosts_ptr;
    gpuErrchk(hipMemcpy(&min_reduced_cost, &d_reducedCosts_ptr[min_indx], sizeof(MatrixCell), hipMemcpyDeviceToHost));

    if (min_reduced_cost.cost < 0 && std::abs(min_reduced_cost.cost) > 10e-3) {

        int cell_index = min_reduced_cost.row*numDemands + min_reduced_cost.col;
        int pivot_row =  cell_index/numDemands;
        int pivot_col = cell_index - (pivot_row*numDemands);

        // Preprocess before sequencial pivot
        if (!(CALCULATE_DUAL=="host_bfs")) {
                
                // Copy Adjacency list on host - reuse the same struct for efficient DFS >> 
                gpuErrchk(hipMemcpy(graph.h_vertex_degree, &graph.d_vertex_degree[1], sizeof(int)*graph.V, hipMemcpyDeviceToHost));
                gpuErrchk(hipMemcpy(graph.h_vertex_start, graph.d_vertex_start, sizeof(int)*graph.V, hipMemcpyDeviceToHost));
                gpuErrchk(hipMemcpy(graph.h_adjVertices, graph.d_adjVertices, sizeof(int)*2*(graph.V-1), hipMemcpyDeviceToHost)); 

            }
        
        auto _pivot_start = std::chrono::high_resolution_clock::now();
        auto _pivot_end = std::chrono::high_resolution_clock::now();
        auto _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    
        // *******************************************
        // STEP: Traverse tree and find a cycle
        // *******************************************
        int _depth = 1; // Stores length of cycle discovered for each thread
        pivot.backtracker[0] = pivot_row;
        memset(pivot.visited, 0, graph.V*sizeof(bool));

        // Find a path by performing DFS from pivot_col reaching pivot row to complete cycle >>
        // SEQUENCIAL PROCEDURE to find An incoming edge to vertex = pivot_row from vertex = numSupplies + pivot_col        
        _pivot_start = std::chrono::high_resolution_clock::now();

        perform_dfs_sequencial_on_i(graph.h_adjMtx_ptr, graph.h_vertex_start, graph.h_vertex_degree, graph.h_adjVertices, 
            pivot.stack, pivot.backtracker, pivot.visited, &_depth, 
            pivot_col+numSupplies, pivot_row, graph.V);
    
        _pivot_end = std::chrono::high_resolution_clock::now();
        _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
        timer.cycle_discovery += _pivot_duration.count();

        // If loop not discovered, this is a foolproof check
        // Beacuse graph is a tree, this should not happen anytime 
        // BUT just in case u know this many not be your day :D
        _pivot_start = std::chrono::high_resolution_clock::now();

        if (_depth <= 1) {
            std::cout<<" !! Error !! : Pivot cannot be performed, this is probably not a tree but forest!"<<std::endl;
            std::cout<<"Solution IS NOT OPTIMAL!"<<std::endl;
            result = true;
            return;
        }
        
        // As expected cycle was discovered and stored in backtracker array
        else {
            // *******************************************
            // STEP : Performing the pivot operation 
            // *******************************************
            pivot.backtracker[_depth] = pivot_row;

            // std::cout<<"Printing Cycle :: [ ";
            // for (int i=0; i<= _depth; i++){
            //     std::cout<<pivot.backtracker[i]<<", ";         
            // }
            // std::cout<<"]"<<std::endl;
            // exit(0);

            execute_pivot_on_host_device(graph.h_adjMtx_ptr, graph.h_flowMtx_ptr, 
                    graph.d_adjMtx_ptr, graph.d_flowMtx_ptr, 
                    pivot.backtracker, pivot_row, pivot_col, _depth, 
                    graph.V, numSupplies, numDemands);
        }
        
        _pivot_end = std::chrono::high_resolution_clock::now();
        _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
        timer.adjustment_time += _pivot_duration.count();
    }
    else
    {
        result = true;
        std::cout<<"Pivoting Complete!"<<std::endl;
        return;
    }
}

/*
KERNEL 1 =>
Parallel version of DFS on Device -
On a negative reduced cost cell find a alternating path that improves the objective function
*/
__global__ void find_loops(MatrixCell * d_reducedCosts_ptr, int * d_adjMtx_ptr, float * d_flowMtx_ptr, 
        int * d_vertex_start, int * d_vertex_degree, int * d_adjVertices,
        stackNode * stack, bool * visited, int * backtracker, int * depth, int numSupplies, int numDemands, int bound) {

    int local_id = blockIdx.x*blockDim.x + threadIdx.x;
    MatrixCell c = d_reducedCosts_ptr[local_id];

    // Check bounds and if this reduced cost is negative
    // Bound is number of parallel pivots that need to be performed
    if (local_id < bound && c.cost < -10e-3 ) { 

        int V = numSupplies + numDemands;
        int offset = V * local_id;
        int local_row = c.row;
        int local_col = c.col;
        int _depth = 1;
        backtracker[offset] = local_row;
        
        // then pivot row is - local_row
        // and  pivot col is - local_col

        perform_dfs_sequencial_on_i(d_adjMtx_ptr, d_vertex_start, d_vertex_degree, d_adjVertices,
                &stack[offset], &backtracker[offset], 
                &visited[offset], &_depth, local_col + numSupplies, local_row, V);
    
        if (_depth > 1) {
    
            // A loop was found - complete the book-keeping
            backtracker[offset + _depth] = local_row;
    
            // Update depth and savings for referncing in subsequent kernel //
            depth[local_id] = _depth;
    
        }
        // else depth[local_id] = 0 (remains default)
    }
}


/*
Fetch and view all parallel discovered cycles 
Function: Copy depth, backtrack from device and print
*/
__host__ void __debug_utility_1(MatrixCell * d_reducedCosts_ptr, int * backtracker, int * depth,  
    int iteration, int numSupplies, int numDemands, int num_threads_launching) 
{
    std::cout<<"DEBUG UTIITY - 1 | Viewing Discovered Loops"<<std::endl;
    int V = numSupplies + numDemands;

    int * h_backtracker = (int *) malloc(num_threads_launching * V * sizeof(int));
    int * h_depth = (int *) malloc(num_threads_launching * sizeof(int));
    MatrixCell * h_reduced_costs = (MatrixCell *) malloc(num_threads_launching * sizeof(MatrixCell));

    int num_cycles = 0;
    
    hipMemcpy(h_backtracker, backtracker, num_threads_launching * V * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_depth, depth, num_threads_launching * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_reduced_costs, d_reducedCosts_ptr, num_threads_launching * sizeof(MatrixCell), hipMemcpyDeviceToHost);

    for (int i=0; i < num_threads_launching; i++) {
        int offset = V*i;
        if (h_depth[i] > 0) {
            std::cout<<"Iteration : "<<iteration<<" : Thread : "<<i<<" : Depth : "<<h_depth[i]<<" : ";
            for (int j = 0; j <= h_depth[i]; j++) {
                std::cout<<h_backtracker[offset+j]<<" ";
            }
            std::cout<<std::endl;
            // std::cout<<"\t Loop Minimum = "<<h_loop_minimum[i]<<" From :"<<h_loop_min_from[i]<<" To : "<<h_loop_min_to[i]<<std::endl;
            std::cout<<"\t Reduced Cost Row = "<<h_reduced_costs[i].row<<std::endl;
            std::cout<<"\t Reduced Cost Col = "<<h_reduced_costs[i].col<<std::endl;
            std::cout<<"\t Reduced Cost = "<<h_reduced_costs[i].cost<<std::endl;
            num_cycles++;
        }
    }

    free(h_backtracker);
    free(h_depth);
    free(h_reduced_costs);

    std::cout<<"\n"<<num_cycles<<" cycles were discovered!"<<std::endl;
    // *********************** END OF DEBUG UTILITY - 1 *************** //
}

/*
API to execute parallel pivot, this uses DFS for dicovering the loops and then
gets the loops to execute pivot on the host (dehati way) - ask Mohit why is this a dehati way

This is meant primarily for testing and deriving insights on parallel cycles
*/
__host__ void perform_a_parallel_pivot(PivotHandler &pivot, PivotTimer &timer, 
    Graph &graph, MatrixCell * d_reducedCosts_ptr, bool &result, int numSupplies, int numDemands, int iteration) {
    
    // Check if termination criteria achieved 
    // (lowest reduced cost is positive)
    MatrixCell min_reduced_cost;
    // have all the reduced costs in the d_reducedCosts_ptr on device
    thrust::sort(thrust::device,
            d_reducedCosts_ptr, d_reducedCosts_ptr + (numSupplies*numDemands), compareCells());
    gpuErrchk(hipMemcpy(&min_reduced_cost, &d_reducedCosts_ptr[0], sizeof(MatrixCell), hipMemcpyDeviceToHost));
    // Termination criteria achieved
    if (!(min_reduced_cost.cost < 0 && std::abs(min_reduced_cost.cost) > 10e-3)) {    
        result = true;
        std::cout<<"Pivoting Complete!"<<std::endl;
        return;
    }

    auto _pivot_start = std::chrono::high_resolution_clock::now();
    auto _pivot_end = std::chrono::high_resolution_clock::now();
    auto _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    
    /*
    Strategy is to execute multiple pivots at the same time
        Step 1 : Go to all the cells with negative reduced costs -> find the cycles (Discover cycles)
        Step 2 : Whatever cycles were discovered, get them on host
        Step 3 : Execute cycles one by one and discard the ones that conflict with the ones discovered earlier 
    */

    // Discover Cycles
    _pivot_start = std::chrono::high_resolution_clock::now();
    
    int num_threads_launching = NUM_THREADS_LAUNCHING(numSupplies, numDemands, PARALLEL_PIVOT_IDEA);

    dim3 __blockDim(blockSize, 1, 1);
    dim3 __gridDim(ceil(1.0*num_threads_launching/blockSize), 1, 1);
    
    // Set initial values
    thrust::fill(thrust::device, pivot.depth, pivot.depth + (num_threads_launching), 0);
    thrust::fill(thrust::device, pivot.visited, pivot.visited + (graph.V * num_threads_launching), false);

    find_loops <<<__gridDim, __blockDim>>> (d_reducedCosts_ptr, graph.d_adjMtx_ptr, graph.d_flowMtx_ptr, // Lookups 
        graph.d_vertex_start, graph.d_vertex_degree, graph.d_adjVertices,
        pivot.stack, pivot.visited,  // Intermediates
        pivot.backtracker, pivot.depth, // book-keeping
        numSupplies, numDemands, num_threads_launching); // Params
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize()); 
        // xxxxxx - Barrier 1 - xxxxxx
    
    // DEBUG UTILITY 1 ::
    // __debug_utility_1(d_reducedCosts_ptr, pivot.backtracker, pivot.depth, iteration, 
    // numSupplies, numDemands, num_threads_launching);

    _pivot_end = std::chrono::high_resolution_clock::now();
    _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    timer.cycle_discovery += _pivot_duration.count();
    
    /* ******************************
        Multi-Pivot Method
    ******************************* */

    // Copy Discovered cycles to host and sequencially execute pivots on the host 
    // Making sure no edge is used twice // 
    bool * edge_visited = (bool *) malloc(numSupplies*numDemands*sizeof(bool)); 
    
    int * h_backtracker = (int *) malloc(num_threads_launching * graph.V * sizeof(int));
    int * h_depth = (int *) malloc(num_threads_launching * sizeof(int));
    MatrixCell * h_reduced_costs = (MatrixCell *) malloc(num_threads_launching * sizeof(MatrixCell));

    _pivot_start = std::chrono::high_resolution_clock::now();

    int num_cycles_pivoted = 0;
    thrust::fill(thrust::host, edge_visited, edge_visited + (numSupplies*numDemands), false);
    
    gpuErrchk(hipMemcpy(h_backtracker, pivot.backtracker, num_threads_launching * graph.V * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_depth, pivot.depth, num_threads_launching * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_reduced_costs, d_reducedCosts_ptr, num_threads_launching * sizeof(MatrixCell), hipMemcpyDeviceToHost));
    
    // In the running workflow, we start with most negative reduced cost and proceed thereafter 
    for (int i=0; i < num_threads_launching; i++) {

        int offset = graph.V*i;
        if (h_depth[i] > 0) {
            // check if all the edges are available >> 
            bool cycle_valid = true; 
            int _edge_from, _edge_to, _id;

            for (int j = 0; j <= h_depth[i]-1; j++) {
                
                _edge_from = h_backtracker[offset+j] - numSupplies*(j%2);
                _edge_to = h_backtracker[offset+j+1] - numSupplies*((j+1)%2);
                _id = (_edge_from*numDemands + _edge_to)*((j+1)%2) + (_edge_to*numDemands + _edge_from)*(j%2);
                cycle_valid = (cycle_valid && !(edge_visited[_id]));
                // No need to check further if already found an edge that has been used
                if (!cycle_valid) {
                    // std::cout<<"break"<<std::endl;
                    break;
                }
            }

            if (cycle_valid) {

                // Mark edges in thie cycles as used >>
                #pragma omp parallel
                #pragma omp for
                for (int j = 0; j <= h_depth[i]-1; j++) {
                
                    _edge_from = h_backtracker[offset+j] - numSupplies*(j%2);
                    _edge_to = h_backtracker[offset+j+1] - numSupplies*((j+1)%2);
                    _id = (_edge_from*numDemands + _edge_to)*((j+1)%2) + (_edge_to*numDemands + _edge_from)*(j%2);
                    edge_visited[_id] = true;
                
                }

                #pragma omp barrier

                execute_pivot_on_host_device(graph.h_adjMtx_ptr, graph.h_flowMtx_ptr, graph.d_adjMtx_ptr, graph.d_flowMtx_ptr, 
                    &h_backtracker[offset], h_reduced_costs[i].row, h_reduced_costs[i].col, h_depth[i], 
                    graph.V, numSupplies, numDemands);

                num_cycles_pivoted++;

            }
        }
    }

    _pivot_end = std::chrono::high_resolution_clock::now();
    _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    timer.adjustment_time += _pivot_duration.count();

    free(h_backtracker);
    free(h_depth);
    free(h_reduced_costs);

    // End of multi-pivot
    // std::cout<<"Iteration : "<<iteration<<" | Number of cycles pivoted : "<<num_cycles_pivoted<<std::endl;

}


// ***********************************************************************

__host__ void _debug_print_APSP(int * d_adjMtx, int * d_pathMtx, int V) {

    int * h_adjMtx_copy = (int *) malloc(sizeof(int)*V*V);
    int * h_pathMtx = (int *) malloc(sizeof(int)*V*V);

    gpuErrchk(hipMemcpy(h_adjMtx_copy, d_adjMtx, V*V*sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_pathMtx, d_pathMtx, V*V*sizeof(int), hipMemcpyDeviceToHost));
	
    std::cout<<" ********* Distances >>"<<std::endl;
    for (int i=0; i<V; i++) {
        std::cout<<i<<" : ";
        for (int j=0; j<V; j++) {
            std::cout<<h_adjMtx_copy[i*V + j]<<", ";
        }
        std::cout<<std::endl;
    }
    std::cout<<" ********* Path >>"<<std::endl;
    for (int i=0; i<V; i++) {
        std::cout<<i<<" : ";
        for (int j=0; j<V; j++) {
            std::cout<<h_pathMtx[i*V + j]<<", ";
        }
        std::cout<<std::endl;
    }
    std::cout << "All point shortest path printed!"<<std::endl;
}

/*
Step 1: Find all point to all points shortest distance with Floyd Warshall using naive implementation 
    of Floyd Warshall algorithm in CUDA

- Step 2: For all negative reduced costs find the paths
- Step 3: Find edge disjoint paths among the ones obtained in 2
- Step 4: Perfrom flow adjustment on the paths
*/
__host__ void perform_a_parallel_pivot_floyd_warshall(PivotHandler &pivot, PivotTimer &timer, 
    Graph &graph, MatrixCell * d_reducedCosts_ptr, bool &result, int numSupplies, int numDemands, int iteration) {
    
    // Here no need to sort reduced costs - check for termination criteria

    MatrixCell min_reduced_cost;
    
    // Find index of most negative reduced cost negative reduced cost >>
    int min_indx = thrust::min_element(thrust::device,
                d_reducedCosts_ptr, d_reducedCosts_ptr + (numSupplies*numDemands), compareCells()) - d_reducedCosts_ptr;
    gpuErrchk(hipMemcpy(&min_reduced_cost, &d_reducedCosts_ptr[min_indx], sizeof(MatrixCell), hipMemcpyDeviceToHost));

    if (!(min_reduced_cost.cost < 0 && std::abs(min_reduced_cost.cost) > 10e-3)) {
    
        // Terminate
        result = true;
        std::cout<<"Pivoting Complete!"<<std::endl;
        return;
    
    }

    auto _pivot_start = std::chrono::high_resolution_clock::now();
    auto _pivot_end = std::chrono::high_resolution_clock::now();
    auto _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);

    // Discover Cycles
    int * d_adjMtx_transform, * d_pathMtx;
    int simplex_gridDim = graph.V*graph.V;

    // Allocate memory for floyd warshall
	gpuErrchk(hipMalloc((void **) &d_adjMtx_transform, simplex_gridDim*sizeof(int)));
	gpuErrchk(hipMalloc((void **) &d_pathMtx, simplex_gridDim*sizeof(int)));
	
    _pivot_start = std::chrono::high_resolution_clock::now();

	// Make a copy of adjacency matrix to make depth
    // IDEA: run my_signum all at once to get rid of that in the floyd warshall kernel - insted of memcpy run a kernel	
	thrust::fill(thrust::device, d_pathMtx, d_pathMtx + simplex_gridDim, -1);
    
    dim3 dimBlock(blockSize, blockSize, 1);
    dim3 dimGrid(ceil(1.0*graph.V/blockSize),ceil(1.0*graph.V/blockSize),1);
    
    fill_adjMtx <<< dimGrid, dimBlock >>> (d_adjMtx_transform, graph.d_adjMtx_ptr, d_pathMtx, graph.V);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // Initialize the grid and block dimensions here
    dim3 dimGrid2((graph.V - 1) / blockSize + 1, (graph.V - 1) / blockSize + 1, 1);
    dim3 dimBlock2(blockSize, blockSize, 1);

    // /* hipFuncSetCacheConfig(reinterpret_cast<const void*>(_naive_fw_kernel), hipFuncCachePreferL1); */
    for (int vertex = 0; vertex < graph.V; ++vertex) {
        _naive_floyd_warshall_kernel <<< dimGrid2, dimBlock2 >>> (vertex, graph.V, d_adjMtx_transform, d_pathMtx);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }

    // DEBUG UTILITY >>
    // _debug_print_APSP(d_adjMtx_ptr_copy, d_pathMtx, V);

    _pivot_end = std::chrono::high_resolution_clock::now();
    _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    timer.cycle_discovery += _pivot_duration.count();

    std::cout<<"APSP Completed in "<<timer.cycle_discovery<<" microseconds"<<std::endl;
    exit(0);

}


/******************** DUMP ***********************

/*
Fetch and view v_owner and v_savings 
Function: Copy arrays from device and print
*/
__host__ void __debug_utility_2(vertex_conflicts * v_conflicts, int numSupplies, int numDemands)
{
    std::cout<<"DEBUG UTIITY - 2 | Viewing Loop Owners"<<std::endl;

    vertex_conflicts * h_v_savings = (vertex_conflicts *) malloc(numSupplies * numDemands * sizeof(vertex_conflicts));
    hipMemcpy(h_v_savings, v_conflicts,  numSupplies * numDemands * sizeof(vertex_conflicts), hipMemcpyDeviceToHost);

    for (int i=0; i < numSupplies; i++) {
        for (int j=0; j < numDemands; j++) {
            std::cout << "Edge - (" <<i<<", "<<j+numSupplies<< ") by Thread : " << h_v_savings[i*numDemands + j].ints[1]<< std::endl;
        }
    }
    // *********************** END OF DEBUG UTILITY - 2 *************** //
}

__host__ void __debug_utility_3(int * backtracker, int * depth, 
    int numSupplies, int numDemands, int num_threads_launching) 
{
    // // *********************** DEBUG UTILITY - 3 *************** //
    // // Fetch and view the loops that do not conflict and maximize savings 
    
    std::cout<<"DEBUG UTIITY - 3 | Viewing Non-Conflicting loops"<<std::endl;
    int V = numSupplies + numDemands;
    int * h_backtracker = (int *) malloc(num_threads_launching * V * sizeof(int));
    int * h_depth = (int *) malloc(num_threads_launching * sizeof(int));
    int num_cycles = 0;
    
    hipMemcpy(h_backtracker, backtracker, num_threads_launching * V * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_depth, depth, num_threads_launching * sizeof(int), hipMemcpyDeviceToHost);

    for (int i=0; i < num_threads_launching; i++) {
        int offset = V*i;
       if (h_depth[i] > 0){
            std::cout<<"Thread : "<<i<<" : Depth : "<<h_depth[i]<<" : ";
            for (int j = 0; j < h_depth[i]; j++) {
                std::cout<<h_backtracker[offset+j]<<" ";
            }
            std::cout<<std::endl;
           num_cycles++;
       }
    }

    free(h_backtracker);
    free(h_depth);

    std::cout<<"\n"<<num_cycles<<" non conflicting cycles were discovered!"<<std::endl;
    // *********************** END OF DEBUG UTILITY - 3 *************** //
}




/*
The Novel Conflict Selector >>
Reference: https://stackoverflow.com/questions/17411493/how-can-i-implement-a-custom-atomic-function-involving-several-variables 
*/
__device__ unsigned long long int atomicMinAuxillary(unsigned long long int* address, float val1, int val2)
{
    vertex_conflicts loc, loctest;
    loc.floats[0] = val1;
    loc.ints[1] = val2;
    loctest.ulong = *address;
    while (val1  < loctest.floats[0] || (val1 == loctest.floats[0] && val2 < loctest.ints[1])) {
        // condition and tie-braker (bland's rule)
        loctest.ulong = atomicCAS(address, loctest.ulong,  loc.ulong);
    } 
    return loctest.ulong;
}

/*
Kernel 2 :: Step 1
Resolve conflicts |  Search for vertices that fall under conflicting loops
*/
__global__ void resolve_conflicts_step_1(int * depth, int * backtracker, MatrixCell * d_reducedCosts_ptr, 
        vertex_conflicts * v_conflicts, int numSupplies, int numDemands, int bound) {
            
    int local_id = blockIdx.y*blockDim.y + threadIdx.y;
    int V = numSupplies + numDemands;
    int offset = V * local_id;
    
    if (local_id < bound) {

        // Check if this is a cell along which pivoting is performing
        int _depth = depth[local_id];
        // Real loops exists on this edge (cell) -
        if (_depth > 0) {

            // Find Savings
            float r = d_reducedCosts_ptr[local_id].cost;
            int _edge_from, _edge_to, _id;
            // If this loop is pivoted then this is the savings you get 

            for (int i=0; i<_depth-1; i++) {

                _edge_from = backtracker[offset+i] - numSupplies*(i%2);
                _edge_to = backtracker[offset+i+1] - numSupplies*((i+1)%2);
                
                /* 
                Atomically make the comparison and assign

                Essentially the following is performed in an atomic sense
                if (_savings < v_savings[_vtx]) {
                    v_savings[_vtx] = _savings;
                    v_owner[_vtx] = local_id;
                } 
                */

                _id = (_edge_from*numDemands + _edge_to)*((i+1)%2) + (_edge_to*numDemands + _edge_from)*(i%2);
                atomicMinAuxillary(&(v_conflicts[_id].ulong), r, local_id);
            }
        }
    }
}

/*
Resolve conflicts | Step 2 : Kill threads => discard the loops => Set depth = 0
*/
__global__ void resolve_conflicts_step_2(int * depth, int * backtracker, vertex_conflicts * v_conflicts, 
        int numSupplies, int numDemands, int num_threads_launching) {
            
    int local_id = blockIdx.x*blockDim.x + threadIdx.x;
    int V = numSupplies + numDemands;
    int offset = V * local_id;
    
    if (local_id < num_threads_launching) {
        // Check if this is a cell along which pivoting is performing
        int _depth = depth[local_id];
        // Real loops exists on this edge (cell) -
        if (_depth > 0) {
            // Check continuity along all vertices in loop if v_owner is this thread itself >>
            bool _continuity = true;
            int _edge_from, _edge_to, _id, i=0;
            while (i < _depth-1 && _continuity) {
                _edge_from = backtracker[offset+i] - numSupplies*(i%2);
                _edge_to = backtracker[offset+i+1] - numSupplies*((i+1)%2);
                _id = (_edge_from*numDemands + _edge_to)*((i+1)%2) + (_edge_to*numDemands + _edge_from)*(i%2);
                _continuity = (_continuity && (v_conflicts[_id].ints[1] == local_id));
                i++;
            }

            if (!_continuity) { // Kill this thread in this case
                depth[local_id] = 0;
            }
        }
    }
}

/*
Kernel to execute the flow adjustments in parallel >>
*/
__global__ void run_flow_adjustments(int * d_adjMtx_ptr, float * d_flowMtx_ptr, int * depth, 
            int * backtracker, float * loop_minimum,
            int * loop_min_from, int * loop_min_to, int * loop_min_id,
            int numSupplies, int numDemands) {
            
    int local_row = blockIdx.y*blockDim.y + threadIdx.y;
    int local_col = blockIdx.x*blockDim.x + threadIdx.x;
    int local_id = local_row*numDemands + local_col;
    int V = numSupplies + numDemands;
    int offset = V * local_id;
    
    if (local_row < numSupplies && local_col < numDemands) {
        // Check if this is a cell along which pivoting is performing
        int _depth = depth[local_id];
        // Real loops exists on this edge (cell) -
        if (_depth > 0) {

            int _from, _to, id, j=-1, min_from = loop_min_from[local_id], 
                min_to = loop_min_to[local_id], min_flow_id = loop_min_id[local_id];
            float _flow, _min_flow = loop_minimum[local_id];

            for (int i=1; i<_depth; i++)
            {
                _from = backtracker[offset+i];
                _to = backtracker[offset+i+1];
                id = d_adjMtx_ptr[TREE_LOOKUP(_from, _to, V)] - 1;
                _flow = j*_min_flow;
                d_flowMtx_ptr[id] += _flow;
                j *= -1;
            }

            // Do the replacment between exiting i - entering j on both host and device
            // Remove edge
            id = TREE_LOOKUP(min_from, min_to, V);
            d_adjMtx_ptr[id] = 0;
            // Insert edge
            id = TREE_LOOKUP(local_row, local_col+numSupplies, V);
            d_adjMtx_ptr[id] = min_flow_id + 1;
            // Update new flow 
            d_flowMtx_ptr[min_flow_id] = _min_flow;
        }
    }
}

/*

// Resolve Conflicts >> 
    _pivot_start = std::chrono::high_resolution_clock::now();
    // std::cout<<"Parallel Pivoiting : Discovered Loops!"<<std::endl;
    // std::cout<<"Parallel Pivoiting : Resolving Conflicts | Running Step 1 (Discover conflicts) ..."<<std::endl;        
    vertex_conflicts _vtx_conflict_default;
    _vtx_conflict_default.floats[0] = FLT_MAX;
    _vtx_conflict_default.ints[1] = -1;
    thrust::fill(thrust::device, v_conflicts, v_conflicts + (numSupplies*numDemands), _vtx_conflict_default);

    resolve_conflicts_step_1 <<<__gridDim, __blockDim>>> (depth, backtracker, d_reducedCosts_ptr, 
        v_conflicts, numSupplies, numDemands, num_threads_launching);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
        // xxxxxx - Barrier 2 - xxxxxx

    // DEBUG UTILITY 2 ::
    // __debug_utility_2(v_conflicts, numSupplies, numDemands);
    
    // std::cout<<"Parallel Pivoiting : Completed Step 1 | Running Step 2 (Resolve Conflicts) ..."<<std::endl;
    resolve_conflicts_step_2 <<<__gridDim, __blockDim>>> (depth, backtracker, v_conflicts, numSupplies, numDemands, num_threads_launching);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    // xxxxxx - Barrier 3 - xxxxxx

    // DEBUG UTILITY 3 ::
    __debug_utility_3(backtracker, depth, numSupplies, numDemands, num_threads_launching);
    exit(0);
    _pivot_end = std::chrono::high_resolution_clock::now();
    _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    resolve_time += _pivot_duration.count();

    // std::cout<<"Parallel Pivoiting : Conflicts Resolved | Running flow adjustments ..."<<std::endl;        
    // Check if any conflicting pivots still exist >>
    _pivot_start = std::chrono::high_resolution_clock::now();
    int _conflict_flag = thrust::reduce(thrust::device, depth, depth + (numSupplies*numDemands), 0);
        
    std::cout<<"THIS PIVOTING METHOD IS OUT DATED, TRY - hybrid!"<<std::endl;
    exit(-1);
    run_flow_adjustments <<<__gridDim, __blockDim>>> (d_adjMtx_ptr, d_flowMtx_ptr, depth, backtracker, loop_minimum, 
        loop_min_from, loop_min_to, loop_min_id, numSupplies, numDemands);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
        // xxxxxx - Barrier 4 - xxxxxx

    _pivot_end = std::chrono::high_resolution_clock::now();
    _pivot_duration = std::chrono::duration_cast<std::chrono::microseconds>(_pivot_end - _pivot_start);
    adjustment_time += _pivot_duration.count();
*/
