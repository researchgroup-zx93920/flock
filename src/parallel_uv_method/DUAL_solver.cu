#include "hip/hip_runtime.h"
#include "DUAL_solver.h"


__host__ void initialize_device_DUAL(float ** u_vars_ptr, float ** v_vars_ptr, 
        Variable ** U_vars, Variable ** V_vars, 
        int ** length, int ** start, int ** Ea, bool ** Fa, bool ** Xa, float ** variables,
        float ** d_csr_values, int ** d_csr_columns, int ** d_csr_offsets,
        float ** d_A, float ** d_b, float ** d_x, int64_t &nnz, 
        int ** h_length, int ** h_start, int ** h_Ea, bool ** h_visited, float ** h_variables,
        int numSupplies, int numDemands) {
    
    int V = numSupplies + numDemands;
    // Create and Initialize u and v variables 
    // To be allocated regardless 
    gpuErrchk(hipMalloc((void **) u_vars_ptr, sizeof(float)*numSupplies));
    gpuErrchk(hipMalloc((void **) v_vars_ptr, sizeof(float)*numDemands));

    if (CALCULATE_DUAL=="tree") {
        
        std::cout<<"Tree is deprecated, USE bfs INSTEAD!"<<std::endl;
        exit(-1);
        //  empty u and v equations using the Variable Data Type >>
        gpuErrchk(hipMalloc((void **) U_vars, sizeof(Variable)*numSupplies));
        gpuErrchk(hipMalloc((void **) V_vars, sizeof(Variable)*numDemands));
    }

    else if (CALCULATE_DUAL=="bfs") {

        //  empty u and v equations using the Variable Data Type >>
        gpuErrchk(hipMalloc((void **) length, sizeof(int)*(V+1)));
        gpuErrchk(hipMalloc((void **) start, sizeof(int)*(V)));
        gpuErrchk(hipMalloc((void **) Ea, sizeof(int)*2*(V-1)));
        gpuErrchk(hipMalloc((void**) Fa, sizeof(bool)*V));
        gpuErrchk(hipMalloc((void**) Xa, sizeof(bool)*V));
        gpuErrchk(hipMalloc((void**) variables, sizeof(float)*V));

    }

    else if (CALCULATE_DUAL=="bfs_seq") {

        //  empty u and v equations using the Variable Data Type >>
        gpuErrchk(hipMalloc((void **) length, sizeof(int)*(V+1)));
        gpuErrchk(hipMalloc((void **) start, sizeof(int)*(V)));
        gpuErrchk(hipMalloc((void **) Ea, sizeof(int)*2*(V-1)));
        * h_length = (int *) malloc(sizeof(int)*V);
        * h_start = (int *) malloc(sizeof(int)*V);
        * h_Ea = (int *) malloc(sizeof(int)*2*(V-1));
        * h_visited = (bool *) malloc(sizeof(bool)*V);
        * h_variables = (float *) malloc(sizeof(float)*V);
    }

    else if (CALCULATE_DUAL=="sparse_linear_solver") {

        int U_0 = 0;
        float U_0_value = 0.0;

        // Allocate memory to store the sparse linear system
        nnz = 2*V - 1;

        // Values are coefs of u and v, which are always one only position and b-vector changes with iterations, So
        gpuErrchk(hipMalloc((void**) d_csr_values,  nnz * sizeof(float)));
        thrust::fill(thrust::device, *d_csr_values, (*d_csr_values) + nnz, 1.0);

        // U_0 is always set to zero - meaning first element is always 0,0 in csr
        gpuErrchk(hipMalloc((void**) d_csr_columns, nnz * sizeof(int)));
        gpuErrchk(hipMemcpy(*d_csr_columns, &U_0, sizeof(int), hipMemcpyHostToDevice));

        // The row pointers also remain constant {0,1,3,5, ... , 2V-1}, Custom Filler kernel below
        gpuErrchk(hipMalloc((void**) d_csr_offsets, (V + 1) * sizeof(int)));
        fill_csr_offset <<< ceil(1.0*(V+1)/blockSize), blockSize >>> (*d_csr_offsets, V+1);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        gpuErrchk(hipMalloc((void **) d_b, sizeof(float)*V));
        gpuErrchk(hipMemcpy(*d_b, &U_0_value, sizeof(float), hipMemcpyHostToDevice));
        
        // d_x is only allocated here - it is to be populated by API's
        gpuErrchk(hipMalloc((void **) d_x, V * sizeof(float)));
    }

    else if (CALCULATE_DUAL=="dense_linear_solver") {

        // Allocate memory to store the dense linear system
        gpuErrchk(hipMalloc((void **) d_A, sizeof(float)*V*V));
        gpuErrchk(hipMalloc((void **) d_b, sizeof(float)*V));
        gpuErrchk(hipMalloc((void **) d_x, V * sizeof(float)));

    }
}

__host__ void terminate_device_DUAL(float * u_vars_ptr, float * v_vars_ptr, 
        Variable * U_vars, Variable * V_vars, 
        int * length, int * start, int * Ea, bool * Fa, bool * Xa, float * variables,
        float * d_csr_values, int * d_csr_columns, int * d_csr_offsets,
        float * d_A, float * d_b, float * d_x, 
        int * h_length, int * h_start, int * h_Ea, bool * h_visited, float * h_variables) {
     
        gpuErrchk(hipFree(u_vars_ptr));
        gpuErrchk(hipFree(v_vars_ptr));
        
        if (CALCULATE_DUAL=="tree") {
        
                gpuErrchk(hipFree(U_vars));
                gpuErrchk(hipFree(V_vars));
        
        }
        
        else if (CALCULATE_DUAL=="bfs") {
                
                gpuErrchk(hipFree(length));
                gpuErrchk(hipFree(start));
                gpuErrchk(hipFree(Ea));
                gpuErrchk(hipFree(Fa));
                gpuErrchk(hipFree(Xa));
                gpuErrchk(hipFree(variables));
        }

        else if (CALCULATE_DUAL=="bfs_seq") {
                
                gpuErrchk(hipFree(length));
                gpuErrchk(hipFree(start));
                gpuErrchk(hipFree(Ea));
                free(h_length);
                free(h_start);
                free(h_Ea);
                free(h_visited);
                free(h_variables);
        }
        
        else if (CALCULATE_DUAL=="sparse_linear_solver") {

        gpuErrchk(hipFree(d_csr_values));
        gpuErrchk(hipFree(d_csr_columns));
        gpuErrchk(hipFree(d_csr_offsets));
        gpuErrchk(hipFree(d_b));
        gpuErrchk(hipFree(d_x));
        
        }
        
        else if (CALCULATE_DUAL=="dense_linear_solver") {

        gpuErrchk(hipFree(d_A));
        gpuErrchk(hipFree(d_b));
        gpuErrchk(hipFree(d_x));

        }
}

__host__ void find_dual_using_tree(float * u_vars_ptr, float * v_vars_ptr, 
        int * d_adjMtx_ptr, float * d_costs_ptr, Variable * U_vars, Variable * V_vars, 
        int numSupplies, int numDemands) {

        initialize_U_vars<<<ceil(1.0*numSupplies/blockSize), blockSize>>>(U_vars, numSupplies);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        initialize_V_vars<<<ceil(1.0*numDemands/blockSize), blockSize>>>(V_vars, numDemands);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        
        // Set u[0] = 0 on device >> // This can be done more smartly - low prioirity
        Variable default_variable;
        default_variable.assigned = true;
        default_variable.value = 0;
        gpuErrchk(hipMemcpy(U_vars, &default_variable, sizeof(Variable), hipMemcpyHostToDevice));

        // Perform the assignment
        dim3 __blockDim(blockSize, blockSize, 1); 
        dim3 __gridDim(ceil(1.0*numDemands/blockSize), ceil(1.0*numSupplies/blockSize), 1);
        for (int i=0; i < (numSupplies+numDemands-1); i++) {
                assign_next <<< __gridDim, __blockDim >>> (d_adjMtx_ptr, d_costs_ptr, 
                U_vars, V_vars, numSupplies, numDemands);
                gpuErrchk(hipPeekAtLastError());
                gpuErrchk(hipDeviceSynchronize()); // Potential performance bottleneck
        }

        // Once done - copy the final values to u_vars_ptr and v_vars_ptr and free device memory
        // This one dumps the unnecessary data associated with equation solve
        copy_row_shadow_prices<<<ceil(1.0*numSupplies/blockSize), blockSize>>>(U_vars, u_vars_ptr, numSupplies);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        
        copy_row_shadow_prices<<<ceil(1.0*numDemands/blockSize), blockSize>>>(V_vars, v_vars_ptr, numDemands);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
}


__host__ void find_dual_using_sparse_solver(float * u_vars_ptr, float * v_vars_ptr, 
        float * d_costs_ptr, int * d_adjMtx_ptr,
        float * d_csr_values, int * d_csr_columns, int * d_csr_offsets, float * d_x, float * d_b, 
        int64_t nnz, int numSupplies, int numDemands)
{
        int V = numSupplies + numDemands;

        // Nice thing is that csr values and offsets remain static over the iterations
        dim3 __blockDim(blockSize, blockSize, 1); 
        dim3 __gridDim(ceil(1.0*numDemands/blockSize), ceil(1.0*numSupplies/blockSize), 1);
        initialize_sparse_u_v_system <<< __gridDim, __blockDim >>> (d_csr_columns, d_b, d_adjMtx_ptr, d_costs_ptr, 
                numSupplies, numDemands);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());


        /* *********************
        DEBUG UTILITY :: Print the csr matrix for u-v system
         ************************/
        // float * h_csr_values = (float *) malloc(sizeof(float)*nnz);
        // int * h_csr_columns = (int *) malloc(sizeof(int)*nnz);
        // int * h_csr_offsets = (int *) malloc(sizeof(int)*(V+1));
        // gpuErrchk(hipMemcpy(h_csr_values, d_csr_values, sizeof(float)*nnz, hipMemcpyDeviceToHost));
        // gpuErrchk(hipMemcpy(h_csr_columns, d_csr_columns, sizeof(int)*nnz, hipMemcpyDeviceToHost));
        // gpuErrchk(hipMemcpy(h_csr_offsets, d_csr_offsets, sizeof(int)*(V+1), hipMemcpyDeviceToHost));
        // std::cout<<"CSR Values = [";
        // for (int i =0; i< nnz; i++){
        //         std::cout<<h_csr_values[i]<<", ";
        // }
        // std::cout<<"]"<<std::endl;
        // std::cout<<"CSR Columns = [";
        // for (int i =0; i< nnz; i++){
        //         std::cout<<h_csr_columns[i]<<", ";
        // }
        // std::cout<<"]"<<std::endl;
        // std::cout<<"CSR Offsets = [";
        // for (int i =0; i < V+1; i++){
        //         std::cout<<h_csr_offsets[i]<<", ";
        // }
        // std::cout<<"]"<<std::endl;
        /* ********** END OF UTILITY ************* */

        // Core >>		
        hipsolverSpHandle_t solver_handle;
	CUSOLVER_CHECK(hipsolverSpCreate(&solver_handle));

        hipsparseMatDescr_t descrA;
        CUSPARSE_CHECK(hipsparseCreateMatDescr(&descrA));
	CUSPARSE_CHECK(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
	CUSPARSE_CHECK(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO)); 

        int singularity;
        // Cholesky : 1 : symrcm, 2 : symamd, or 3 : csrmetisnd
        int reorder = 0;

        if (SPARSE_SOLVER=="qr") {
                
                CUSOLVER_CHECK(hipsolverSpScsrlsvqr(solver_handle, V, nnz, descrA, 
                                        d_csr_values, d_csr_offsets, d_csr_columns, d_b, 
                                        10e-9, reorder, d_x, &singularity));
        
        }

	else if (SPARSE_SOLVER=="chol") {

                CUSOLVER_CHECK(hipsolverSpScsrlsvchol(solver_handle, V, nnz, descrA,
                     d_csr_values, d_csr_offsets, d_csr_columns, d_b,
                     10e-9, reorder, d_x, &singularity));

        }

        else {
        
                std::cout<<" Invalid sparse solver!"<<std::endl;
                exit(0);
        }
        
        // Clean up ! 
        CUSOLVER_CHECK(hipsolverSpDestroy(solver_handle));
        CUSPARSE_CHECK(hipsparseDestroyMatDescr(descrA));

        if (singularity == -1) {

                dim3 __blockDim2(blockSize, 1, 1);
                dim3 __gridDim2(ceil(1.0*V/blockSize), 1, 1);
                retrieve_uv_solution <<< __gridDim2, __blockDim2 >>> (d_x, u_vars_ptr, v_vars_ptr, numSupplies, numDemands);
                gpuErrchk(hipPeekAtLastError());
                gpuErrchk(hipDeviceSynchronize());
        }
        
        else {
        
                std::cout<<" ========== !! Unexpected ERROR :: Matrix A is singular !!"<<std::endl;
                std::cout<<" ========== Return singularity = "<<singularity<<std::endl;
                exit(0);
                // float * h_x = (float *) malloc(sizeof(float)*V);
                // hipMemcpy(h_x, d_x, sizeof(float)*V, hipMemcpyDeviceToHost);
                // for (int i=0; i<V; i++) {
                //     std::cout<< "X [" <<i<<"] = "<<h_x[i]<<std::endl;
                // }
        }
}


__host__ void find_dual_using_dense_solver(float * u_vars_ptr, float * v_vars_ptr, 
        float * d_costs_ptr, int * d_adjMtx_ptr,
        float * d_A, float * d_x, float * d_b, 
        int numSupplies, int numDemands) 
{
        int V = numSupplies + numDemands;
        thrust::fill(thrust::device, d_A, d_A + (V * V), 0.0f);
        thrust::fill(thrust::device, d_b, d_b + (V), 0.0f);

        // Nice thing is that csr values and offsets remain static over the iterations
        dim3 __blockDim(blockSize, blockSize, 1); 
        dim3 __gridDim(ceil(1.0*numDemands/blockSize), ceil(1.0*numSupplies/blockSize), 1);
        initialize_dense_u_v_system <<< __gridDim, __blockDim >>> (d_A, d_b, d_adjMtx_ptr, d_costs_ptr, 
                numSupplies, numDemands);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

}

/*
DEBUG UTILITY : VIEW ADJACENCY LIST STRCTURE 
*/
__host__ void __debug_view_adjList(int * length, int * start, int * Ea, int V) 
{        
        int * h_length = (int *) malloc(sizeof(int)*V);
        int * h_start = (int *) malloc(sizeof(int)*V);
        int * h_Ea = (int *) malloc(sizeof(int)*2*(V-1));

        hipMemcpy(h_length, length, sizeof(int)*V, hipMemcpyDeviceToHost);
        hipMemcpy(h_start, start, sizeof(int)*V, hipMemcpyDeviceToHost);
        hipMemcpy(h_Ea, Ea, sizeof(int)*2*(V-1), hipMemcpyDeviceToHost);

        std::cout<<"Str = [ ";
        for (int i =0; i < V; i++){
                std::cout<<h_start[i]<<", ";
        }
        std::cout<<"]"<<std::endl;
        std::cout<<"Len = [ ";
        for (int i =0; i < V; i++){
                std::cout<<h_length[i]<<", ";
        }
        std::cout<<"]"<<std::endl;
        std::cout<<"Ea = [ ";
        for (int i =0; i < 2*(V-1); i++){
                std::cout<<h_Ea[i]<<", ";
        }
        std::cout<<"]"<<std::endl;
        // *************** END OF DEBUG UTILITY ***************
}


__host__ void find_dual_using_bfs(float * u_vars_ptr, float * v_vars_ptr, 
        int * length, int * start, int * Ea, bool * Fa, bool * Xa, float * variables,
        int * d_adjMtx_ptr, float * d_costs_ptr, int numSupplies, int numDemands) {

        int V = numSupplies + numDemands;
        bool f0 = true;

        // Kernel Dimensions >>
        dim3 __blockDim(blockSize, 1, 1); 
        dim3 __gridDim(ceil(1.0*V/blockSize), 1, 1);

        determine_length <<< __gridDim, __blockDim >>> (length, d_adjMtx_ptr, V);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        
        thrust::inclusive_scan(thrust::device, length, length + V, start);
        
        fill_Ea <<< __gridDim, __blockDim >>> (start, Ea, d_adjMtx_ptr, V, numSupplies);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        int * _length = &length[1];

        // DEBUG ::
        // __debug_view_adjList(_length, start, Ea, V);
        
	thrust::fill(thrust::device, Fa, Fa + V, false);
        thrust::fill(thrust::device, Xa, Xa + V, false);
        thrust::fill(thrust::device, variables, variables + V, 0.0);
        gpuErrchk(hipMemcpy(&Fa[0], &f0, sizeof(bool), hipMemcpyHostToDevice));

        // >>> Running BFS
        // std::cout<<"Running BFS"<<std::endl;
        bool done;
	bool * d_done;
	gpuErrchk(hipMalloc((void**) &d_done, sizeof(bool)));
	int count = 0;

	do {
		count++;
		done = true;
		gpuErrchk(hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice));
		CUDA_BFS_KERNEL <<<__gridDim, __blockDim >>>(start, _length, Ea, Fa, Xa, variables, d_costs_ptr, d_done, numSupplies, numDemands, V);
		gpuErrchk(hipPeekAtLastError());
                gpuErrchk(hipDeviceSynchronize());
                gpuErrchk(hipMemcpy(&done, d_done , sizeof(bool), hipMemcpyDeviceToHost));

	} while (!done && count < (numSupplies+numDemands-1));

        // std::cout<<"BFS Complete!"<<std::endl;
	gpuErrchk(hipMemcpy(u_vars_ptr, &variables[0], sizeof(float)*numSupplies, hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(v_vars_ptr, &variables[numSupplies], sizeof(float)*numDemands, hipMemcpyDeviceToDevice));
}

__host__ void find_dual_using_seq_bfs(float * u_vars_ptr, float * v_vars_ptr, 
        int * length, int * start, int * Ea, int * d_adjMtx_ptr, float * h_costs_ptr, 
        int * h_length, int * h_start, int * h_Ea, bool * h_visited, float * h_variables,
        int numSupplies, int numDemands) {

        int V = numSupplies + numDemands;

        // Kernel Dimensions >>
        dim3 __blockDim(blockSize, 1, 1); 
        dim3 __gridDim(ceil(1.0*V/blockSize), 1, 1);

        determine_length <<< __gridDim, __blockDim >>> (length, d_adjMtx_ptr, V);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        
        thrust::inclusive_scan(thrust::device, length, length + V, start);
        
        fill_Ea <<< __gridDim, __blockDim >>> (start, Ea, d_adjMtx_ptr, V, numSupplies);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        int * _length = &length[1];
        
        gpuErrchk(hipMemcpy(h_length, _length, sizeof(int)*V, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(h_start, start, sizeof(int)*V, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(h_Ea, Ea, sizeof(int)*2*(V-1), hipMemcpyDeviceToHost));

        thrust::fill(thrust::host, h_visited, h_visited + V, false);
        thrust::fill(thrust::host, h_variables, h_variables + V, 0.0f);

        // Initialize >>
        std::queue<int> assigned_parents;
        h_visited[0] = true;
        // The value of u0 is already zero as initialized
        assigned_parents.push(0);

        // Perform a BFS on Host (trickle down) >> 
        int parent, child, row, col;

        while (!assigned_parents.empty()) {
                parent = assigned_parents.front();
                for (int i = h_start[parent]; i < h_start[parent] + h_length[parent]; i++) {
                        child = h_Ea[i];
                        if (!h_visited[child]) {
                                h_visited[child] = true;
                                row = min(parent, child);
                                col = max(parent, child) - numSupplies;
                                h_variables[child] = h_costs_ptr[row*numDemands + col] - h_variables[parent];
                                assigned_parents.push(child);
                        }

                }
                assigned_parents.pop();
        }

        // Transfer back to GPU >> 
        gpuErrchk(hipMemcpy(u_vars_ptr, &h_variables[0], sizeof(int)*numSupplies, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(v_vars_ptr, &h_variables[numSupplies], sizeof(int)*numDemands, hipMemcpyHostToDevice));

}