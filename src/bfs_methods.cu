#include "hip/hip_runtime.h"
#include "bfs_methods.h"

void find_nw_corner_bfs_seq(int * supplies, int * demands, double * costMatrix, int * flows, 
        int matrixSupplies, int matrixDemands) {
    
    std::cout<<"Running Northwest Corner Seq BFS Method"<<std::endl;

    // Step 1 :: Jumpt to NW corner >>
    int current_row_number = 0;
    int current_col_number = 0;
    int current_demand = demands[current_row_number];
    int current_supply = supplies[current_col_number];

    // Allocate flow equal to minimum of demand and supply and update the buffer accordingly >>
    while (current_row_number < matrixSupplies && current_col_number < matrixDemands) {
        
        // std::cout<<"Current Supply Index : "<<current_row_number<<std::endl;
        // std::cout<<"Current Supply : "<<current_supply<<std::endl;
        // std::cout<<"Current Demand Index : "<<current_col_number<<std::endl;
        // std::cout<<"Current Demand : "<<current_demand<<std::endl;
        
        if (current_demand >= current_supply) {
            flows[current_row_number*matrixDemands + current_col_number] = current_supply;
            current_demand = current_demand -  current_supply;
            current_row_number++;
            current_supply = supplies[current_row_number];
        }
        else {
            flows[current_row_number*matrixDemands + current_col_number] = current_demand;
            current_supply = current_supply -  current_demand;
            current_col_number++;
            current_demand = demands[current_col_number];
        }
    }
    std::cout<<"Feasible BFS Generated!"<<std::endl;
}

void find_vogel_bfs_seq(int * supplies, int * demands, double * costMatrix, int * flows, 
        int matrixSupplies, int matrixDemands) {
    
    std::cout<<"Vogel's Approximation seq BFS Method"<<std::endl;
    // Book-keeping stuff >>
    int coveredRows = 0 , coveredColumns = 0;
    int *residual_supply = (int *) malloc(matrixSupplies*sizeof(int));
    std::memcpy(residual_supply, supplies, matrixSupplies*sizeof(int));

    int *residual_demand = (int *) malloc(matrixDemands*sizeof(int));
    std::memcpy(residual_demand, demands, matrixDemands*sizeof(int));

    int * rowCovered = (int *) calloc(matrixSupplies, sizeof(int));
    int * colCovered = (int *) calloc(matrixDemands, sizeof(int));    
    int * differences = (int *) calloc(matrixSupplies + matrixDemands, sizeof(int));
    std::cout<<"\tCreated all book-keeping structs"<<std::endl;

    std::cout<<"\tIterating Vogel's Heuristic"<<std::endl;
    while (coveredRows + coveredColumns < matrixDemands+matrixSupplies-1) {
        // std::cout<<"Iteration - "<<coveredColumns+coveredRows<<std::endl;
        double temp1, temp2, tempDiff;
        double costTemp;
        int i_tempDiff, i_minCost;

        // Calculate row differences >> 
        for (int i=0; i< matrixSupplies; i++){
            if (rowCovered[i] == 0) {
                temp1 = INT_FAST16_MAX;
                temp2 = INT_FAST16_MAX;
                for (int j=0; j< matrixDemands; j++) {
                    // Only look at columns not covered >> 
                    if (colCovered[j] == 0) {
                        double entry = costMatrix[i*matrixDemands + j];
                        if (entry <= temp1) {
                            temp2 = temp1;
                            temp1 = entry;
                        }
                        else if (entry <= temp2) {
                            temp2 = entry;
                        }
                    }
                }
                differences[i] = temp2 - temp1;
            }
            else {
                differences[i] = INT_FAST16_MIN;
            }
        }

        // Calculate col differences >> 
        for (int j=0; j< matrixDemands; j++){
            if (colCovered[j] == 0) {
                temp1 = INT_FAST16_MAX;
                temp2 = INT_FAST16_MAX;
                // Only look at rows not covered >>
                for (int i=0; i< matrixSupplies; i++) {
                    if (rowCovered[i] == 0) {
                        double entry = costMatrix[i*matrixDemands + j];
                        if (entry <= temp1) {
                            temp2 = temp1;
                            temp1 = entry;
                        }
                        else if (entry <= temp2) {
                            temp2 = entry;
                        }
                    }
                }
                differences[matrixSupplies + j] = temp2 - temp1;
            }
            else {
                differences[matrixSupplies + j] = INT_FAST16_MIN;
            }
        }

        // Determine the maximum of differences - (Reduction)
        tempDiff = INT_FAST16_MIN;
        i_tempDiff = -1;
        for (int i=0; i < matrixSupplies + matrixDemands; i++) {
            if (differences[i] > tempDiff) {
                // tie broken by first seen
                tempDiff = differences[i];
                i_tempDiff = i;
            }
        }
        
        // Check if row or col difference and determine correspinding min cost - Another Reduction
        // Update flow accordingly and increment coveredRows/Columns and row/colCovered - Minor Update
        // Now we have Basic row and col
        // Assign flow based on availability 
        if (i_tempDiff >= matrixSupplies) {
            // This is a col difference
            i_tempDiff -= matrixSupplies;
            // In this column index find the min cost
            costTemp = INT_FAST16_MAX;
            for (int i=0; i<matrixSupplies; i++) {
                double entry = costMatrix[i*matrixDemands + i_tempDiff];
                if (entry < costTemp && rowCovered[i] == 0) {
                    costTemp = entry;
                    i_minCost = i;
                }
            }

            // std::cout<<"Col: Index-1 "<<i_tempDiff<<std::endl;
            // std::cout<<"Col: Index-2 "<<i_minCost<<std::endl;

            // std::cout<<" Res-Sup "<<residual_supply[i_minCost]<<std::endl;
            // std::cout<<" Res-Demand "<<residual_demand[i_tempDiff]<<std::endl;

            // Min cost row is i_minCost
            if (residual_demand[i_tempDiff] > residual_supply[i_minCost]){
                flows[i_minCost*matrixDemands + i_tempDiff] = residual_supply[i_minCost];
                residual_demand[i_tempDiff] -= residual_supply[i_minCost];
                rowCovered[i_minCost] = 1;
                coveredRows += 1;
            }
            else {
                flows[i_minCost*matrixDemands + i_tempDiff] = residual_demand[i_tempDiff];
                residual_supply[i_minCost] -= residual_demand[i_tempDiff];
                colCovered[i_tempDiff] = 1;
                coveredColumns += 1;
            }
        }
        else {
            // Then this is a row difference
            // In this row find the min cost
            costTemp = INT_FAST16_MAX;
            
            for (int j=0; j<matrixDemands; j++) {
                double entry = costMatrix[i_tempDiff*matrixDemands + j];
                if (entry < costTemp && colCovered[j] == 0) {
                    costTemp = entry;
                    i_minCost = j;
                }
            }
            // minCost column is i_minCost
            // std::cout<<"Row: Index-1 "<<i_tempDiff<<std::endl;
            // std::cout<<"Row: Index-2 "<<i_minCost<<std::endl;

            // std::cout<<" Res-Sup "<<residual_supply[i_tempDiff]<<std::endl;
            // std::cout<<" Res-Demand "<<residual_demand[i_minCost]<<std::endl;

            if (residual_demand[i_minCost] > residual_supply[i_tempDiff]){
                flows[i_tempDiff*matrixDemands + i_minCost] = residual_supply[i_tempDiff];
                residual_demand[i_minCost] -= residual_supply[i_tempDiff];
                rowCovered[i_tempDiff] = 1;
                coveredRows += 1;
            }
            else {
                flows[i_tempDiff*matrixDemands + i_minCost] = residual_demand[i_minCost];
                residual_supply[i_tempDiff] -= residual_demand[i_minCost];
                colCovered[i_minCost] = 1;
                coveredColumns += 1;
            }  
        }
        // printLocalDebugArray(flows, matrixSupplies, matrixDemands, "Flows");
    }
    std::cout<<"\tVogel complete!"<<std::endl;
}


__device__ int colIndxInFlat(int c, int w, int i) {
    return i*w + c;
}

__device__ int rowIndxInFlat(int r, int w, int i) {
    return r*w + i;
}

__device__ vogelDifference getMaxDiff(vogelDifference a, vogelDifference b) {
    if (a.diff > b.diff) {    
        return a;
    }
    else{
        return b;
    }
}

__global__ void find_max(vogelDifference * d_diff, vogelDifference *output, int len) {
  
  //@@ Load a segment of the input vector into shared memory
  __shared__ vogelDifference partialSegment[2*blockSize];
  vogelDifference default_diff = {.indx = -1, .diff = INT_FAST16_MIN, .ileast_1 = -1, .ileast_2 = -1};
  unsigned int start = 2*blockDim.x*blockIdx.x;
  if (threadIdx.x+start < len) {
    partialSegment[threadIdx.x] = d_diff[threadIdx.x+start];  
  }
  else {
    partialSegment[threadIdx.x] = default_diff;
  }
  
  if (start + blockDim.x + threadIdx.x < len) {
    partialSegment[blockDim.x + threadIdx.x] = d_diff[start + blockDim.x + threadIdx.x];  
  }
  
  else {
    partialSegment[blockDim.x + threadIdx.x] = default_diff;
  }
  
  //@@ Traverse the reduction tree
  for (unsigned int s = blockDim.x; s >= 1; s /= 2) {
    __syncthreads();
    if (threadIdx.x < s) {
      partialSegment[threadIdx.x] = getMaxDiff(partialSegment[threadIdx.x], partialSegment[threadIdx.x + s]);
    }
  }
  
  //@@ Write the computed sum of the block to the output vector at the
  //@@ correct index
  if (threadIdx.x == 0) {
    output[blockIdx.x] = partialSegment[0];  
  }  
}

/* 
    Find least two elements in a row/column of a matrix
    flatMatrix2D : ptr to a flattened 2D matrix
    diff: container for storing differences
    orientation: 0 for rows, 1 for columns >> bug: doesn't check for integrity here
    vectorIndex: index or row/col
    width: matrixWidth
    height: matrixHeight
*/
__global__ void find_least_two_with_indexes(double * flatMatrix2D, vogelDifference * diff, int orientation, 
    int * rowCovered, int *colCovered,
    int width, int height, int offset) {
    
    int indx = blockDim.x*blockIdx.x + threadIdx.x;
    
    int iterations = orientation == 0?width:height; 
    // If computing row differences - for indx'th row - iterate over all cols, else iterate over all rows 
    
    int max_indx = orientation == 0?height:width;
    // If computing row differences - for max indx is height, else if col diff then max index is width

    bool skip_flag = orientation == 0?rowCovered[indx]:colCovered[indx];
    // Skip flag tells if a row-col is to be ignored

    int (* fx) (int a, int b, int c) = orientation==0?&rowIndxInFlat:&colIndxInFlat;
    // fx is an indexing method for flattened arrays =>
    //  if computing row differences, indx is the row ID, iteration over columns
    //  if computing col differences, indx is the col ID, iteration over rows

    double temp1, temp2;
    int itemp1, itemp2;
    
    temp1 = INT_FAST16_MAX; // Find some C-eqvlnt of this thing
    temp2 = INT_FAST16_MAX;
    
    if (indx < max_indx && !skip_flag) {

        for (int i=0; i< iterations; i++) {
            // Only look at columns not covered >>     
            double entry = flatMatrix2D[fx(indx, width, i)];
            if (entry <= temp1) {
                temp2 = temp1;
                itemp2 = itemp1;
                temp1 = entry;
                itemp1 = i;
            }
            else if (entry <= temp2) {
                temp2 = entry;
                itemp2 = i;
            }
        }
        vogelDifference this_diff = {.indx = offset+indx, .diff = temp2-temp1, .ileast_1 = itemp1, .ileast_2 = itemp2};
        // .least_1 = temp1, .least_2 = temp2,
        diff[offset+indx] = this_diff;
    }
}


__global__ void consumeDemandSupply(int * d_supplies, int * d_demands, int * rowCovered, int * columnCovered, 
        int * f, int row_indx, int col_indx) {
        
        int sup = d_supplies[row_indx];
        int dem = d_demands[col_indx];
        if (sup >= dem) {
            // Demand point consumed the supply and got eliminated
            f[0] = dem; // Message to host
            f[1] = 1; // Message to host
            d_supplies[row_indx] = sup - dem;
            columnCovered[col_indx] = 1;
        }
        
        else {
            // Supply point consumed the demand and got eliminated
            f[0] = sup; // Message to host
            f[1] = 0; // Message to host
            d_demands[col_indx] = dem - sup;
            rowCovered[row_indx] = 1;
        }
}

/* 
Step 0 : Initialize
    
    0.1 : Allocate and copy Structures to GPU global memory
        
        Allocate : (supplies, demands, costMatrix)
        Copy : (supplies, demands, costMatrix)

        Flow Assignment for BFS are O(1) - Do them on the host

Initialize while loop >> 

Step 1 : Find Row and Columns Differences

    1.1 Launch Kernel - 1D grid (Row differences) - async
    1.2 Launch Kernel - 1D grid (Columns Differences) - async

Step 2: Reduce Max of row and col differences

Step 3: Find 

hipDeviceSynchronize();
*/

void find_vogel_bfs_parallel(int * supplies, int * demands, double * costMatrix, 
        int * flows, int matrixSupplies, int matrixDemands) {
        
        // Step 0 :
        std::cout<<"Vogel Kernel - Step 0"<<std::endl; 

        int number_of_blocks = ceil(1.0*matrixSupplies/blockSize);
        int * d_supplies, * d_demands, *rowCovered, *columnCovered, * h_f, *d_f, row_indx, col_indx;
        double * d_costMatrix;
        
        vogelDifference * d_diff, *d_diff_buffer, * h_diff_buffer, tempDiff;
        h_diff_buffer = (vogelDifference *) malloc((number_of_blocks)*sizeof(vogelDifference));
        
        hipMalloc((void **) &d_supplies, matrixSupplies*sizeof(int));
        hipMemcpy(d_supplies, supplies, matrixSupplies*sizeof(int), hipMemcpyHostToDevice);
        
        hipMalloc((void **) &d_demands, matrixDemands*sizeof(int));
        hipMemcpy(d_demands, demands, matrixDemands*sizeof(int), hipMemcpyHostToDevice);

        hipMalloc((void **) &d_costMatrix, matrixSupplies*matrixDemands*sizeof(double));
        hipMemcpy(d_costMatrix, costMatrix, matrixSupplies*matrixDemands*sizeof(double), hipMemcpyHostToDevice);

        // Booking Structures on device and host >>
        hipMalloc((void **) &d_diff, (matrixSupplies+matrixDemands)*sizeof(vogelDifference));
        hipMalloc((void **) &d_diff_buffer, (number_of_blocks)*sizeof(vogelDifference));
        
        hipMalloc((void **) &rowCovered, matrixSupplies*sizeof(int));
        hipMemset(rowCovered, 0, matrixSupplies*sizeof(int));

        hipMalloc((void **) &columnCovered, matrixDemands*sizeof(int));
        hipMemset(columnCovered, 0, matrixDemands*sizeof(int));

        h_f = (int *) malloc(sizeof(int)*2);
        hipMalloc((void **) &d_f, sizeof(int)*2);

        // Step 1 : 
        
        // Preparation for Step-1 >>
        // IDEA : use hipStream_t for all memcopies, add Error Catcher, 
        //      avoid_recompute with smarter lookups, on top of lookups

        dim3 blockD(blockSize, 1, 1);
        dim3 gridD(number_of_blocks, 1, 1);

        for (int iter=0; iter<matrixSupplies+matrixDemands-1; iter++) {

            std::cout<<"Vogel Kernel - Step 1 : Row Differences"<<std::endl; 
            find_least_two_with_indexes<<<gridD, blockD>>>(d_costMatrix, d_diff, 0, rowCovered, columnCovered, matrixDemands, matrixSupplies, 0);
            std::cout<<"Vogel Kernel - Step 1 : Col Differences"<<std::endl;
            find_least_two_with_indexes<<<gridD, blockD>>>(d_costMatrix, d_diff, 1, rowCovered, columnCovered, matrixDemands, matrixSupplies, matrixSupplies);
            
            hipDeviceSynchronize();

            // Find max of Row and Col Differences >>
            // d_diff is still on device => Directly Call the reduction kernel
            find_max<<<blockD, gridD>>>(d_diff, d_diff_buffer, matrixSupplies+ matrixDemands);
            hipDeviceSynchronize();

            hipMemcpy(h_diff_buffer, d_diff_buffer, sizeof(vogelDifference)*number_of_blocks, hipMemcpyDeviceToHost);
            // Now Reduce a small segment on device for h_diff >>
            // Recall - We're still finding max of differences but now from a very small set
            tempDiff = h_diff_buffer[0];
            for (int i=1; i<number_of_blocks; i++) {
                if (h_diff_buffer[i].diff >= tempDiff.diff) {
                    tempDiff = h_diff_buffer[i];
                }
            }
            
            // vogelDifference d = tempDiff;
            // std::cout<<"Max Diff : "<<d.diff<<std::endl;
            // std::cout<<"Max Diff : indx "<<d.indx<<std::endl;
            // std::cout<<"Max Diff : leastCost "<<d.ileast_1<<std::endl;
            
            // Identify this is a row difference or col difference
            // Now the flow assignment cell is - tempDiff.idx and tempDiff.ileast_1 

            if (tempDiff.indx > matrixSupplies) {
                // This is a col difference
                row_indx = tempDiff.ileast_1;
                col_indx = tempDiff.indx - matrixSupplies;
            }
            else {
                col_indx = tempDiff.ileast_1;
                row_indx = tempDiff.indx;
            }
            
            // Assign Supply Demand smartly with kernel - Direcly update device copy of demands and 
            // supplies to determine residual, use 1 thread for this and update flow in host
            
            // Get some way around this - do we have a device struc messaging protocol >> 
            consumeDemandSupply<<<1, 1>>>(d_supplies, d_demands, rowCovered, columnCovered, d_f, row_indx, col_indx);
            hipDeviceSynchronize();

            hipMemcpy(h_f, d_f, sizeof(int)*2, hipMemcpyHostToDevice);
            flows[row_indx*matrixSupplies+col_indx] = h_f[0];

        }
        
        printLocalDebugArray(flows, matrixSupplies, matrixDemands, "flows");

        hipFree(d_supplies);
        hipFree(d_demands);
        hipFree(d_costMatrix);
        hipFree(d_diff);
        hipFree(rowCovered);
        hipFree(columnCovered);

    }