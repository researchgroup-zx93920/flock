#include "hip/hip_runtime.h"
/*
PARALLEL KERNELS are classified as simple and special

(^_^) All simple kernels are here 

Some of the kernels in parallel simplex are specialized and slightly complicated,
they are stored in a separate module for cleanliness, follow the
usage in uv_model_parallel.cu (aka parent) file. 

FYI Directly reviewing this file wouldn't make sense. If you see a kernel in parent
You'll either find it here or there's a comment that would take you to the 
appropriate place
*/

#include <iostream>

#include "parallel_structs.h"

#ifndef KERNELS
#define KERNELS

// ##################################################
// PREPROCESS and POSTPROCESS  >>
// ##################################################

/*
Kernel to convert float cost matrix to the MatrixCell objects
*/
__global__ void createCostMatrix(MatrixCell *d_costMtx, float * d_costs_ptr, int n_supplies, int n_demands)
{

    int d = blockIdx.x * blockDim.x + threadIdx.x;
    int s = blockIdx.y * blockDim.y + threadIdx.y;

    if (s < n_supplies && d < n_demands)
    {
        int id = s * n_demands + d;
        MatrixCell _c = {.row = s, .col = d, .cost = d_costs_ptr[id]};
        d_costMtx[id] = _c;
    }
}

/*
Generate a tree on the global memory using the initial set of feasible flows
*/
__global__ void create_initial_tree(flowInformation * d_flows_ptr, int * d_adjMtx_ptr, float * d_flowMtx_ptr,
    int numSupplies, int numDemands)
{
    
    int V = numSupplies+numDemands;
    int gid = blockIdx.x*blockDim.x + threadIdx.x;

    if (gid < V - 1) {
    
        flowInformation _this_flow = d_flows_ptr[gid];
        int row = _this_flow.source;
        int column =  _this_flow.destination;
        int idx = TREE_LOOKUP(row, numSupplies+column, V); // Index in adjacency matrix
        float _qty = 1.0*_this_flow.qty;
        if (_qty==0){
            // Handling degeneracy - Flow purturbation
            _qty=epsilon;
        }
        d_flowMtx_ptr[gid] = _qty;
        d_adjMtx_ptr[idx] = gid+1;
    }
}

/*
Reverse operation of generating a tree from the feasible flows - unordered allocation
*/
__global__ void retrieve_final_tree(flowInformation * d_flows_ptr, int * d_adjMtx_ptr, float * d_flowMtx_ptr,
        int numSupplies, int numDemands) 
{

    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;
    int V = numSupplies+numDemands;
    
    // Upper triangle scope of adj matrix
    if (col_indx < V && col_indx >= numSupplies && row_indx < numSupplies) {
        
        // Check if this is a flow edge - 
        int gid = TREE_LOOKUP(row_indx, col_indx, V);
        int flow_id = d_adjMtx_ptr[gid];
        if (flow_id > 0) {

            flowInformation _this_flow;
            _this_flow.qty = round(d_flowMtx_ptr[flow_id - 1]);
            _this_flow.source = row_indx;
            _this_flow.destination = col_indx - numSupplies;
            d_flows_ptr[flow_id - 1] = _this_flow;

        }
    }
}


/* 
Transfer flows on device and prepare an adjacency and flow matrix using the flows from IBFS
In case of sequencial pivoting - one would need a copy of adjMatrix on the host to traverse the graph
*/
__host__ void create_IBF_tree_on_host_device(Graph &graph, flowInformation * feasible_flows, 
    int numSupplies, int numDemands) {

    int V = numSupplies+numDemands;
    int _utm_entries = (V*(V+1))/2; // Number of entries in upper triangular matrix 

    gpuErrchk(hipMalloc((void **) &graph.d_adjMtx_ptr, sizeof(int)*_utm_entries)); 
    thrust::fill(thrust::device, graph.d_adjMtx_ptr, (graph.d_adjMtx_ptr) + _utm_entries, 0);

    gpuErrchk(hipMalloc((void **) &graph.d_flowMtx_ptr, sizeof(float)*(V-1)));
    thrust::fill(thrust::device, graph.d_flowMtx_ptr, (graph.d_flowMtx_ptr) + (V-1), 0);

    // Make a replica of feasible flows on device
    flowInformation * d_flows_ptr;
    gpuErrchk(hipMalloc((void **) &d_flows_ptr, sizeof(flowInformation)*(V-1)));
    gpuErrchk(hipMemcpy(d_flows_ptr, feasible_flows, sizeof(flowInformation)*(V-1), hipMemcpyHostToDevice));

    // Small kernel to parallely create a tree using the flows
    create_initial_tree <<< ceil(1.0*(V-1)/blockSize), blockSize >>> (d_flows_ptr, graph.d_adjMtx_ptr, graph.d_flowMtx_ptr, numSupplies, numDemands);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
    // Now device_flows are useless; 
    // All information about graph is now contained within d_adjMatrix, d_flowMatrix on device =>
    gpuErrchk(hipFree(d_flows_ptr));
    
    // Make a copy on host >>
    graph.h_adjMtx_ptr = (int *) malloc(sizeof(int)*(_utm_entries));
    gpuErrchk(hipMemcpy(graph.h_adjMtx_ptr, graph.d_adjMtx_ptr, sizeof(int)*(_utm_entries), hipMemcpyDeviceToHost));
    graph.h_flowMtx_ptr = (float *) malloc(sizeof(float)*(V-1));
    gpuErrchk(hipMemcpy(graph.h_flowMtx_ptr, graph.d_flowMtx_ptr, sizeof(float)*(V-1), hipMemcpyDeviceToHost));

    // Iterations would also work with a adjacency list
    gpuErrchk(hipMalloc((void **) &graph.d_vertex_start, sizeof(int)*(V)));
    gpuErrchk(hipMalloc((void **) &graph.d_vertex_degree, sizeof(int)*(V+1)));
    gpuErrchk(hipMalloc((void **) &graph.d_adjVertices, sizeof(int)*2*(V-1)));
    graph.h_vertex_start = (int *) malloc(sizeof(int)*V);
    graph.h_vertex_degree = (int *) malloc(sizeof(int)*V);
    graph.h_adjVertices = (int *) malloc(sizeof(int)*2*(V-1));

}

/*
Given a feasible tree on device, load a feasible solution to transportation problem on the host
*/
__host__ void retrieve_solution_on_current_tree(flowInformation * feasible_flows, Graph &graph,
    int &active_flows, int numSupplies, int numDemands)
{
    
    // Recreate device flows using the current adjMatrix
    flowInformation default_flow;
    default_flow.qty = 0;

    flowInformation * d_flows_ptr;
    gpuErrchk(hipMalloc((void **) &d_flows_ptr, sizeof(flowInformation)*(numSupplies*numDemands)));
    thrust::fill(thrust::device, d_flows_ptr, d_flows_ptr + (numSupplies*numDemands), default_flow);

    dim3 __blockDim(blockSize, blockSize, 1);
    int grid_size = ceil(1.0*(numSupplies+numDemands)/blockSize); // VxV threads
    dim3 __gridDim(grid_size, grid_size, 1);
    retrieve_final_tree <<< __gridDim, __blockDim >>> (d_flows_ptr, graph.d_adjMtx_ptr, graph.d_flowMtx_ptr, numSupplies, numDemands);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
    // Copy the (flows > 0) back on the host >>
    auto flow_end = thrust::remove_if(thrust::device,
        d_flows_ptr, d_flows_ptr + (numSupplies*numDemands), is_zero());
    int flow_count = flow_end - d_flows_ptr;
    // Update active flows in result 
    active_flows = flow_count;
    gpuErrchk(hipMemcpy(feasible_flows, d_flows_ptr, (flow_count)*sizeof(flowInformation), hipMemcpyDeviceToHost));

}

/* Clear up the memory occupied by graph on host on device */
__host__ void close_solver(Graph &graph)
{

    gpuErrchk(hipFree(graph.d_vertex_start));
    gpuErrchk(hipFree(graph.d_vertex_degree));
    gpuErrchk(hipFree(graph.d_adjVertices));
    gpuErrchk(hipFree(graph.d_adjMtx_ptr));
    gpuErrchk(hipFree(graph.d_flowMtx_ptr));
    
    free(graph.h_vertex_start);
    free(graph.h_vertex_degree);
    free(graph.h_adjVertices);
    free(graph.h_adjMtx_ptr);
    free(graph.h_flowMtx_ptr);

}

__global__ void determine_length(int * length, int * d_adjMtx_ptr, int V) {
        int L = 0;
        int i = blockIdx.x *blockDim.x + threadIdx.x;
        // No data re-use (this is a straight fwd kernel)
        if (i < V) 
        {    
                for (int j=0; j<V; j++) {
                        int idx = TREE_LOOKUP(i, j, V);
                        if (d_adjMtx_ptr[idx] > 0) {
                                L++;
                        }
                }
                length[i+1] = L;
                length[0] = 0;
        }
}

__global__ void fill_Ea(int * start, int * Ea, int * d_adjMtx_ptr, int V, int numSupplies) {
        int i = blockIdx.x*blockDim.x + threadIdx.x;
        int offset = start[i];
        int L = 0;
        if (i < V) {
                for (int j=0; j<V; j++) {
                        int idx = TREE_LOOKUP(i, j, V);
                        if (d_adjMtx_ptr[idx] > 0) {
                                Ea[offset + L] = j;
                                L++;
                        }
                }
        }
}

/*
DEBUG UTILITY : VIEW ADJACENCY LIST STRCTURE 
*/
__host__ void __debug_view_adjList(int * start, int * length, int * Ea, int V) 
{        
        int * h_length = (int *) malloc(sizeof(int)*V);
        int * h_start = (int *) malloc(sizeof(int)*V);
        int * h_Ea = (int *) malloc(sizeof(int)*2*(V-1));

        hipMemcpy(h_length, length, sizeof(int)*V, hipMemcpyDeviceToHost);
        hipMemcpy(h_start, start, sizeof(int)*V, hipMemcpyDeviceToHost);
        hipMemcpy(h_Ea, Ea, sizeof(int)*2*(V-1), hipMemcpyDeviceToHost);

        std::cout<<"Str = [ ";
        for (int i =0; i < V; i++){
                std::cout<<h_start[i]<<", ";
        }
        std::cout<<"]"<<std::endl;
        std::cout<<"Len = [ ";
        for (int i =0; i < V; i++){
                std::cout<<h_length[i]<<", ";
        }
        std::cout<<"]"<<std::endl;
        std::cout<<"Ea = [ ";
        for (int i =0; i < 2*(V-1); i++){
                std::cout<<h_Ea[i]<<", ";
        }
        std::cout<<"]"<<std::endl;
        
        free(h_length);
        free(h_Ea);
        free(h_start);
        // *************** END OF DEBUG UTILITY ***************

}

__host__ void make_adjacency_list(Graph &graph, int numSupplies, int numDemands) {

        // Kernel Dimensions >> 
        dim3 __blockDim(blockSize, 1, 1); 
        dim3 __gridDim(ceil(1.0*graph.V/blockSize), 1, 1);

        determine_length <<< __gridDim, __blockDim >>> (graph.d_vertex_degree, graph.d_adjMtx_ptr, graph.V);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        
        thrust::inclusive_scan(thrust::device, graph.d_vertex_degree, graph.d_vertex_degree + graph.V, graph.d_vertex_start);
        
        fill_Ea <<< __gridDim, __blockDim >>> (graph.d_vertex_start, graph.d_adjVertices, graph.d_adjMtx_ptr, graph.V, numSupplies);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // Entries in Matrix >>
        // int _utm_entries = (V*(V+1))/2; // Number of entries in upper triangular matrix
        // auto result_end = thrust::copy_if(thrust::device, d_adjMtx_ptr, d_adjMtx_ptr + _utm_entries, 
        //                     Ea, is_nonzero_entry()); // --> need col indices of non-zeros

        // DEBUG :: 
        // __debug_view_adjList(start, &length[1], Ea, V);
        // exit(0);
}


// ##################################################
// SOLVING DUAL >>
// ##################################################

/*
APPROACH 1 :
Kernels concerned with solving the UV System using a BFS Traversal Approach
*/

__global__ void copy_row_shadow_prices(Variable * U_vars, float * u_vars_ptr, int numSupplies) 
{    
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    if (gid < numSupplies) {
        u_vars_ptr[gid] = U_vars[gid].value;
    }
}

__global__ void copy_col_shadow_prices(Variable * V_vars, float * v_vars_ptr, int numDemands) 
{
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    if (gid < numDemands) {
        v_vars_ptr[gid] = V_vars[gid].value;
    }
}

__global__ void initialize_U_vars(Variable * U_vars, int numSupplies) {
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    Variable default_var;
    if (gid < numSupplies) {
        U_vars[gid] = default_var;
    }
}

__global__ void initialize_V_vars(Variable * V_vars, int numDemands) {
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    Variable default_var;
    if (gid < numDemands) {
        V_vars[gid] = default_var;
    }
}

/*
Breadth First Traversal on UV
*/
__global__ void assign_next(int * d_adjMtx_ptr, float * d_costs_ptr, 
    Variable *u_vars, Variable *v_vars, int numSupplies, int numDemands) {
    
    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;
    int V = numSupplies + numDemands;

    // Within the scope of the adj matrix
    if (row_indx < numSupplies && col_indx < numDemands) {
        // Check if these are adjacent - (checks in upper triangular matrix, because row < adj-col-index)
        int indx = TREE_LOOKUP(row_indx, col_indx + numSupplies, V); // Adjusted destination vertex ID
        if (d_adjMtx_ptr[indx] > 0) {

            Variable u_i = u_vars[row_indx];
            Variable v_j = v_vars[col_indx];
            
            // Check if any of the u or v has not been assigned and adjacent is assigned - then assign it
            if (u_vars[row_indx].assigned && (!v_vars[col_indx].assigned)) {
                // In this case >> v_j = c_ij - u_i
                Variable var;
                var = d_costs_ptr[row_indx*numDemands+col_indx] - u_vars[row_indx].value;
                // var.assigned = true;
                v_vars[col_indx] = var;
            }
            else if ((!u_vars[row_indx].assigned) && v_vars[col_indx].assigned) {
                // In this case >> u_j = c_ij - v_j
                Variable var;
                var = d_costs_ptr[row_indx*numDemands+col_indx] -  v_vars[col_indx].value;
                // var.assigned = true;
                u_vars[row_indx] = var;
            }
        }
    }
}

// Credits: https://github.com/siddharths2710/cuda_bfs/blob/master/cuda_bfs/kernel.cu
__global__ void CUDA_BFS_KERNEL(int * start, int * length, int *Ea, bool * Fa, bool * Xa, 
        float * variables, float * d_costs_ptr, bool * done, int numSupplies, int numDemands, int V)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > V)
		*done = false;

	if (Fa[id] == true && Xa[id] == false)
	{
		// printf("%d ", id); //This printf gives the order of vertices in BFS	
		Fa[id] = false;
		Xa[id] = true;
		__syncthreads(); 
		int k = 0;
		int start_ptr = start[id];
		int end_ptr = start_ptr + length[id];
		for (int i = start_ptr; i < end_ptr; i++) 
		{
			int nid = Ea[i];
			if (Xa[nid] == false)
			{       
                int row_indx = min(nid, id);
                int col_indx = max(nid, id) - numSupplies;
				variables[nid] = d_costs_ptr[row_indx*numDemands+col_indx] - variables[id];
				Fa[nid] = true;
				*done = false;
			}
		}
	}
}

/*
APPROACH 2:
Kernels concerned with solving the UV System using a using a matrix solver
*/

// Custom Fill kernel for csr row pointers
__global__ void fill_csr_offset (int * d_csr_offsets, int length) {
        
        int idx = blockIdx.x*blockDim.x + threadIdx.x;
        if (idx < length) {
                if (idx == 0) {
                        d_csr_offsets[idx] = 0;
                }
                else {
                        d_csr_offsets[idx] = 2*idx - 1; 
                }
        }
}

/*
Create a dense linear system in parallel by looking at current feasible tree 
*/
__global__ void initialize_dense_u_v_system(float * d_A, float * d_b, int * d_adjMtx_ptr, 
    float * d_costs_ptr, int numSupplies, int numDemands) {
        
    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;
    int V = numSupplies + numDemands;

    if (row_indx < numSupplies && col_indx < numDemands) {
        int indx = TREE_LOOKUP(row_indx, col_indx + numSupplies, V); // Adjusted destination vertex ID
        int flow_indx = d_adjMtx_ptr[indx];
        if (flow_indx > 0) {
            // This is a flow - flow_indx = row_number, u = row_number, v = col_number
            d_A[flow_indx * V + row_indx] = 1;
            d_A[flow_indx * V + numSupplies + col_indx] = 1;
            d_b[flow_indx] = d_costs_ptr[row_indx*numDemands + col_indx];
        }
    }
}

/*
Create a sparse linear system in parallel by looking at current feasible tree 
*/
__global__ void initialize_sparse_u_v_system(int * d_csr_columns, float * d_b, int * d_adjMtx_ptr, 
    float * d_costs_ptr, int numSupplies, int numDemands) {
        
    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;
    int V = numSupplies + numDemands;

    if (row_indx < numSupplies && col_indx < numDemands) {
        int indx = TREE_LOOKUP(row_indx, col_indx + numSupplies, V); // Adjusted destination vertex ID
        int flow_indx = d_adjMtx_ptr[indx];
        if (flow_indx > 0) {
            // This is a flow - flow_indx = row_number, u = row_number, v = col_number
            d_csr_columns[2*flow_indx-1] = row_indx;
            d_csr_columns[2*flow_indx] = numSupplies + col_indx;
            d_b[flow_indx] = d_costs_ptr[row_indx*numDemands + col_indx];
        }
    }
}

/*
Load the solution of system to the appropriate place
*/
__global__ void retrieve_uv_solution(float * d_x, float * u_vars_ptr, float * v_vars_ptr, int numSupplies, int numDemands) 
{
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    int V = numSupplies + numDemands;
    if (gid < V) {
        if (gid < numSupplies) {
            u_vars_ptr[gid] = d_x[gid];
        } 
        else {
            v_vars_ptr[gid - numSupplies] = d_x[gid];
        }
    }
}

// ##################################################
// COMPUTING REDUCED COSTS >>
// ##################################################


/*
Kernel to compute Reduced Costs in the transportation table
*/
__global__ void computeReducedCosts(float * u_vars_ptr, float * v_vars_ptr, float * d_costs_ptr, float * d_reducedCosts_ptr, 
    int numSupplies, int numDemands)
{
        int row_indx = blockIdx.y*blockDim.y + threadIdx.y;
        int col_indx = blockIdx.x*blockDim.x + threadIdx.x;

        if (row_indx < numSupplies && col_indx < numDemands) {
            // r =  C_ij - (u_i + v_j);
            float r = d_costs_ptr[row_indx*numDemands+col_indx] - u_vars_ptr[row_indx] - v_vars_ptr[col_indx];
            d_reducedCosts_ptr[row_indx*numDemands+col_indx] = r;
        }
}


/* Optimized function for the above kernel to compute reduced costs */
__global__ void computeReducedCosts(float * u_vars_ptr, float * v_vars_ptr, float * d_costs_ptr, MatrixCell * d_reducedCosts_ptr, 
    int numSupplies, int numDemands)
{

        __shared__ float U[blockSize];
        __shared__ float V[blockSize];
        
        int row_indx = blockIdx.y*blockDim.y + threadIdx.y;
        int col_indx = blockIdx.x*blockDim.x + threadIdx.x;

        if (row_indx < numSupplies && col_indx < numDemands) {
            // r =  C_ij - (u_i + v_j);
            U[threadIdx.y] =  u_vars_ptr[row_indx];
            V[threadIdx.x] = v_vars_ptr[col_indx];
            __syncthreads();
            float r = d_costs_ptr[row_indx*numDemands+col_indx] - U[threadIdx.y] - V[threadIdx.x];
            MatrixCell _m = {.row = row_indx, .col = col_indx, .cost = r};
            d_reducedCosts_ptr[row_indx*numDemands+col_indx] = _m;
        }
}


/*
 Naive CUDA kernel implementation of Floyd Wharshall
 check if path from vertex x -> y will be shorter using a via. vertex k 
 for all vertices in graph:
    check if (x -> k -> y) < (x -> k)

*/
__global__ void _naive_floyd_warshall_kernel(const int k, const int V, int * d_adjMtx, int * path) {
    
    int col_indx = blockDim.x * blockIdx.x + threadIdx.x;
    int row_indx = blockDim.y * blockIdx.y + threadIdx.y;

    if (col_indx < V && row_indx < V) {
        int indexYX = row_indx * V + col_indx;
        int indexKX = k * V + col_indx;
        int indexYK = row_indx*V + k;

        int newPath = d_adjMtx[indexYK] + d_adjMtx[indexKX];
        int oldPath = d_adjMtx[indexYX];
        if (oldPath > newPath) {
            d_adjMtx[indexYX] = newPath;
            path[indexYX] = path[indexKX];
        }
    }
}


__device__ int my_signum(const int x) {
    return (((x) > 0)?(1):(INT16_MAX));
}

/* Set initial values in adj Matrix and path matrix */
__global__ void fill_adjMtx(int * d_adjMtx_transform, int * d_adjMtx_actual, int * d_pathMtx, int V) {
    
    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;

    if (row_indx < V && col_indx < V) {

        int dist = my_signum(d_adjMtx_actual[TREE_LOOKUP(row_indx, col_indx, V)]);
        
        d_adjMtx_transform[row_indx*V + col_indx] = dist; // Setting B - A
        d_adjMtx_transform[col_indx*V + row_indx] = dist; // setting A - B 
        d_adjMtx_transform[row_indx*V + row_indx]  = 0; // setting the diagonal entries 0 
        
        if (dist == 1) {
            d_pathMtx[row_indx*V + col_indx] = row_indx;
            d_pathMtx[col_indx*V + row_indx] = col_indx;
        }        
    }
}


/*
Recursively explore pathMtx and store all the discovered cycles in the expanded form 
Can spped this up using a 3D grid? 
*/
__global__ void expand_all_cycles(int * d_adjMtx_transform, int * d_pathMtx, int * d_pivot_cycles, int diameter, int numSupplies, int numDemands) {

    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;  // demand point
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;  // supply point 
    int offset_1 = diameter*(row_indx*numDemands + col_indx); // offset for cycle store
    int V = numSupplies + numDemands;
    // Entering Edge is from supply - i to demand - j we discover a cycle by finding a shortest path from [j] -> [i]
    int offset_2 = row_indx*V + (col_indx+numSupplies); // offset for path and adjMtx 

    if (row_indx < numSupplies && col_indx < numDemands) {

        int depth = d_adjMtx_transform[offset_2] + 1;
        int current_vtx = col_indx + numSupplies; // backtrack from - j
        int target_vtx = row_indx; // reach - i 

        d_pivot_cycles[offset_1] = target_vtx;
        
        int d = 1;

        while (d < depth) {
            d_pivot_cycles[offset_1 + d] = current_vtx;
            current_vtx = d_pathMtx[target_vtx*V + current_vtx];
            d++;
        }

        d_pivot_cycles[offset_1+depth] = target_vtx;
    }
}

/*
Check if a cycle is still feasible if any of the edges of the cycle located at min_indx has been used
If it is infeasible then set the reduced cost of this cell as non-negative to deactivate pivot here
*/
__global__ void check_pivot_feasibility(int * d_adjMtx_transform, int * d_pivot_cycles, 
    MatrixCell * d_reducedCosts_ptr, int min_r_index, int diameter, int numSupplies, int numDemands) {

    // Nomenclature : 
    // this_cycle - cycle located on this thread
    // earlier_cycle - cycle located on the min_r_index 
    // Context : Edges in earlier cycle will be used by pivot (because min_reduced cost)
    //      Then if there are any common edges between this_cycle and earlier_cycle
    //      then this_cycle conflicts with earlier_cycle and it has to die
    //      Set reduced cost = non_negative for this cycle

    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;  // demand point
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;  // supply point 
    int offset_1 = diameter*(row_indx*numDemands + col_indx); // offset for this_cycle store
    int V = numSupplies + numDemands;
    int offset_2 = row_indx*V + (col_indx+numSupplies); // offset for path and adjMtx 
    
    // Load the earlier cycle and it's depth - enough memory is available through diameter
    extern __shared__ int earlier_cycle[];

    int _pivot_row =  min_r_index/numDemands;
    int _pivot_col = min_r_index - (_pivot_row*numDemands);
    int earlier_cycle_depth = d_adjMtx_transform[_pivot_row*V + (_pivot_col+numSupplies)] + 1; // depth of earlier_cycle
    // Set reduced cost of earlier_cycle nonNegative >>
    MatrixCell _nonNegative = {.row = _pivot_row, .col = _pivot_col, .cost = epsilon};
    d_reducedCosts_ptr[_pivot_row*numDemands + _pivot_col] = _nonNegative;
    // all threads in block load the same value

    // Load the earlier cycle in parallel 
    int _stride = blockDim.x*blockDim.y; 
    int _local_index = threadIdx.y*blockDim.x + threadIdx.x;
    int offset_3 = diameter*(_pivot_row*numSupplies + _pivot_col); // offset for earlier_cycle_store
    while (_local_index < earlier_cycle_depth + 1) {
        earlier_cycle[_local_index] = d_pivot_cycles[offset_3 + _local_index];
        _local_index = _local_index + _stride;
    }
    __syncthreads();

    // Now earlier cycle is available in shared memory - traverse this_cycle and check for common edges
    if (row_indx < numSupplies && col_indx < numDemands) {

        int vtx_i; // i^th vertex in this_cycle 
        int vtx_j; // j^th vertex in earlier_cycle
        int this_cycle_depth = d_adjMtx_transform[offset_2] + 1;
        int vtx_i1, vtx_j1; // i+1^th and j+1^th vertices in corresponding columns
        int edge_i, edge_j; // edge id of edges (i,i+1) and (j,j+1) - edgeid is the index of edge in supply x demand matrix

        for (int i = 1; i < this_cycle_depth + 1; i++) {
            
            vtx_i = d_pivot_cycles[offset_1+i] - numSupplies*(i%2);
            vtx_i1 = d_pivot_cycles[offset_1+i+1] - numSupplies*((i+1)%2);
            edge_i = (vtx_i*numDemands + vtx_i1)*((i+1)%2) + (vtx_i1*numDemands + vtx_i)*(i%2);

            for (int j = 1; j < earlier_cycle_depth + 1; j++) {
                
                vtx_j = earlier_cycle[j] - numSupplies*(j%2);
                vtx_j1 = earlier_cycle[j+1] - numSupplies*((j+1)%2);
                edge_j = (vtx_j*numDemands + vtx_j1)*((j+1)%2) + (vtx_j1*numDemands + vtx_j)*(j%2);
                // Note that we're looking up undirected edges, so we compare unique identifiers of both
                
                // Whenever the cycles intersect
                if (edge_i == edge_j) {
                    // set reduced cost of this_cycles as non-negative
                    MatrixCell _nonNegative = {.row = row_indx, .col = col_indx, .cost = epsilon};
                    d_reducedCosts_ptr[row_indx*numDemands + col_indx] = _nonNegative;
                    return;
                }
            }
        }
    }
}


/*
Check if a cycle is still feasible if any of the edges of the cycle located at min_indx has been used
If it is infeasible then set the reduced cost of this cell as non-negative to deactivate pivot here
*/
__global__ void check_pivot_feasibility(int * d_adjMtx_transform, int * d_pivot_cycles, 
    float * d_opportunity_costs, int min_r_index, int diameter, int numSupplies, int numDemands) {

    // Nomenclature : 
    // this_cycle - cycle located on this thread
    // earlier_cycle - cycle located on the min_r_index 
    // Context : Edges in earlier cycle will be used by pivot (because min_reduced cost)
    //      Then if there are any common edges between this_cycle and earlier_cycle
    //      then this_cycle conflicts with earlier_cycle and it has to die
    //      Set reduced cost = non_negative for this cycle

    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;  // demand point
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;  // supply point 
    int offset_1 = diameter*(row_indx*numDemands + col_indx); // offset for this_cycle store
    int V = numSupplies + numDemands;
    int offset_2 = row_indx*V + (col_indx+numSupplies); // offset for path and adjMtx 
    
    // Load the earlier cycle and it's depth - enough memory is available through diameter
    extern __shared__ int earlier_cycle[];

    int _pivot_row =  min_r_index/numDemands;
    int _pivot_col = min_r_index - (_pivot_row*numDemands);
    int earlier_cycle_depth = d_adjMtx_transform[_pivot_row*V + (_pivot_col+numSupplies)] + 1; // depth of earlier_cycle
    
    // Set opportunity cost of earlier_cycle nonNegative >>
    d_opportunity_costs[_pivot_row*numDemands + _pivot_col] = epsilon;
    // all threads in block load the same value

    // Load the earlier cycle in parallel 
    int _stride = blockDim.x*blockDim.y; 
    int _local_index = threadIdx.y*blockDim.x + threadIdx.x;
    int offset_3 = diameter*(_pivot_row*numSupplies + _pivot_col); // offset for earlier_cycle_store
    while (_local_index < earlier_cycle_depth + 1) {
        earlier_cycle[_local_index] = d_pivot_cycles[offset_3 + _local_index];
        _local_index = _local_index + _stride;
    }
    __syncthreads();

    // Now earlier cycle is available in shared memory - traverse this_cycle and check for common edges
    if (row_indx < numSupplies && col_indx < numDemands) {

        int vtx_i; // i^th vertex in this_cycle 
        int vtx_j; // j^th vertex in earlier_cycle
        int this_cycle_depth = d_adjMtx_transform[offset_2] + 1;
        int vtx_i1, vtx_j1; // i+1^th and j+1^th vertices in corresponding columns
        int edge_i, edge_j; // edge id of edges (i,i+1) and (j,j+1) - edgeid is the index of edge in supply x demand matrix

        for (int i = 1; i < this_cycle_depth + 1; i++) {
            
            vtx_i = d_pivot_cycles[offset_1+i] - numSupplies*(i%2);
            vtx_i1 = d_pivot_cycles[offset_1+i+1] - numSupplies*((i+1)%2);
            edge_i = (vtx_i*numDemands + vtx_i1)*((i+1)%2) + (vtx_i1*numDemands + vtx_i)*(i%2);

            for (int j = 1; j < earlier_cycle_depth + 1; j++) {
                
                vtx_j = earlier_cycle[j] - numSupplies*(j%2);
                vtx_j1 = earlier_cycle[j+1] - numSupplies*((j+1)%2);
                edge_j = (vtx_j*numDemands + vtx_j1)*((j+1)%2) + (vtx_j1*numDemands + vtx_j)*(j%2);
                // Note that we're looking up undirected edges, so we compare unique identifiers of both
                
                // Whenever the cycles intersect
                if (edge_i == edge_j) {
                    // set opportunity cost of this_cycle as non-negative
                    d_opportunity_costs[row_indx*numDemands + col_indx] = epsilon;
                    return;
                }
            }
        }
    }
}



/* 
Compute Oppotunity costs and delta -
Logic : For each edge retreive cost and flow - 
        Track their sum and minimum as you traverse along
        Store the final value in appropriate array
*/
__global__ void compute_opportunity_cost_and_delta(int * d_adjMtx_ptr, float * d_flowMtx_ptr, float * d_costs_ptr, 
    int * d_adjMtx_transform, int * d_pivot_cycles, float * d_opportunity_costs, 
    int diameter, int numSupplies, int numDemands) {

    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;  // demand point
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;  // supply point 
    int offset_1 = diameter*(row_indx*numDemands + col_indx); // offset for cycle store
    int V = numSupplies + numDemands;
    int offset_2 = row_indx*V + (col_indx+numSupplies); // offset for adjMtx_transformed

    if (row_indx < numSupplies && col_indx < numDemands) {

        int id_graph, id_costs, _from = -1, _to = -1;
        int this_cycle_depth = d_adjMtx_transform[offset_2] + 1;
        float _flow, min_flow = INT_MAX, opportunity_cost = 0.0f;
        
        for (int i = 0; i < this_cycle_depth + 1; i++) {

            _from = d_pivot_cycles[offset_1+i];
            _to = d_pivot_cycles[offset_1+i+1];
            id_costs = (_from*numDemands + _to)*((i+1)%2) + (_from*numDemands + _to)*(i%2);
            
            // ########### PART - 1 | Finding the opportunity costs >>
            // Add evens and substract odds
            opportunity_cost = opportunity_cost + pow(-1, i%2)*d_costs_ptr[id_costs];

            // ########### PART - 2 | Finding the minimum flow >>
            // Traverse the loop find the minimum flow that could be increased
            // on the incoming edge - (Look for minimum of flows on odd indexed edges)
            if (i%2==1) 
            {
                id_graph = d_adjMtx_ptr[TREE_LOOKUP(_from, _to, V)] - 1;
                _flow = d_flowMtx_ptr[id_graph];
                
                if (_flow < min_flow) 
                {
                    min_flow = _flow;
                }
            }
        }

        // Load the values the in books for next kernel
        d_opportunity_costs[row_indx*numDemands + col_indx] = opportunity_cost*min_flow;
    }
}


/* 
Compute Oppotunity costs and delta -
Logic : For each edge retreive costs - 
        Track their sum as you traverse along
        Store the final value in appropriate array
*/
__global__ void compute_opportunity_cost(int * d_adjMtx_ptr, float * d_flowMtx_ptr, float * d_costs_ptr, 
    int * d_adjMtx_transform, int * d_pivot_cycles, float * d_opportunity_costs, 
    int diameter, int numSupplies, int numDemands) {

    int col_indx = blockIdx.x*blockDim.x + threadIdx.x;  // demand point
    int row_indx = blockIdx.y*blockDim.y + threadIdx.y;  // supply point 
    int offset_1 = diameter*(row_indx*numDemands + col_indx); // offset for cycle store
    int V = numSupplies + numDemands;
    int offset_2 = row_indx*V + (col_indx+numSupplies); // offset for adjMtx_transformed

    if (row_indx < numSupplies && col_indx < numDemands) {

        int id_costs, _from = -1, _to = -1;
        int this_cycle_depth = d_adjMtx_transform[offset_2] + 1;
        float opportunity_cost = 0.0f;
        
        for (int i = 0; i < this_cycle_depth + 1; i++) {

            _from = d_pivot_cycles[offset_1+i];
            _to = d_pivot_cycles[offset_1+i+1];
            id_costs = (_from*numDemands + _to)*((i+1)%2) + (_from*numDemands + _to)*(i%2);
            
            // Finding the opprotunity costs >>
            // Traverse the loop find the minimum flow that could be increased
            // on the incoming edge - (Look for minimum of flows on odd indexed edges)
            opportunity_cost = opportunity_cost + pow(-1, i%2)*d_costs_ptr[id_costs];
        }

        // Load the values the in books for next kernel
        d_opportunity_costs[row_indx*numDemands + col_indx] = opportunity_cost;  
    }
}

#endif