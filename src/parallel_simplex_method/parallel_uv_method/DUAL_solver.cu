#include "hip/hip_runtime.h"
#include "DUAL_solver.h"

namespace UV_METHOD {

__host__ void dualMalloc(DualHandler &dual, int numSupplies, int numDemands) {
    
    int V = numSupplies + numDemands;
    // Create and Initialize u and v variables 
    //  empty u and v equations using the Variable Data Type >>
    dual.h_visited = (bool *) malloc(sizeof(bool)*V);
    dual.h_variables = (float *) malloc(sizeof(float)*V);
    
    if (REDUCED_COST_MODE == "parallel") {
        
        gpuErrchk(hipMalloc((void **) &dual.u_vars_ptr, sizeof(float)*numSupplies));
        gpuErrchk(hipMalloc((void **) &dual.v_vars_ptr, sizeof(float)*numDemands));

    }
}

__host__ void dualFree(DualHandler &dual) {
     
     if (REDUCED_COST_MODE == "parallel") {

        gpuErrchk(hipFree(dual.u_vars_ptr));
        gpuErrchk(hipFree(dual.v_vars_ptr));

     }
        
        free(dual.h_visited);
        free(dual.h_variables);

}

__host__ void find_dual_using_host_bfs(DualHandler &dual,  Graph &graph, float * h_costs_ptr, 
        int numSupplies, int numDemands) {

        // Copy Adjacency list on host >> assuming Tranformation already occured at the start of pivoting 
        gpuErrchk(hipMemcpy(graph.h_vertex_degree, &graph.d_vertex_degree[1], sizeof(int)*graph.V, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(graph.h_vertex_start, graph.d_vertex_start, sizeof(int)*graph.V, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(graph.h_adjVertices, graph.d_adjVertices, sizeof(int)*2*(graph.V-1), hipMemcpyDeviceToHost));

        thrust::fill(thrust::host, dual.h_visited, dual.h_visited + graph.V, false);
        thrust::fill(thrust::host, dual.h_variables, dual.h_variables + graph.V, 0.0f);

        // Initialize >>
        std::queue<int> assigned_parents;
        dual.h_visited[0] = true;
        // The value of u0 is already zero as initialized
        assigned_parents.push(0);

        // Perform a BFS on Host (trickle down) >> 
        int parent, child, row, col;
        
        int * h_start = graph.h_vertex_start;
        int * h_length = graph.h_vertex_degree;
        int * h_Ea = graph.h_adjVertices;


        while (!assigned_parents.empty()) {
                parent = assigned_parents.front();
                for (int i = h_start[parent]; i < h_start[parent] + h_length[parent]; i++) {
                        child = h_Ea[i];
                        if (!dual.h_visited[child]) {
                                dual.h_visited[child] = true;
                                row = min(parent, child);
                                col = max(parent, child) - numSupplies;
                                dual.h_variables[child] = h_costs_ptr[row*numDemands + col] - dual.h_variables[parent];
                                assigned_parents.push(child);
                        }
                }
                assigned_parents.pop();
        }

        // Transfer back to GPU >> 
        if (REDUCED_COST_MODE == "parallel") {

                gpuErrchk(hipMemcpy(dual.u_vars_ptr, &dual.h_variables[0], sizeof(int)*numSupplies, hipMemcpyHostToDevice));
                gpuErrchk(hipMemcpy(dual.v_vars_ptr, &dual.h_variables[numSupplies], sizeof(int)*numDemands, hipMemcpyHostToDevice));
        }
}

} // End of namespace