#include "hip/hip_runtime.h"
#include "DUAL_solver.h"


__host__ void dualMalloc(DualHandler &dual, int numSupplies, int numDemands) {
    
    int V = numSupplies + numDemands;
    // Create and Initialize u and v variables 
    // To be allocated regardless 
    gpuErrchk(hipMalloc((void **) &dual.u_vars_ptr, sizeof(float)*numSupplies));
    gpuErrchk(hipMalloc((void **) &dual.v_vars_ptr, sizeof(float)*numDemands));

    if (CALCULATE_DUAL=="device_bfs") {

        //  empty u and v equations using the Variable Data Type >>
        gpuErrchk(hipMalloc((void **) &dual.Fa, sizeof(bool)*V));
        gpuErrchk(hipMalloc((void **) &dual.Xa, sizeof(bool)*V));
        gpuErrchk(hipMalloc((void **) &dual.variables, sizeof(float)*V));

    }

    else if (CALCULATE_DUAL=="host_bfs") {

        //  empty u and v equations using the Variable Data Type >>
        dual.h_visited = (bool *) malloc(sizeof(bool)*V);
        dual.h_variables = (float *) malloc(sizeof(float)*V);
    }

    else if (CALCULATE_DUAL=="device_sparse_linear_solver") {

        int U_0 = 0;
        float U_0_value = 0.0;

        // Allocate memory to store the sparse linear system
        dual.nnz = 2*V - 1;

        // Values are coefs of u and v, which are always one only position and b-vector changes with iterations, So
        gpuErrchk(hipMalloc((void **) &dual.d_csr_values,  dual.nnz * sizeof(float)));
        thrust::fill(thrust::device, dual.d_csr_values, (dual.d_csr_values) + dual.nnz, 1.0);

        // U_0 is always set to zero - meaning first element is always 0,0 in csr
        gpuErrchk(hipMalloc((void **) &dual.d_csr_columns, dual.nnz * sizeof(int)));
        gpuErrchk(hipMemcpy(dual.d_csr_columns, &U_0, sizeof(int), hipMemcpyHostToDevice));

        // The row pointers also remain constant {0,1,3,5, ... , 2V-1}, Custom Filler kernel below
        gpuErrchk(hipMalloc((void**) &dual.d_csr_offsets, (V + 1) * sizeof(int)));
        fill_csr_offset <<< ceil(1.0*(V+1)/blockSize), blockSize >>> (dual.d_csr_offsets, V+1);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        gpuErrchk(hipMalloc((void **) &dual.d_b, sizeof(float)*V));
        gpuErrchk(hipMemcpy(dual.d_b, &U_0_value, sizeof(float), hipMemcpyHostToDevice));
        
        // d_x is only allocated here - it is to be populated by API's
        gpuErrchk(hipMalloc((void **) &dual.d_x, V * sizeof(float)));
    }

    else if (CALCULATE_DUAL=="device_dense_linear_solver") {

        // Allocate memory to store the dense linear system
        gpuErrchk(hipMalloc((void **) &dual.d_A, sizeof(float)*V*V));
        gpuErrchk(hipMalloc((void **) &dual.d_b, sizeof(float)*V));
        gpuErrchk(hipMalloc((void **) &dual.d_x, V * sizeof(float)));
    }
}

__host__ void dualFree(DualHandler &dual) {
     
        gpuErrchk(hipFree(dual.u_vars_ptr));
        gpuErrchk(hipFree(dual.v_vars_ptr));
        
        if (CALCULATE_DUAL=="device_bfs") {

                gpuErrchk(hipFree(dual.Fa));
                gpuErrchk(hipFree(dual.Xa));
                gpuErrchk(hipFree(dual.variables));
        }

        else if (CALCULATE_DUAL=="host_bfs") {
                
                free(dual.h_visited);
                free(dual.h_variables);
        }
        
        else if (CALCULATE_DUAL=="device_sparse_linear_solver") {

        gpuErrchk(hipFree(dual.d_csr_values));
        gpuErrchk(hipFree(dual.d_csr_columns));
        gpuErrchk(hipFree(dual.d_csr_offsets));
        gpuErrchk(hipFree(dual.d_b));
        gpuErrchk(hipFree(dual.d_x));
        
        }
        
        else if (CALCULATE_DUAL=="device_dense_linear_solver") {

        gpuErrchk(hipFree(dual.d_A));
        gpuErrchk(hipFree(dual.d_b));
        gpuErrchk(hipFree(dual.d_x));

        }
}


__host__ void find_dual_using_sparse_solver(DualHandler &dual, Graph &graph, float * d_costs_ptr, int numSupplies, int numDemands)
{

        // Nice thing is that csr values and offsets remain static over the iterations
        // So a bunch of things are assigned here statically during the dualMalloc phase
        dim3 __blockDim(blockSize, blockSize, 1); 
        dim3 __gridDim(ceil(1.0*numDemands/blockSize), ceil(1.0*numSupplies/blockSize), 1);
        initialize_sparse_u_v_system <<< __gridDim, __blockDim >>> (dual.d_csr_columns, dual.d_b, graph.d_adjMtx_ptr, d_costs_ptr, 
                numSupplies, numDemands);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());


        /* *********************
        DEBUG UTILITY :: Print the csr matrix for u-v system
         ************************/
        // float * h_csr_values = (float *) malloc(sizeof(float)*nnz);
        // int * h_csr_columns = (int *) malloc(sizeof(int)*nnz);
        // int * h_csr_offsets = (int *) malloc(sizeof(int)*(V+1));
        // gpuErrchk(hipMemcpy(h_csr_values, d_csr_values, sizeof(float)*nnz, hipMemcpyDeviceToHost));
        // gpuErrchk(hipMemcpy(h_csr_columns, d_csr_columns, sizeof(int)*nnz, hipMemcpyDeviceToHost));
        // gpuErrchk(hipMemcpy(h_csr_offsets, d_csr_offsets, sizeof(int)*(V+1), hipMemcpyDeviceToHost));
        // std::cout<<"CSR Values = [";
        // for (int i =0; i< nnz; i++){
        //         std::cout<<h_csr_values[i]<<", ";
        // }
        // std::cout<<"]"<<std::endl;
        // std::cout<<"CSR Columns = [";
        // for (int i =0; i< nnz; i++){
        //         std::cout<<h_csr_columns[i]<<", ";
        // }
        // std::cout<<"]"<<std::endl;
        // std::cout<<"CSR Offsets = [";
        // for (int i =0; i < V+1; i++){
        //         std::cout<<h_csr_offsets[i]<<", ";
        // }
        // std::cout<<"]"<<std::endl;
        // exit(0);
        /* ********** END OF UTILITY ************* */

        // Core >>		
        hipsolverSpHandle_t solver_handle;
	CUSOLVER_CHECK(hipsolverSpCreate(&solver_handle));

        hipsparseMatDescr_t descrA;
        CUSPARSE_CHECK(hipsparseCreateMatDescr(&descrA));
	CUSPARSE_CHECK(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
	CUSPARSE_CHECK(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO)); 

        int singularity;
        // Cholesky : 1 : symrcm, 2 : symamd, or 3 : csrmetisnd
        int reorder = 0;

        if (SPARSE_SOLVER=="qr") {
                
                CUSOLVER_CHECK(hipsolverSpScsrlsvqr(solver_handle, graph.V, dual.nnz, descrA, 
                                        dual.d_csr_values, dual.d_csr_offsets, dual.d_csr_columns, dual.d_b, 
                                        10e-9, reorder, dual.d_x, &singularity));
        
        }

	else if (SPARSE_SOLVER=="chol") {

                CUSOLVER_CHECK(hipsolverSpScsrlsvchol(solver_handle, graph.V, dual.nnz, descrA,
                     dual.d_csr_values, dual.d_csr_offsets, dual.d_csr_columns, dual.d_b,
                     10e-9, reorder, dual.d_x, &singularity));
        }

        else {
        
                std::cout<<" Invalid sparse solver!"<<std::endl;
                exit(0);
        }
        
        // Clean up ! 
        CUSOLVER_CHECK(hipsolverSpDestroy(solver_handle));
        CUSPARSE_CHECK(hipsparseDestroyMatDescr(descrA));

        if (singularity == -1) {

                dim3 __blockDim2(blockSize, 1, 1);
                dim3 __gridDim2(ceil(1.0*graph.V/blockSize), 1, 1);
                retrieve_uv_solution <<< __gridDim2, __blockDim2 >>> (dual.d_x, dual.u_vars_ptr, dual.v_vars_ptr, numSupplies, numDemands);
                gpuErrchk(hipPeekAtLastError());
                gpuErrchk(hipDeviceSynchronize());
        }
        
        else {
        
                std::cout<<" ========== !! Unexpected ERROR :: Matrix A is singular !!"<<std::endl;
                std::cout<<" ========== Return singularity = "<<singularity<<std::endl;
                exit(0);
                // float * h_x = (float *) malloc(sizeof(float)*V);
                // hipMemcpy(h_x, d_x, sizeof(float)*V, hipMemcpyDeviceToHost);
                // for (int i=0; i<V; i++) {
                //     std::cout<< "X [" <<i<<"] = "<<h_x[i]<<std::endl;
                // }
        }
        // Make tree 
}


__host__ void find_dual_using_dense_solver(DualHandler &dual, Graph &graph, float * d_costs_ptr, 
        int numSupplies, int numDemands)
{

        thrust::fill(thrust::device, dual.d_A, dual.d_A + (graph.V * graph.V), 0.0f);
        thrust::fill(thrust::device, dual.d_b, dual.d_b + (graph.V), 0.0f);

        // Nice thing is that csr values and offsets remain static over the iterations
        dim3 __blockDim(blockSize, blockSize, 1); 
        dim3 __gridDim(ceil(1.0*numDemands/blockSize), ceil(1.0*numSupplies/blockSize), 1);
        initialize_dense_u_v_system <<< __gridDim, __blockDim >>> (dual.d_A, dual.d_b, 
                graph.d_adjMtx_ptr, d_costs_ptr, 
                numSupplies, numDemands);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
}


__host__ void find_dual_using_device_bfs(DualHandler &dual, Graph &graph, float * d_costs_ptr, int numSupplies, int numDemands) {

        bool f0 = true;
        // Initialize BFS >>
	thrust::fill(thrust::device, dual.Fa, dual.Fa + graph.V, false);
        thrust::fill(thrust::device, dual.Xa, dual.Xa + graph.V, false);
        thrust::fill(thrust::device, dual.variables, dual.variables + graph.V, 0.0);
        gpuErrchk(hipMemcpy(&dual.Fa[0], &f0, sizeof(bool), hipMemcpyHostToDevice));

        // >>> Running BFS
        // std::cout<<"Running BFS"<<std::endl;
        bool done;
	bool * d_done;
	gpuErrchk(hipMalloc((void**) &d_done, sizeof(bool)));
	int count = 0;
        dim3 __blockDim(blockSize, 1, 1); 
        dim3 __gridDim(ceil(1.0*graph.V/blockSize), 1, 1);

	do {
		count++;
		done = true;
		gpuErrchk(hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice));
		CUDA_BFS_KERNEL <<<__gridDim, __blockDim >>>(graph.d_vertex_start, &graph.d_vertex_degree[1], graph.d_adjVertices, 
                                        dual.Fa, dual.Xa, dual.variables, d_costs_ptr, 
                                        d_done, numSupplies, numDemands, graph.V);
		gpuErrchk(hipPeekAtLastError());
                gpuErrchk(hipDeviceSynchronize());
                gpuErrchk(hipMemcpy(&done, d_done , sizeof(bool), hipMemcpyDeviceToHost));

	} while (!done && count < (numSupplies+numDemands-1));

        // std::cout<<"BFS Complete!"<<std::endl;
	gpuErrchk(hipMemcpy(dual.u_vars_ptr, &dual.variables[0], sizeof(float)*numSupplies, hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(dual.v_vars_ptr, &dual.variables[numSupplies], sizeof(float)*numDemands, hipMemcpyDeviceToDevice));
}

__host__ void find_dual_using_host_bfs(DualHandler &dual,  Graph &graph, float * h_costs_ptr, 
        int numSupplies, int numDemands) {

        // Copy Adjacency list on host >> assuming Tranformation already occured at the start of pivoting 
        gpuErrchk(hipMemcpy(graph.h_vertex_degree, &graph.d_vertex_degree[1], sizeof(int)*graph.V, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(graph.h_vertex_start, graph.d_vertex_start, sizeof(int)*graph.V, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(graph.h_adjVertices, graph.d_adjVertices, sizeof(int)*2*(graph.V-1), hipMemcpyDeviceToHost));

        thrust::fill(thrust::host, dual.h_visited, dual.h_visited + graph.V, false);
        thrust::fill(thrust::host, dual.h_variables, dual.h_variables + graph.V, 0.0f);

        // Initialize >>
        std::queue<int> assigned_parents;
        dual.h_visited[0] = true;
        // The value of u0 is already zero as initialized
        assigned_parents.push(0);

        // Perform a BFS on Host (trickle down) >> 
        int parent, child, row, col;
        
        int * h_start = graph.h_vertex_start;
        int * h_length = graph.h_vertex_degree;
        int * h_Ea = graph.h_adjVertices;


        while (!assigned_parents.empty()) {
                parent = assigned_parents.front();
                for (int i = h_start[parent]; i < h_start[parent] + h_length[parent]; i++) {
                        child = h_Ea[i];
                        if (!dual.h_visited[child]) {
                                dual.h_visited[child] = true;
                                row = min(parent, child);
                                col = max(parent, child) - numSupplies;
                                dual.h_variables[child] = h_costs_ptr[row*numDemands + col] - dual.h_variables[parent];
                                assigned_parents.push(child);
                        }
                }
                assigned_parents.pop();
        }

        // Transfer back to GPU >> 
        gpuErrchk(hipMemcpy(dual.u_vars_ptr, &dual.h_variables[0], sizeof(int)*numSupplies, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(dual.v_vars_ptr, &dual.h_variables[numSupplies], sizeof(int)*numDemands, hipMemcpyHostToDevice));

}